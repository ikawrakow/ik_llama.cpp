#include "hip/hip_runtime.h"
//
// Copyright (C) 2024 Iwan Kawrakow
// MIT license
// SPDX-License-Identifier: MIT
//

#include "iqk_mmvq.cuh"
#include "iqk_cuda_common.h"
#include "mmvq-args.h"

typedef void (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float *);

template<>
struct ggml_cuda_type_traits<GGML_TYPE_IQ1_M_R4> {
    static constexpr int qk = 32;
    static constexpr int qr = 2;
    static constexpr int qi = 4;
};

//  Reminder:
//    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
//    constexpr int qi  = ggml_cuda_type_traits<type>::qi;
//    constexpr int vdr = get_vdr_mmvq(type);

// QI4_XS = 256/(4*2) = 32
// vdr = 4, qi = 32 -> qi/vdr = 8, kqs = 4*(tid%8),  blocks_per_iter = 4*1*32/32 = 4
// vdr = 2, qi = 32 -> qi/vdr =16, kqs = 2*(tid%16), blocks_per_iter = 2*1*32/32 = 2
namespace {
template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda, int ncols_y, int n_interleaved = 1>
__device__ void iqk_mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst, const int64_t row_size) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int nwarps              = 1;
    constexpr int rows_per_cuda_block = n_interleaved;
#else
    constexpr int nwarps              = n_interleaved == 1 ? ncols_y <= 4 ? 4 : 2 : 1;
    constexpr int rows_per_cuda_block = n_interleaved == 1 ? ncols_y == 1 ? 1 : 2 : n_interleaved;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
            if constexpr (n_interleaved == 1) {
#pragma unroll
                for (int i = 0; i < rows_per_cuda_block; ++i) {
                    vec_dot_q_cuda((const void *)((const char *)vx + (row0 + i)*row_size),
                            &y[j*blocks_per_col_y + kby], kbx, kqs, &tmp[j][i]);
                }
            } else {
                vec_dot_q_cuda((const void *)((const char *)vx + row0*row_size),
                    &y[j*blocks_per_col_y + kby], kbx, kqs, tmp[j]);
            }
        }
    }

    __shared__ float tmp_shared[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared[threadIdx.y-1][j][i][threadIdx.x] = tmp[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp[j][i] += tmp_shared[l][j][i][threadIdx.x];
            }
            tmp[j][i] = warp_reduce_sum(tmp[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + threadIdx.x < nrows_dst)) {
            dst[j*nrows_dst + row0 + threadIdx.x] = tmp[j][threadIdx.x];
        }
    }
}

template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda, int ncols_y, int n_interleaved = 1>
__device__ void iqk_fused_mul_mat_vec_q(
    const void * __restrict__ vup, const void * __restrict__ vgate, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst, const int64_t row_size,
    ggml_unary_op unary_op) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int nwarps              = 1;
    constexpr int rows_per_cuda_block = n_interleaved;
#else
    constexpr int nwarps              = n_interleaved == 1 ? ncols_y <= 4 ? 4 : 2 : 1;
    constexpr int rows_per_cuda_block = n_interleaved == 1 ? ncols_y == 1 ? 1 : 2 : n_interleaved;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp_u[ncols_y][rows_per_cuda_block] = {0.0f};
    float tmp_g[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
            if constexpr (n_interleaved == 1) {
#pragma unroll
                for (int i = 0; i < rows_per_cuda_block; ++i) {
                    vec_dot_q_cuda((const void *)((const char *)vup + (row0 + i)*row_size),
                            &y[j*blocks_per_col_y + kby], kbx, kqs, &tmp_u[j][i]);
                    vec_dot_q_cuda((const void *)((const char *)vgate + (row0 + i)*row_size),
                            &y[j*blocks_per_col_y + kby], kbx, kqs, &tmp_g[j][i]);
                }
            } else {
                vec_dot_q_cuda((const void *)((const char *)vup + row0*row_size),
                    &y[j*blocks_per_col_y + kby], kbx, kqs, tmp_u[j]);
                vec_dot_q_cuda((const void *)((const char *)vgate + row0*row_size),
                    &y[j*blocks_per_col_y + kby], kbx, kqs, tmp_u[j]);
            }
        }
    }

    __shared__ float tmp_shared_u[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    __shared__ float tmp_shared_g[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared_u[threadIdx.y-1][j][i][threadIdx.x] = tmp_u[j][i];
                tmp_shared_g[threadIdx.y-1][j][i][threadIdx.x] = tmp_g[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp_u[j][i] += tmp_shared_u[l][j][i][threadIdx.x];
                tmp_g[j][i] += tmp_shared_g[l][j][i][threadIdx.x];
            }
            tmp_u[j][i] = warp_reduce_sum(tmp_u[j][i]);
            tmp_g[j][i] = warp_reduce_sum(tmp_g[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + threadIdx.x < nrows_dst)) {
            float u = tmp_u[j][threadIdx.x];
            float g = tmp_g[j][threadIdx.x];
            float r;
            switch (unary_op) {
                case GGML_UNARY_OP_SILU: r = u*g/(1 + expf(-g)); break;
                case GGML_UNARY_OP_RELU: r = fmaxf(g, 0.0f) * u; break;
                // we assume that the supported ops have been checked by the caller
                default: {
                    constexpr float GELU_COEF_A    = 0.044715f;
                    constexpr float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
                    r = 0.5f*g*u*(1.0f + tanhf(SQRT_2_OVER_PI*g*(1.0f + GELU_COEF_A*g*g)));
                } break;
            }
            dst[j*nrows_dst + row0 + threadIdx.x] = r;
        }
    }
}

template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda, int ncols_y, int n_interleaved = 1>
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
// tell the compiler to use as many registers as it wants, see nwarps definition below
__launch_bounds__((ncols_y <= 4 ? 4 : 2)*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__global__ void iqk_mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst, const char * __restrict__ ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst, const int64_t row_size,
    const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, const int64_t ids_nb0) {
    int i2 = blockIdx.y;
    int i02 = ids_data ? *(const int *)(ids_data + i2*ids_nb0) : i2;
    if (i02 < 0) return;
    const char * cx = (const char *)vx + i02*nb02;
    const char * cy = (const char *)vy + i2*nb12;
    char * cdst = (char *)dst + i2*nb2;
    iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, ncols_y, n_interleaved>(cx, cy, (float *)cdst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
}

template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda, int n_interleaved = 1>
void iqk_mul_mat_vec_q_cuda(const mmvq_args & args, hipStream_t stream) {

    GGML_ASSERT(args.ncols_x % ggml_blck_size(type) == 0);
    //GGML_ASSERT(ncols_y <= MMVQ_MAX_BATCH_SIZE);

    int id = ggml_cuda_get_device();

    int64_t nwarps = 1;
    int64_t rows_per_cuda_block = n_interleaved;

    if (ggml_cuda_info().devices[id].cc < CC_RDNA2) { // NVIDIA and AMD older than RDNA2
        switch(args.ncols_y) {
            case 1:
                nwarps = n_interleaved == 1 ? 4 : 1;
                rows_per_cuda_block = n_interleaved == 1 ? 1 : n_interleaved;
                break;
            case 2:
            case 3:
            case 4:
                nwarps = n_interleaved == 1 ? 4 : 1;
                rows_per_cuda_block = n_interleaved == 1 ? 2 : n_interleaved;
                break;
            case 5:
            case 6:
            case 7:
            case 8:
                nwarps = n_interleaved == 1 ? 2 : 1;
                rows_per_cuda_block = n_interleaved == 1 ? 2 : n_interleaved;
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
    }
    const int64_t nblocks = (args.nrows_x + rows_per_cuda_block - 1) / rows_per_cuda_block;
    const dim3 block_nums(nblocks, args.ne2, 1);
    const dim3 block_dims(WARP_SIZE, nwarps, 1);

    const int64_t row_size = ggml_row_size(type, args.ncols_x);

    switch (args.ncols_y) {
        case 1:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 1, n_interleaved><<<block_nums, block_dims, 0, stream>>>(args.vx, args.vy, args.dst, args.ids_data, args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, row_size, args.nb02, args.nb12, args.nb2, args.ids_nb0);
            break;
        case 2:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 2, n_interleaved><<<block_nums, block_dims, 0, stream>>>(args.vx, args.vy, args.dst, args.ids_data, args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, row_size, args.nb02, args.nb12, args.nb2, args.ids_nb0);
            break;
        case 3:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 3, n_interleaved><<<block_nums, block_dims, 0, stream>>>(args.vx, args.vy, args.dst, args.ids_data, args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, row_size, args.nb02, args.nb12, args.nb2, args.ids_nb0);
            break;
        case 4:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 4, n_interleaved><<<block_nums, block_dims, 0, stream>>>(args.vx, args.vy, args.dst, args.ids_data, args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, row_size, args.nb02, args.nb12, args.nb2, args.ids_nb0);
            break;
        case 5:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 5, n_interleaved><<<block_nums, block_dims, 0, stream>>>(args.vx, args.vy, args.dst, args.ids_data, args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, row_size, args.nb02, args.nb12, args.nb2, args.ids_nb0);
            break;
        case 6:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 6, n_interleaved><<<block_nums, block_dims, 0, stream>>>(args.vx, args.vy, args.dst, args.ids_data, args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, row_size, args.nb02, args.nb12, args.nb2, args.ids_nb0);
            break;
        case 7:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 7, n_interleaved><<<block_nums, block_dims, 0, stream>>>(args.vx, args.vy, args.dst, args.ids_data, args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, row_size, args.nb02, args.nb12, args.nb2, args.ids_nb0);
            break;
        case 8:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 8, n_interleaved><<<block_nums, block_dims, 0, stream>>>(args.vx, args.vy, args.dst, args.ids_data, args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, row_size, args.nb02, args.nb12, args.nb2, args.ids_nb0);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

__device__ __forceinline__ void get_int_from_table_16_shift(const uint32_t & q4, uint16_t shift, const uint8_t * all_values,
        int & val1, int & val2) {

    uint32_t aux32; const uint8_t * q8 = (const uint8_t *)&aux32;
    aux32 = q4 & 0x0f0f0f0f;
    const uint8_t * values = all_values + 16*(shift & 1);
    uint16_t v1 = values[q8[0]] | (values[q8[1]] << 8);
    uint16_t v2 = values[q8[2]] | (values[q8[3]] << 8);
    val1 = v1 | (v2 << 16);
    aux32 = (q4 >> 4) & 0x0f0f0f0f;
    values = all_values + 8*(shift & 2);
    v1 = values[q8[0]] | (values[q8[1]] << 8);
    v2 = values[q8[2]] | (values[q8[3]] << 8);
    val2 = v1 | (v2 << 16);
}

#define VDR_IQ4_K_Q8_1_MMVQ 4
#define VDR_IQ4_K_Q8_1_MMQ  4

__device__ __forceinline__ void vec_dot_iq4_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq4_k * bq4 = (const block_iq4_k *) vbq + kbx;
    const uint8_t * all_values = (const uint8_t *)iq4k_values;

    // iqs is 0...28
    const int ib32 = iqs/4;
    // Why iqs/4 ?
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const uint16_t * q4 = (const uint16_t *)bq4->qs + 8*ib32;
    const uint16_t extra = bq4->extra >> 2*ib32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        const uint32_t aux32 = q4[2*j+0] | (q4[2*j+1] << 16);
        get_int_from_table_16_shift(aux32, extra, all_values, v1, v2);
        sumi1 = ggml_cuda_dp4a(v1, q8[j+0], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8[j+4], sumi2);
    }
    const float d = __half2float(bq4->d) * __low2float(bq8_1[ib32].ds);
    const uint8_t sh = bq4->scales_h[ib32/2] >> 4*(ib32%2);
    const int ls1 = ((bq4->scales_l[ib32] & 0xf) | ((sh << 4) & 0x30)) - 32;
    const int ls2 = ((bq4->scales_l[ib32] >>  4) | ((sh << 2) & 0x30)) - 32;
    *result += d * (sumi1 * ls1 + sumi2 * ls2);
}

static __device__ __forceinline__ int2 get_int_from_table_16(const int & q4, const int8_t * values) {
#if defined(__CUDA_ARCH__)
    uint32_t v1, v2, v3, v4, mask;
    const uint32_t * values32 = (const uint32_t *)values;

    mask = (0x32103210 | ((q4 & 0x88888888) >> 1));
    // Perform lookups in the lower half of the table (indices 0-7).
    v1 = __byte_perm(values32[0], values32[1], q4);
    // Perform lookups in the upper half of the table (indices 8-15).
    v2 = __byte_perm(values32[2], values32[3], q4);
    // Select between the low and high results based on the MSB of each index nibble.
    v3 = __byte_perm(v1, v2, mask);
    // Same for the upper part of q4.
    v1 = __byte_perm(values32[0], values32[1], q4 >> 16);
    v2 = __byte_perm(values32[2], values32[3], q4 >> 16);
    v4 = __byte_perm(v1, v2, mask >> 16);

    // Mix the results to get the final int2.
    return make_int2(__byte_perm(v3, v4, 0x6420), __byte_perm(v3, v4, 0x7531));
#else
    const int      q0_32  = (q4 >> 0) & 0x0F0F0F0F;
    const int8_t * q0_8   = (const int8_t *) &q0_32;
    const char4    val0_8 = make_char4(values[q0_8[0]], values[q0_8[1]], values[q0_8[2]], values[q0_8[3]]);

    const int      q1_32  = (q4 >> 4) & 0x0F0F0F0F;
    const int8_t * q1_8   = (const int8_t *) &q1_32;
    const char4    val1_8 = make_char4(values[q1_8[0]], values[q1_8[1]], values[q1_8[2]], values[q1_8[3]]);

    return make_int2(*((const int *) &val0_8), *((const int *) &val1_8));
#endif
}

__device__ __forceinline__ void vec_dot_iq4_k_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq4_k_r4 * bq4 = (const block_iq4_k_r4 *)vbq + kbx;

    // iqs is 0...28 in steps of 2
    const int ib16 = iqs/2;
    const float d8 = __low2float(bq8_1[ib16/2].ds);
    const int32_t  * q8 = (const int *)bq8_1[ib16/2].qs + 4*(ib16%2);

    int ib32 = ib16/2;
    int is   = ib16%2;
    int scales;
    const uint32_t * scales_l = (const uint32_t *)bq4->scales_l;
    const uint32_t * scales_h = (const uint32_t *)bq4->scales_h;
    scales = __vsub4(((scales_l[2*(ib32%4)+is] >> 4*(ib32/4)) & 0x0f0f0f0f) | (((scales_h[2*(ib32%2)+is] >> 2*(ib32/2)) & 0x03030303) << 4), 0x20202020);
    const int8_t * s8 = (const int8_t *)&scales;
    int2 val1;
    const int * q4 = (const int *)bq4->qs + 16*ib32;
    for (int i = 0; i < 4; ++i) {
        auto values1 = iq4k_values + (((bq4->extra[i+4*is] >> ib32) & 1) << 4);
        int sumi1 = 0;
        val1  = get_int_from_table_16(q4[i+4*is+0], values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[0], ggml_cuda_dp4a(val1.y, q8[2], sumi1));
        val1  = get_int_from_table_16(q4[i+4*is+8], values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[1], ggml_cuda_dp4a(val1.y, q8[3], sumi1));
        const float d = __half2float(bq4->d[i]) * d8;
        result[i] += d * sumi1 * s8[i];
    }
}

__device__ __forceinline__ void vec_dot_iq4_ks_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const float * dptr = (const float *)vbq;
    const block_iq4_ks_r4 * bq4 = (const block_iq4_ks_r4 *)(dptr + 4) + kbx;

    // iqs is 0...28 in steps of 2
    const int ib16 = iqs/2;
    const float d8 = __low2float(bq8_1[ib16/2].ds);
    const int32_t  * q8 = (const int *)bq8_1[ib16/2].qs + 4*(ib16%2);

    int ib32 = ib16/2;
    int is   = ib16%2;
    const uint32_t * scales32 = (const uint32_t *)bq4->scales;
    int scales = __vsub4(scales32[ib32] & 0xfefefefe, 0x7f7f7f7f);
    const int8_t * s8 = (const int8_t *)&scales;
    int2 val;
    const int * q4 = (const int *)bq4->qs + 16*ib32;
    for (int i = 0; i < 4; ++i) {
        auto values = iq4k_values + ((bq4->scales[4*ib32+i] & 1) << 4);
        int sumi = 0;
        val  = get_int_from_table_16(q4[i+4*is+0], values);
        sumi = ggml_cuda_dp4a(val.x, q8[0], ggml_cuda_dp4a(val.y, q8[2], sumi));
        val  = get_int_from_table_16(q4[i+4*is+8], values);
        sumi = ggml_cuda_dp4a(val.x, q8[1], ggml_cuda_dp4a(val.y, q8[3], sumi));
        const float d = dptr[i] * d8;
        result[i] += d * sumi * s8[i];
    }
}

__device__ __forceinline__ void vec_dot_iq1_s_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const half * dptr = (const half *)vbq;
    const block_iq1_s_r4 * bq1 = (const block_iq1_s_r4 *)(dptr + 4) + kbx;

    // iqs is 0 or 2
    const float d8 = __low2float(bq8_1->ds);
    const int32_t  * q8 = (const int *)bq8_1->qs;

    int32_t grid32[2];
    const int * igrid = (const int *)grid32;

    int minus = 0;
    for (int k = 0; k < 4; ++k) minus = ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+k], minus);

    for (int i = 0; i < 4; ++i) {
        float dl = __half2float(dptr[i])*(2*((bq1->qh[i] >> 12) & 7) + 1) * d8;
        float ml = dl * (bq1->qh[i] & 0x8000 ? -1-IQ1S_DELTA : -1+IQ1S_DELTA);
        grid32[0] = iq1s_grid_gpu[bq1->qs[4*iqs+i] | (((bq1->qh[i] >> 3*iqs) & 7) << 8)];
        grid32[1] = (grid32[0] >> 4) & 0x0f0f0f0f;
        grid32[0] &= 0x0f0f0f0f;
        int sumi = ggml_cuda_dp4a(igrid[0], q8[4*(iqs/2)+0], ggml_cuda_dp4a(igrid[1], q8[4*(iqs/2)+1], 0));
        grid32[0] = iq1s_grid_gpu[bq1->qs[4*iqs+i+4] | (((bq1->qh[i] >> (3*iqs+3)) & 7) << 8)];
        grid32[1] = (grid32[0] >> 4) & 0x0f0f0f0f;
        grid32[0] &= 0x0f0f0f0f;
        sumi = ggml_cuda_dp4a(igrid[0], q8[4*(iqs/2)+2], ggml_cuda_dp4a(igrid[1], q8[4*(iqs/2)+3], sumi));
        result[i] += dl * sumi + ml * minus;
    }
}

__device__ __forceinline__ void vec_dot_iq1_m_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const half * dptr = (const half *)vbq;
    const block_iq1_m_r4 * bq1 = (const block_iq1_m_r4 *)(dptr + 4) + kbx;

    // iqs is 0 or 2
    const float d8 = __low2float(bq8_1->ds);
    const int32_t  * q8 = (const int *)bq8_1->qs;

    int32_t grid32[2];
    const int * igrid = (const int *)grid32;

    int minus1 = ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+0], ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+1], 0));
    int minus2 = ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+2], ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+3], 0));

    for (int i = 0; i < 4; ++i) {
        float dl = __half2float(dptr[i])*((bq1->scales[i] >> 4*(iqs/2)) & 0xf) * d8;
        float ml1 = dl * (bq1->qh[4*(iqs/2)+i] & 0x08 ? -1-IQ1M_DELTA : -1+IQ1M_DELTA);
        float ml2 = dl * (bq1->qh[4*(iqs/2)+i] & 0x80 ? -1-IQ1M_DELTA : -1+IQ1M_DELTA);
        grid32[0] = iq1s_grid_gpu[bq1->qs[4*iqs+i] | ((bq1->qh[4*(iqs/2)+i] & 0x07) << 8)];
        grid32[1] = (grid32[0] >> 4) & 0x0f0f0f0f;
        grid32[0] &= 0x0f0f0f0f;
        int sumi = ggml_cuda_dp4a(igrid[0], q8[4*(iqs/2)+0], ggml_cuda_dp4a(igrid[1], q8[4*(iqs/2)+1], 0));
        grid32[0] = iq1s_grid_gpu[bq1->qs[4*iqs+i+4] | ((bq1->qh[4*(iqs/2)+i] & 0x70) << 4)];
        grid32[1] = (grid32[0] >> 4) & 0x0f0f0f0f;
        grid32[0] &= 0x0f0f0f0f;
        sumi = ggml_cuda_dp4a(igrid[0], q8[4*(iqs/2)+2], ggml_cuda_dp4a(igrid[1], q8[4*(iqs/2)+3], sumi));
        result[i] += dl * sumi + ml1 * minus1 + ml2*minus2;
    }
}

#define VDR_IQ4_KS_Q8_1_MMVQ 4
#define VDR_IQ4_KS_Q8_1_MMQ  4

__device__ __forceinline__ void vec_dot_iq4_ks_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    float scale = *(const float *)vbq;
    const block_iq4_ks * bq4 = (const block_iq4_ks *)((const char *)vbq + sizeof(float)) + kbx;

    // iqs is 0...28
    const int ib32 = iqs/4; // Why iqs/4 ?
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const uint32_t * q4 = (const uint32_t *)bq4->qs + 4*ib32;
    const float dl = scale * ((bq4->scales[ib32] & 254) - 127);
    auto values = iq4k_values + ((bq4->scales[ib32] & 1) << 4);
    int sumi = 0;
    for (int j = 0; j < 4; ++j) {
        auto v = get_int_from_table_16(q4[j], values);
        sumi = ggml_cuda_dp4a(v.x, q8[j+0], sumi);
        sumi = ggml_cuda_dp4a(v.y, q8[j+4], sumi);
    }
    *result += dl * __low2float(bq8_1[ib32].ds) * sumi;
}

__device__ __forceinline__ void vec_dot_iq4_kt_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    constexpr uint32_t ka = 0xCBAC1FED;
    constexpr uint32_t km = 0x3f3f3f3f;

    float scale = *(const float *)vbq;
    const block_iq4_kt * bq4 = (const block_iq4_kt *)((const char *)vbq + sizeof(float)) + kbx;

    // iqs is 0...28
    const int ib32 = iqs/4; // Why iqs/4 ?
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    //const int8_t  * q8 = bq8_1[ib32].qs;
    const int ls = (bq4->qs[ib32] & 0xff) >> 1;
    const float dl = scale * (ls - 64);
    const uint32_t idx0 = ((bq4->qs[ib32] & 1) << 15) + 4096;
    auto ql = (const uint8_t *)(bq4->qs + 8);
    auto qh = ql + 64;
    ql += 8*ib32;
    qh += 8*(ib32%4);
    const int shift1 = 8 - 4*(ib32/4);
    int sumi = 0;
    for (int j = 0; j < 8; ++j) {
        const uint32_t sh = bq4->qs[ib32] >> (8 + 3*j);
        uint32_t val = ql[j] + ((qh[j] << shift1) & 0xf00) + ((sh & 7) << 12) + idx0;
        int v4 = 0;
        for (int k = 0; k < 4; ++k) {
            val *= ka;
            //int s = val & km;
            //sumi += q8[4*j+k] * ggml_cuda_dp4a(s, 0x01010101, -126);
            v4 |= (ggml_cuda_dp4a(val & km, 0x01010101, -126) & 0xff) << 8*k;
        }
        sumi = ggml_cuda_dp4a(v4, q8[j], sumi);
    }
    *result += dl * __low2float(bq8_1[ib32].ds) * sumi;
}

__device__ __forceinline__ void vec_dot_iq1_kt_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    constexpr uint32_t ka = 0xCBAC1FED;
    constexpr uint32_t km = 0x3f3f3f3f;

    float scale = *(const float *)vbq;
    const block_iq1_kt * bq1 = (const block_iq1_kt *)((const char *)vbq + sizeof(float)) + kbx;

    // iqs is 0...28
    const int ib32 = iqs/4;
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const int ls = iq4k_values[bq1->sh[ib32] & 0xf];
    const float dl = scale * ls;
    int sumi = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t val = bq1->ql[4*ib32+j] + 4096 + ((bq1->qh[4*(ib32%4)+j] << (8 - 4*(ib32/4))) & 0xf00) + ((bq1->sh[ib32] << (8 - j)) & 0x1000);
        int v4 = 0;
        for (int k = 0; k < 4; ++k) {
            val *= ka;
            v4 |= (ggml_cuda_dp4a(val & km, 0x01010101, -126) & 0xff) << 8*k;
        }
        sumi = ggml_cuda_dp4a(v4, q8[2*j+0], sumi);
        v4 = 0;
        for (int k = 0; k < 4; ++k) {
            val *= ka;
            v4 |= (ggml_cuda_dp4a(val & km, 0x01010101, -126) & 0xff) << 8*k;
        }
        sumi = ggml_cuda_dp4a(v4, q8[2*j+1], sumi);
    }
    *result += dl * __low2float(bq8_1[ib32].ds) * sumi;
}

__device__ __forceinline__ void vec_dot_iq2_kt_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    constexpr uint32_t ka = 0xCBAC1FED;
    constexpr uint32_t km = 0x3f3f3f3f;

    float scale = *(const float *)vbq;
    const block_iq2_kt * bq2 = (const block_iq2_kt *)((const char *)vbq + sizeof(float)) + kbx;

    // iqs is 0...28
    const int ib32 = iqs/4;
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const int ls = iq4k_values[(bq2->scales[ib32%4] >> 4*(ib32/4)) & 0xf];
    const float dl = scale * ls * 1.05f;
    auto ql = (const uint16_t *)bq2->ql;
    int sumi = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t val = ql[4*ib32+j] + 4096;
        int v4 = 0;
        for (int k = 0; k < 4; ++k) {
            val *= ka;
            v4 |= (ggml_cuda_dp4a(val & km, 0x01010101, -126) & 0xff) << 8*k;
        }
        sumi = ggml_cuda_dp4a(v4, q8[2*j+0], sumi);
        v4 = 0;
        for (int k = 0; k < 4; ++k) {
            val *= ka;
            v4 |= (ggml_cuda_dp4a(val & km, 0x01010101, -126) & 0xff) << 8*k;
        }
        sumi = ggml_cuda_dp4a(v4, q8[2*j+1], sumi);
    }
    *result += dl * __low2float(bq8_1[ib32].ds) * sumi;
}

__device__ __forceinline__ void vec_dot_iq3_kt_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    constexpr uint32_t ka = 0xCBAC1FED;
    constexpr uint32_t km = 0x3f3f3f3f;

    float scale = *(const float *)vbq;
    const block_iq3_kt * bq3 = (const block_iq3_kt *)((const char *)vbq + sizeof(float)) + kbx;

    // iqs is 0...28
    const int ib32 = iqs/4;
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const int ls = (bq3->scales[ib32%4] >> 4*(ib32/4)) & 0xf;
    const float dl = scale * ls * 1.015f;
    auto ql = (const uint16_t *)bq3->ql;
    uint32_t mask = 0x01010101 << ib32;
    const uint32_t * qh = (const uint32_t *)bq3->qh;
    int sumi = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t val = ql[4*ib32+j] + 4096;
        int v4 = 0;
        for (int k = 0; k < 4; ++k) {
            val *= ka;
            int8_t q = std::abs(ggml_cuda_dp4a(val & km, 0x01010101, -126));
            v4 |= q << 8*k;
        }
        uint32_t signs = __vcmpne4(qh[2*j+0] & mask, 0);
        v4 = __vsub4(v4 ^ signs, signs);
        sumi = ggml_cuda_dp4a(v4, q8[2*j+0], sumi);
        v4 = 0;
        for (int k = 0; k < 4; ++k) {
            val *= ka;
            int8_t q = std::abs(ggml_cuda_dp4a(val & km, 0x01010101, -126));
            v4 |= q << 8*k;
        }
        signs = __vcmpne4(qh[2*j+1] & mask, 0);
        v4 = __vsub4(v4 ^ signs, signs);
        sumi = ggml_cuda_dp4a(v4, q8[2*j+1], sumi);
    }
    *result += dl * __low2float(bq8_1[ib32].ds) * sumi;
}

#define VDR_IQ4_KSS_Q8_1_MMVQ 4
#define VDR_IQ4_KSS_Q8_1_MMQ  4

__device__ __forceinline__ void vec_dot_iq4_kss_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    float scale = *(const float *)vbq;
    const block_iq4_kss * bq4 = (const block_iq4_kss *)((const char *)vbq + sizeof(float)) + kbx;

    // iqs is 0...28
    const int ib32 = iqs/4; // Why iqs/4 ?
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const uint32_t * q4 = (const uint32_t *)bq4->qs + 4*ib32;
    uint32_t s32 = (q4[0] & 0x00010001) | ((q4[1] & 0x00010001) << 2) | ((q4[2] & 0x00010001) << 4) | ((q4[3] & 0x00010001) << 6);
    uint8_t ls = (s32 | (s32 >> 15)) & 0xff;
    const float dl = scale * ((ls & 254) - 127);
    auto values = iq4k_values + ((ls & 1) << 4);
    int sumi = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t aux32 = q4[j] & 0xfffefffe;
        aux32 ^= (aux32 >> 1);
        auto v = get_int_from_table_16(aux32, values);
        sumi = ggml_cuda_dp4a(v.x, q8[j+0], sumi);
        sumi = ggml_cuda_dp4a(v.y, q8[j+4], sumi);
    }
    *result += dl * __low2float(bq8_1[ib32].ds) * sumi;
}

#define VDR_IQ5_K_Q8_1_MMVQ 4
#define VDR_IQ5_K_Q8_1_MMQ  4

__device__ __forceinline__ int int_from_table(const uint8_t * a8, const uint8_t * values) {
    uint16_t v1 = values[a8[0]] | (values[a8[1]] << 8);
    uint16_t v2 = values[a8[2]] | (values[a8[3]] << 8);
    return v1 | (v2 << 16);
}

__device__ __forceinline__ void vec_dot_iq5_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq5_k * bq5 = (const block_iq5_k *) vbq + kbx;
    const uint8_t * all_values = (const uint8_t *)iq5nl_values;

    int i4 = iqs/4;  // 0...7.  Blocks of 16 index is 4*(i4/2) + (i4%2) + (0 and 2)

    const int32_t  * q8_1 = (const int *)bq8_1[2*(i4/2)+0].qs + 4*(i4%2);
    const int32_t  * q8_2 = (const int *)bq8_1[2*(i4/2)+1].qs + 4*(i4%2);
    const uint32_t * q4 = (const uint32_t *)bq5->qs + 8*(i4/2) + 4*(i4%2);
    const uint32_t * qh = (const uint32_t *)bq5->qh + 4*(i4%2);
    const uint16_t extra = bq5->extra >> (4*(i4/2) + (i4%2));
    const uint8_t * values1 = all_values + 32*(extra & 1);
    const uint8_t * values2 = all_values +  8*(extra & 4);
    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)aux32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t h = qh[j] >> 2*(i4/2);
        aux32[0] = ((q4[j] >> 0) & 0x0f0f0f0f) | ((h << 4) & 0x10101010);
        aux32[1] = ((q4[j] >> 4) & 0x0f0f0f0f) | ((h << 3) & 0x10101010);
        v1 = int_from_table(a8+0, values1);
        v2 = int_from_table(a8+4, values2);
        sumi1 = ggml_cuda_dp4a(v1, q8_1[j], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8_2[j], sumi2);
    }
    const float d5 = __half2float(bq5->d);
    const uint8_t sh = bq5->scales_h[i4/2] >> 2*(i4%2);
    const int ls1 = (((bq5->scales_l[2*(i4/2)+0] >> 4*(i4%2)) & 0xf) | ((sh << 4) & 0x30)) - 32;
    const int ls2 = (((bq5->scales_l[2*(i4/2)+1] >> 4*(i4%2)) & 0xf) | ((sh << 0) & 0x30)) - 32;
    *result += d5 * (__low2float(bq8_1[2*(i4/2)+0].ds) * sumi1 * ls1 + __low2float(bq8_1[2*(i4/2)+1].ds) * sumi2 * ls2);
}

__device__ __forceinline__ void vec_dot_iq5_k_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq5_k_r4 * bq5 = (const block_iq5_k_r4 *)vbq + kbx;

    // iqs is 0...28 in steps of 2
    const int ib16 = iqs/2;
    const float d8 = __low2float(bq8_1[ib16/2].ds);
    const int32_t  * q8 = (const int *)bq8_1[ib16/2].qs + 4*(ib16%2);

    int ib32 = ib16/2;
    int is   = ib16%2;
    int scales;
    const uint32_t * scales_l = (const uint32_t *)bq5->scales_l;
    const uint32_t * scales_h = (const uint32_t *)bq5->scales_h;
    scales = __vsub4(((scales_l[2*(ib32%4)+is] >> 4*(ib32/4)) & 0x0f0f0f0f) | (((scales_h[2*(ib32%2)+is] >> 2*(ib32/2)) & 0x03030303) << 4), 0x20202020);
    const int8_t * s8 = (const int8_t *)&scales;
    int2 val1;
    const int * q4 = (const int *)bq5->qs + 16*ib32;
    const int * qh = (const int *)bq5->qh +  4*ib32;
    int aux32[2];
    const uint8_t * aux8 = (const uint8_t *)aux32;
    for (int i = 0; i < 4; ++i) {
        auto values1 = iq5nl_values + (((bq5->extra[i+4*is] >> ib32) & 1) << 5);
        int sumi1 = 0;
        aux32[0] = ((q4[i+4*is+0] >> 0) & 0x0f0f0f0f) | (((qh[i] >> (2*is+0)) & 0x01010101) << 4);
        aux32[1] = ((q4[i+4*is+0] >> 4) & 0x0f0f0f0f) | (((qh[i] >> (2*is+1)) & 0x01010101) << 4);
        val1.x  = int_from_table(aux8+0, (const uint8_t *)values1);
        val1.y  = int_from_table(aux8+4, (const uint8_t *)values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[0], ggml_cuda_dp4a(val1.y, q8[2], sumi1));
        aux32[0] = ((q4[i+4*is+8] >> 0) & 0x0f0f0f0f) | (((qh[i] >> (2*is+4)) & 0x01010101) << 4);
        aux32[1] = ((q4[i+4*is+8] >> 4) & 0x0f0f0f0f) | (((qh[i] >> (2*is+5)) & 0x01010101) << 4);
        val1.x  = int_from_table(aux8+0, (const uint8_t *)values1);
        val1.y  = int_from_table(aux8+4, (const uint8_t *)values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[1], ggml_cuda_dp4a(val1.y, q8[3], sumi1));
        const float d = __half2float(bq5->d[i]) * d8;
        result[i] += d * sumi1 * s8[i];
    }
}

__device__ __forceinline__ void vec_dot_iq5_ks_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const float * dptr = (const float *)vbq;
    const block_iq5_ks_r4 * bq5 = (const block_iq5_ks_r4 *)(dptr + 4) + kbx;

    // iqs is 0...28 in steps of 2
    const int ib16 = iqs/2;
    const float d8 = __low2float(bq8_1[ib16/2].ds);
    const int32_t  * q8 = (const int *)bq8_1[ib16/2].qs + 4*(ib16%2);

    int ib32 = ib16/2;
    int is   = ib16%2;
    const uint32_t * scales32 = (const uint32_t *)bq5->scales;
    int scales = __vsub4(scales32[ib32] & 0xfefefefe, 0x7f7f7f7f);
    const int8_t * s8 = (const int8_t *)&scales;
    int2 val;
    const int * q4 = (const int *)bq5->qs + 16*ib32;
    const int * qh = (const int *)bq5->qh +  4*ib32;
    int aux32[2];
    const uint8_t * aux8 = (const uint8_t *)aux32;
    for (int i = 0; i < 4; ++i) {
        auto values = iq5nl_values + ((bq5->scales[4*ib32+i] & 1) << 5);
        int sumi = 0;
        aux32[0] = ((q4[i+4*is+0] >> 0) & 0x0f0f0f0f) | (((qh[i] >> (2*is+0)) & 0x01010101) << 4);
        aux32[1] = ((q4[i+4*is+0] >> 4) & 0x0f0f0f0f) | (((qh[i] >> (2*is+1)) & 0x01010101) << 4);
        val.x  = int_from_table(aux8+0, (const uint8_t *)values);
        val.y  = int_from_table(aux8+4, (const uint8_t *)values);
        sumi = ggml_cuda_dp4a(val.x, q8[0], ggml_cuda_dp4a(val.y, q8[2], sumi));
        aux32[0] = ((q4[i+4*is+8] >> 0) & 0x0f0f0f0f) | (((qh[i] >> (2*is+4)) & 0x01010101) << 4);
        aux32[1] = ((q4[i+4*is+8] >> 4) & 0x0f0f0f0f) | (((qh[i] >> (2*is+5)) & 0x01010101) << 4);
        val.x  = int_from_table(aux8+0, (const uint8_t *)values);
        val.y  = int_from_table(aux8+4, (const uint8_t *)values);
        sumi = ggml_cuda_dp4a(val.x, q8[1], ggml_cuda_dp4a(val.y, q8[3], sumi));
        result[i] += dptr[i] * d8 * sumi * s8[i];
    }
}

__device__ __forceinline__ void vec_dot_iq5_ks_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    float scale = *(const float *)vbq;
    const block_iq5_ks * bq5 = (const block_iq5_ks *)((const char *)vbq + sizeof(float)) + kbx;
    const uint8_t * all_values = (const uint8_t *)iq5nl_values;

    int i4 = iqs/4;  // 0...7.  Blocks of 16 index is 4*(i4/2) + (i4%2) + (0 and 2)

    const int32_t  * q8_1 = (const int *)bq8_1[2*(i4/2)+0].qs + 4*(i4%2);
    const int32_t  * q8_2 = (const int *)bq8_1[2*(i4/2)+1].qs + 4*(i4%2);
    const uint32_t * q4 = (const uint32_t *)bq5->qs + 8*(i4/2) + 4*(i4%2);
    const uint32_t * qh = (const uint32_t *)bq5->qh + 4*(i4%2);
    const uint8_t * values1 = all_values + ((bq5->scales[2*(i4/2)+0] & 1) << 5);
    const uint8_t * values2 = all_values + ((bq5->scales[2*(i4/2)+1] & 1) << 5);
    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)aux32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t h = qh[j] >> 2*(i4/2);
        aux32[0] = ((q4[j] >> 0) & 0x0f0f0f0f) | ((h << 4) & 0x10101010);
        aux32[1] = ((q4[j] >> 4) & 0x0f0f0f0f) | ((h << 3) & 0x10101010);
        v1 = int_from_table(a8+0, values1);
        v2 = int_from_table(a8+4, values2);
        sumi1 = ggml_cuda_dp4a(v1, q8_1[j], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8_2[j], sumi2);
    }
    const int ls1 = (bq5->scales[2*(i4/2)+0] & 254) - 127;
    const int ls2 = (bq5->scales[2*(i4/2)+1] & 254) - 127;
    *result += scale * (__low2float(bq8_1[2*(i4/2)+0].ds) * sumi1 * ls1 + __low2float(bq8_1[2*(i4/2)+1].ds) * sumi2 * ls2);
}

__device__ __forceinline__ void vec_dot_iq3_k_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq3_k_r4 * bq3 = (const block_iq3_k_r4 *)vbq + kbx;

    // iqs is 0...30 in steps of 2
    const int ib16 = iqs/2;
    const float d8 = __low2float(bq8_1[ib16/2].ds);
    const int32_t  * q8 = (const int *)bq8_1[ib16/2].qs + 4*(ib16%2);

    int ib32 = ib16/2;
    int is   = ib16%2;
    int scales[2];
    const uint32_t * scales_l = (const uint32_t *)bq3->scales_l;
    const uint32_t * scales_h = (const uint32_t *)bq3->scales_h;

    scales[0] = (((scales_l[2*(ib32%4)+is] >> 4*(ib32/4)) & 0x0f0f0f0f) << 1) | 0x01010101;
    scales[1] = (scales_h[is] >> ib32) & 0x01010101;
    // This is not faster. Why?
    //scales[1] = __vcmpeq4((scales_h[is] >> ib32) & 0x01010101, 0x01010101);
    //scales[0] = __vsub4(scales[0] ^ scales[1], scales[1]);
    const int8_t * s8 = (const int8_t *)scales;
    const uint32_t * q2 = (const uint32_t *)bq3->qs + 8*ib32 + 4*is;
    const uint32_t * qh = (const uint32_t *)bq3->qh + 4*ib32;
    for (int i = 0; i < 4; ++i) {
        uint32_t extra32 = uint32_t((bq3->extra[i+4*is] >> ib32) & 1) * 0x88888888;

        int sumi1 = 0;
        uint32_t h = qh[i] >> 4*is;
        uint32_t val1 = ((q2[i] >> 0) & 0x33333333) | extra32 | ((h << 2) & 0x04040404) | ((h << 4) & 0x40404040);
        uint32_t val2 = ((q2[i] >> 2) & 0x33333333) | extra32 | ((h << 1) & 0x04040404) | ((h << 3) & 0x40404040);
        int2 v1 = get_int_from_table_16(val1, iq3nl_values);
        int2 v2 = get_int_from_table_16(val2, iq3nl_values);
        sumi1 = ggml_cuda_dp4a(v1.x, q8[0], ggml_cuda_dp4a(v2.x, q8[1], sumi1));
        sumi1 = ggml_cuda_dp4a(v1.y, q8[2], ggml_cuda_dp4a(v2.y, q8[3], sumi1));
        const float d = __half2float(bq3->d[i]) * d8;
        result[i] += d * sumi1 * s8[i] * (s8[i+4] ? -1 : 1);
    }
}

#define VDR_IQ6_K_Q8_1_MMVQ 4
#define VDR_IQ6_K_Q8_1_MMQ  4

__device__ __forceinline__ void vec_dot_iq6_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq6_k * bq6 = (const block_iq6_k *) vbq + kbx;
    const uint8_t * all_values = (const uint8_t *)iq6nl_values;

    int i4 = iqs/4;  // 0...7.  Blocks of 16 index is 4*(i4/2) + (i4%2) + (0 and 2)
                     //         Blocks of 32 index is 2*(i4/2) + 0 or 1

    const int32_t  * q8_1 = (const int *)bq8_1[2*(i4/2)+0].qs + 4*(i4%2);
    const int32_t  * q8_2 = (const int *)bq8_1[2*(i4/2)+1].qs + 4*(i4%2);
    const uint32_t * q4 = (const uint32_t *)bq6->qs + 8*(i4/2) + 4*(i4%2);
    const uint32_t * qh = (const uint32_t *)bq6->qh + 8*(i4/4) + 4*(i4%2);
    const uint16_t extra = bq6->extra >> (4*(i4/2) + (i4%2));
    const uint8_t * values1 = all_values + 64*(extra & 1);
    const uint8_t * values2 = all_values + 16*(extra & 4);
    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)aux32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t h = qh[j] >> 4*((i4/2)%2);
        aux32[0] = ((q4[j] >> 0) & 0x0f0f0f0f) | ((h << 4) & 0x30303030);
        aux32[1] = ((q4[j] >> 4) & 0x0f0f0f0f) | ((h << 2) & 0x30303030);
        v1 = int_from_table(a8+0, values1);
        v2 = int_from_table(a8+4, values2);
        sumi1 = ggml_cuda_dp4a(v1, q8_1[j], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8_2[j], sumi2);
    }
    const float d6 = __half2float(bq6->d);
    *result += d6 * (__low2float(bq8_1[2*(i4/2)+0].ds) * sumi1 * bq6->scales[4*(i4/2)+(i4%2)] + __low2float(bq8_1[2*(i4/2)+1].ds) * sumi2 * bq6->scales[4*(i4/2)+(i4%2)+2]);
}

#define VDR_IQ2_K_Q8_1_MMVQ 4
#define VDR_IQ2_K_Q8_1_MMQ  4

__device__ __forceinline__ void vec_dot_iq2_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    // iqs is 0, 4, 8, 12, 16, 20, 24, 28
    // we have 16 packed quants (when cast to int)

    int i4 = iqs/4;  // 0...7. We will process q8 blocks 4*(i4/4), 4*(i4/4)+1, 4*(i4/4)+2, 4*(i4/4)+3
    const int32_t  * q8_1 = (const int *)bq8_1[4*(i4/4)+0].qs + 2*(i4%4);
    const int32_t  * q8_2 = (const int *)bq8_1[4*(i4/4)+1].qs + 2*(i4%4);
    const int32_t  * q8_3 = (const int *)bq8_1[4*(i4/4)+2].qs + 2*(i4%4);
    const int32_t  * q8_4 = (const int *)bq8_1[4*(i4/4)+3].qs + 2*(i4%4);

    const block_iq2_k * bq2 = (const block_iq2_k *) vbq + kbx;
    const uint32_t * q2 = (const uint32_t *)bq2->qs + 8*(i4/4) + 2*(i4%4);
    const uint16_t extra = bq2->extra >> (8*(i4/4) + (i4%4)/2);

    const uint32_t * scales = (const uint32_t *)bq2->scales;
    uint32_t s32 = __vsub4((scales[i4/4] >> 4*(((i4%4)/2)%2)) & 0x0f0f0f0f, 0x08080808);
    const int8_t * s8 = (const int8_t *)&s32;

    // Block of 16: (32*(4*(i4/4)+k)+8*(i4%4))/16 = 8*(i4/4) + 2*k + (i4%4)/2
    // -> scales_l[4*(i4/4) + k] >> 4*(((i4%4)/2)%2)

#ifdef __CUDA_ARCH__
    uint32_t extra32 = uint32_t(extra & 0xff) * 0x01010101;
    uint32_t extra32_1 = (extra32 << 2) & 0x44444444;
    uint32_t extra32_2 = (extra32 << 0) & 0x44444444;

    uint32_t val1, val2;

    val1 = ((q2[0] >> 0) & 0x33333333) | extra32_1; val2 = ((q2[1] >> 0) & 0x33333333) | extra32_1;
    int2 v1 = get_int_from_table_8(val1, iq2nl_values);
    int2 v2 = get_int_from_table_8(val2, iq2nl_values);
    int sumi1 = ggml_cuda_dp4a(v2.x, q8_1[1], ggml_cuda_dp4a(v1.x, q8_1[0], 0)) * s8[0];
    int sumi3 = ggml_cuda_dp4a(v2.y, q8_3[1], ggml_cuda_dp4a(v1.y, q8_3[0], 0)) * s8[2];

    val1 = ((q2[0] >> 2) & 0x33333333) | extra32_2; val2 = ((q2[1] >> 2) & 0x33333333) | extra32_2;
    v1 = get_int_from_table_8(val1, iq2nl_values);
    v2 = get_int_from_table_8(val2, iq2nl_values);
    int sumi2 = ggml_cuda_dp4a(v2.x, q8_2[1], ggml_cuda_dp4a(v1.x, q8_2[0], 0)) * s8[1];
    int sumi4 = ggml_cuda_dp4a(v2.y, q8_4[1], ggml_cuda_dp4a(v1.y, q8_4[0], 0)) * s8[3];

#else

    const int * all_values = (const int *)iq2k_table;
    const int * values;

    uint32_t val1 = q2[0], val2 = q2[1];

    uint32_t aux32[2];
    int v1, v2;

    aux32[0] = ((val1 >> 0) & 0x03030303); aux32[1] = ((val2 >> 0) & 0x03030303); values = all_values + ((extra & 0x01) << 8);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi1 = ggml_cuda_dp4a(v2, q8_1[1], ggml_cuda_dp4a(v1, q8_1[0], 0)) * s8[0];

    aux32[0] = ((val1 >> 2) & 0x03030303); aux32[1] = ((val2 >> 2) & 0x03030303); values = all_values + ((extra & 0x04) << 6);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi2 = ggml_cuda_dp4a(v2, q8_2[1], ggml_cuda_dp4a(v1, q8_2[0], 0)) * s8[1];

    aux32[0] = ((val1 >> 4) & 0x03030303); aux32[1] = ((val2 >> 4) & 0x03030303); values = all_values + ((extra & 0x10) << 4);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi3 = ggml_cuda_dp4a(v2, q8_3[1], ggml_cuda_dp4a(v1, q8_3[0], 0)) * s8[2];

    aux32[0] = ((val1 >> 6) & 0x03030303); aux32[1] = ((val2 >> 6) & 0x03030303); values = all_values + ((extra & 0x40) << 2);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi4 = ggml_cuda_dp4a(v2, q8_4[1], ggml_cuda_dp4a(v1, q8_4[0], 0)) * s8[3];
#endif

    *result += __half2float(bq2->d) * (__low2float(bq8_1[4*(i4/4)+0].ds) * sumi1
                                    +  __low2float(bq8_1[4*(i4/4)+1].ds) * sumi2
                                    +  __low2float(bq8_1[4*(i4/4)+2].ds) * sumi3
                                    +  __low2float(bq8_1[4*(i4/4)+3].ds) * sumi4);
}

#define VDR_IQ2_KS_Q8_1_MMVQ 4
#define VDR_IQ2_KS_Q8_1_MMQ  4

__device__ __forceinline__ void vec_dot_iq2_ks_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    float scale = *(const half *)vbq;
    const block_iq2_ks * bq2 = (const block_iq2_ks *)((const char *)vbq + sizeof(half)) + kbx;

    int i4 = iqs/4;  // 0...7. We will process q8 blocks 4*(i4/4), 4*(i4/4)+1, 4*(i4/4)+2, 4*(i4/4)+3
    const int32_t  * q8_1 = (const int *)bq8_1[4*(i4/4)+0].qs + 2*(i4%4);
    const int32_t  * q8_2 = (const int *)bq8_1[4*(i4/4)+1].qs + 2*(i4%4);
    const int32_t  * q8_3 = (const int *)bq8_1[4*(i4/4)+2].qs + 2*(i4%4);
    const int32_t  * q8_4 = (const int *)bq8_1[4*(i4/4)+3].qs + 2*(i4%4);

    const uint16_t * q2 = (const uint16_t *)bq2->qs + 16*(i4/4) + 4*(i4%4);
    const uint16_t extra = bq2->extra >> 4*(i4/4);

    uint32_t val1 = q2[0] | (q2[1] << 16), val2 = q2[2] | (q2[3] << 16);

    int32_t scales32;
    const uint16_t * scales16 = (const uint16_t *)bq2->scales;
    scales32 = __vsub4((scales16[i4/4] | (scales16[i4/4] << 12)) & 0x0f0f0f0f, 0x10101010);
    int8_t * s8 = (int8_t *)&scales32;
    s8[0] += ((extra >> 4) & 0x10);
    s8[1] += ((extra >> 6) & 0x10);
    s8[2] += ((extra >> 5) & 0x10);
    s8[3] += ((extra >> 7) & 0x10);

#ifdef __CUDA_ARCH__

    uint32_t extra32 = uint32_t(extra & 0xf) * 0x01010101;

    uint32_t this_extra = ((extra32 << 2) & 0x04040404) | ((extra32 << 4) & 0x40404040);
    uint32_t idx1 = ((val1 >> 0) & 0x33333333) | this_extra;
    uint32_t idx2 = ((val2 >> 0) & 0x33333333) | this_extra;
    int2 v1 = get_int_from_table_8(idx1, iq2nl_values);
    int2 v2 = get_int_from_table_8(idx2, iq2nl_values);

    int sumi1 = ggml_cuda_dp4a(v2.x, q8_1[1], ggml_cuda_dp4a(v1.x, q8_1[0], 0)) * s8[0];
    int sumi3 = ggml_cuda_dp4a(v2.y, q8_3[1], ggml_cuda_dp4a(v1.y, q8_3[0], 0)) * s8[1];

    this_extra = ((extra32 << 1) & 0x04040404) | ((extra32 << 3) & 0x40404040);
    idx1 = ((val1 >> 2) & 0x33333333) | this_extra;
    idx2 = ((val2 >> 2) & 0x33333333) | this_extra;
    v1 = get_int_from_table_8(idx1, iq2nl_values);
    v2 = get_int_from_table_8(idx2, iq2nl_values);

    int sumi2 = ggml_cuda_dp4a(v2.x, q8_2[1], ggml_cuda_dp4a(v1.x, q8_2[0], 0)) * s8[2];
    int sumi4 = ggml_cuda_dp4a(v2.y, q8_4[1], ggml_cuda_dp4a(v1.y, q8_4[0], 0)) * s8[3];

#else

    uint32_t aux32[2];
    int v1, v2;
    const int * all_values = (const int *)iq2k_table;
    const int * values;

    aux32[0] = ((val1 >> 0) & 0x03030303); aux32[1] = ((val2 >> 0) & 0x03030303); values = all_values + ((extra & 0x01) << 8);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi1 = ggml_cuda_dp4a(v2, q8_1[1], ggml_cuda_dp4a(v1, q8_1[0], 0)) * s8[0];

    aux32[0] = ((val1 >> 2) & 0x03030303); aux32[1] = ((val2 >> 2) & 0x03030303); values = all_values + ((extra & 0x02) << 7);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi2 = ggml_cuda_dp4a(v2, q8_2[1], ggml_cuda_dp4a(v1, q8_2[0], 0)) * s8[2];

    aux32[0] = ((val1 >> 4) & 0x03030303); aux32[1] = ((val2 >> 4) & 0x03030303); values = all_values + ((extra & 0x04) << 6);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi3 = ggml_cuda_dp4a(v2, q8_3[1], ggml_cuda_dp4a(v1, q8_3[0], 0)) * s8[1];

    aux32[0] = ((val1 >> 6) & 0x03030303); aux32[1] = ((val2 >> 6) & 0x03030303); values = all_values + ((extra & 0x08) << 5);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi4 = ggml_cuda_dp4a(v2, q8_4[1], ggml_cuda_dp4a(v1, q8_4[0], 0)) * s8[3];
#endif

    *result += scale * (__low2float(bq8_1[4*(i4/4)+0].ds) * sumi1
                     +  __low2float(bq8_1[4*(i4/4)+1].ds) * sumi2
                     +  __low2float(bq8_1[4*(i4/4)+2].ds) * sumi3
                     +  __low2float(bq8_1[4*(i4/4)+3].ds) * sumi4);
}

__device__ __forceinline__ void vec_dot_iq2_k_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq2_k_r4 * bq2 = (const block_iq2_k_r4 *)vbq + kbx;

    // iqs is 0...30 in steps of 2
    const int ib16 = iqs/2;
    const float d8 = __low2float(bq8_1[ib16/2].ds);
    const int32_t  * q8 = (const int *)bq8_1[ib16/2].qs + 4*(ib16%2);

    int ib32 = ib16/2;
    int is   = ib16%2;
    const int * scales_l = (const int *)bq2->scales;

    int scales = __vsub4(((scales_l[2*(ib32%4)+is] >> 4*(ib32/4)) & 0x0f0f0f0f), 0x08080808);
    const int8_t * s8 = (const int8_t *)&scales;

    const int * q2 = (const int *)bq2->qs + 8*ib32 + 4*is;

#ifdef __CUDA_ARCH__

#pragma unroll
    for (int i = 0; i < 4; ++i) {
        uint32_t extra32 = uint32_t((bq2->extra[i+4*is] >> ib32) & 1) * 0x04040404;
        extra32 |= (extra32 << 4);
        uint32_t val1 = ((q2[i] >> 0) & 0x33333333) | extra32;
        uint32_t val2 = ((q2[i] >> 2) & 0x33333333) | extra32;
        int2 v1 = get_int_from_table_8(val1, iq2nl_values);
        int2 v2 = get_int_from_table_8(val2, iq2nl_values);
        int sumi = 0;
        sumi = ggml_cuda_dp4a(v1.x, q8[0], ggml_cuda_dp4a(v2.x, q8[1], sumi));
        sumi = ggml_cuda_dp4a(v1.y, q8[2], ggml_cuda_dp4a(v2.y, q8[3], sumi));
        const float d = __half2float(bq2->d[i]) * d8;
        result[i] += d * sumi * s8[i];
    }

#else
    const int * all_values = (const int *)iq2k_table;
    int2 val1;
    int aux32[2];
#pragma unroll
    for (int i = 0; i < 4; ++i) {
        auto values1 = all_values + (((bq2->extra[i+4*is] >> ib32) & 1) << 8);
        int sumi1 = 0;
        aux32[0] = ((q2[i] >> 0) & 0x03030303);
        aux32[1] = ((q2[i] >> 2) & 0x03030303);
        val1.x  = int_from_table_4(aux32[0], values1);
        val1.y  = int_from_table_4(aux32[1], values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[0], ggml_cuda_dp4a(val1.y, q8[1], sumi1));
        aux32[0] = ((q2[i] >> 4) & 0x03030303);
        aux32[1] = ((q2[i] >> 6) & 0x03030303);
        val1.x  = int_from_table_4(aux32[0], values1);
        val1.y  = int_from_table_4(aux32[1], values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[2], ggml_cuda_dp4a(val1.y, q8[3], sumi1));
        const float d = __half2float(bq2->d[i]) * d8;
        result[i] += d * sumi1 * s8[i];
    }
#endif
}

#define VDR_IQ3_K_Q8_1_MMVQ 4
#define VDR_IQ3_K_Q8_1_MMQ  4

__device__ __forceinline__ void vec_dot_iq3_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iiqs, float * result) {
    const block_iq3_k * bq3 = (const block_iq3_k *) vbq + kbx;

    int iqs = iiqs/4;
    const int ib128 = iqs/4;  // 0 or 1. 0 works on quants 0...127, 1 on quants 128...255
                              // Each thread processes 8 quants in each of the 4 32-blocks
    const int il8   = iqs%4;  // 0...3. 0 works on quants 0...7, 1 on quants 8...15, 2 on 16...23, 3 on 24...31
    const int shift = 4*(il8/2);

    const uint16_t * ql = (const uint16_t *)bq3->qs + 16*ib128 + 4*il8;
    const uint16_t * qh = (const uint16_t *)bq3->qh + 4*il8;

    uint32_t aux32;
    const uint8_t * aux8 = (const uint8_t *)&aux32;

    const int hshift = 4*(1-ib128);
    const uint16_t sh = bq3->scales_h >> (8*ib128 + il8/2);

    const uint8_t extra = bq3->extra >> (8*ib128 + il8/2);
    uint32_t extra32 = uint32_t(extra) * 0x01010101;
    uint32_t extra32_1 = ((extra32 << 3) & 0x08080808) | ((extra32 << 5) & 0x80808080);
    uint32_t extra32_2 = ((extra32 << 2) & 0x08080808) | ((extra32 << 4) & 0x80808080);

    const int * q8;
    int sumi[4] = {0, 0, 0, 0};
    for (int i = 0; i < 2; ++i) {
        uint32_t vl = ql[2*i+0] | (ql[2*i+1] << 16);
        uint32_t vh = ((qh[2*i+0] | (qh[2*i+1] << 16)) << hshift);

        uint32_t val1 = ((vl >> 0) & 0x33333333) | extra32_1 | ((vh >> 2) & 0x04040404) | ((vh >> 0) & 0x40404040);
        uint32_t val2 = ((vl >> 2) & 0x33333333) | extra32_2 | ((vh >> 3) & 0x04040404) | ((vh >> 1) & 0x40404040);
        int2 v1 = get_int_from_table_16(val1, iq3nl_values);
        int2 v2 = get_int_from_table_16(val2, iq3nl_values);

        q8 = (const int *)bq8_1[4*ib128+0].qs + 2*il8;
        sumi[0] = ggml_cuda_dp4a(v1.x, q8[i], sumi[0]);

        q8 += sizeof(block_q8_1)/4;
        sumi[1] = ggml_cuda_dp4a(v2.x, q8[i], sumi[1]);

        q8 += sizeof(block_q8_1)/4;
        sumi[2] = ggml_cuda_dp4a(v1.y, q8[i], sumi[2]);

        q8 += sizeof(block_q8_1)/4;
        sumi[3] = ggml_cuda_dp4a(v2.y, q8[i], sumi[3]);
    }
    const float d = __half2float(bq3->d);
    const uint16_t * sl16 = (const uint16_t *)bq3->scales_l + 2*ib128;
    aux32 = ((((sl16[0] | (sl16[1] << 16)) >> shift) & 0x0f0f0f0f) << 1) | 0x01010101;
    *result += d * (__low2float(bq8_1[4*ib128+0].ds) * aux8[0] * (sh & 0x01 ? -1 : 1) * sumi[0] +
                    __low2float(bq8_1[4*ib128+1].ds) * aux8[1] * (sh & 0x04 ? -1 : 1) * sumi[1] +
                    __low2float(bq8_1[4*ib128+2].ds) * aux8[2] * (sh & 0x10 ? -1 : 1) * sumi[2] +
                    __low2float(bq8_1[4*ib128+3].ds) * aux8[3] * (sh & 0x40 ? -1 : 1) * sumi[3]);

}

// TODO
__device__ __forceinline__ void vec_dot_iq2_kl_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iiqs, float * result) {

    float d = __half2float(*(const half *)vbq);
    const block_iq2_kl * bq2 = (const block_iq2_kl *)((const char *)vbq + sizeof(half)) + kbx;

    int iqs = iiqs/4;
    const int ib64 = iqs/2;  // 0...3. 0 works on quants 0...63, 1 on quants 64...127, etc.
                             // Each thread processes 16 quants in each of the 2 32-blocks
    const int il16 = iqs%2;  // 0...3. 0 works on quants 0...7, 1 on quants 8...15, 2 on 16...23, 3 on 24...31

    const uint16_t * ql = (const uint16_t *)bq2->qs + 8*ib64 + 4*il16;
    const uint16_t * qh = (const uint16_t *)bq2->qh + 4*il16;

    int32_t aux32;
    const uint8_t * aux8 = (const uint8_t *)&aux32;

    const int * q8l = (const int *)bq8_1[2*ib64+0].qs + 4*il16;
    const int * q8h = (const int *)bq8_1[2*ib64+1].qs + 4*il16;

    int sumi1 = 0, sumi2 = 0;
    int v1, v2;
    for (int i = 0; i < 2; ++i) {
        uint32_t vl =  ql[2*i+0] | (ql[2*i+1] << 16);
        uint32_t vh = (qh[2*i+0] | (qh[2*i+1] << 16)) >> 2*ib64;

        aux32 = (vl & 0x0f0f0f0f) | ((vh << 4) & 0x10101010);
        v1 = iq2kl_values[aux8[0]] | (iq2kl_values[aux8[1]] << 16);
        v2 = iq2kl_values[aux8[2]] | (iq2kl_values[aux8[3]] << 16);
        sumi1 = ggml_cuda_dp4a(v1, q8l[2*i+0], ggml_cuda_dp4a(v2, q8l[2*i+1], sumi1));

        aux32 = ((vl >> 4) & 0x0f0f0f0f) | ((vh << 3) & 0x10101010);
        v1 = iq2kl_values[aux8[0]] | (iq2kl_values[aux8[1]] << 16);
        v2 = iq2kl_values[aux8[2]] | (iq2kl_values[aux8[3]] << 16);
        sumi2 = ggml_cuda_dp4a(v1, q8h[2*i+0], ggml_cuda_dp4a(v2, q8h[2*i+1], sumi2));
    }

    auto sh = bq2->scales_h >> 4*ib64;
    int ls1 = int(((bq2->scales_l[(2*ib64+0)%4] >> 4*(ib64/2)) & 0xf) | ((sh << 4) & 0x30)) - 32;
    int ls2 = int(((bq2->scales_l[(2*ib64+1)%4] >> 4*(ib64/2)) & 0xf) | ((sh << 2) & 0x30)) - 32;

    *result += d * (__low2float(bq8_1[2*ib64+0].ds) * ls1 * sumi1 + __low2float(bq8_1[2*ib64+1].ds) * ls2 * sumi2);

}

__device__ __forceinline__ void vec_dot_iq3_ks_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iiqs, float * result) {

    float d = __half2float(*(const half *)vbq);
    const block_iq3_ks * bq3 = (const block_iq3_ks *)((const char *)vbq + sizeof(half)) + kbx;

    int iqs = iiqs/4;
    const int ib128 = iqs/4;  // 0 or 1. 0 works on quants 0...127, 1 on quants 128...255
                              // Each thread processes 8 quants in each of the 4 32-blocks
    const int il8   = iqs%4;  // 0...3. 0 works on quants 0...7, 1 on quants 8...15, 2 on 16...23, 3 on 24...31

    const uint16_t * ql = (const uint16_t *)bq3->qs + 16*ib128 + 4*il8;
    const uint16_t * qh = (const uint16_t *)bq3->qh + 4*il8;

    uint16_t extra = bq3->extra >> 4*ib128;
    uint32_t extra_v = uint32_t(extra >> 8) * 0x01010101;

    uint32_t extra32_1 = ((extra_v << 3) & 0x08080808) | ((extra_v << 5) & 0x80808080);
    uint32_t extra32_2 = ((extra_v << 2) & 0x08080808) | ((extra_v << 4) & 0x80808080);

    const int * q8;
    int sumi[4] = {0, 0, 0, 0};
    for (int i = 0; i < 2; ++i) {
        uint32_t vl = ql[2*i+0] | (ql[2*i+1] << 16);
        uint32_t vh = ((qh[2*i+0] | (qh[2*i+1] << 16)) >> 4*ib128);

        uint32_t val1 = ((vl >> 0) & 0x33333333) | extra32_1 | ((vh << 2) & 0x04040404) | ((vh << 4) & 0x40404040);
        uint32_t val2 = ((vl >> 2) & 0x33333333) | extra32_2 | ((vh << 1) & 0x04040404) | ((vh << 3) & 0x40404040);
        int2 v1 = get_int_from_table_16(val1, iq3nl_values);
        int2 v2 = get_int_from_table_16(val2, iq3nl_values);

        q8 = (const int *)bq8_1[4*ib128+0].qs + 2*il8;
        sumi[0] = ggml_cuda_dp4a(v1.x, q8[i], sumi[0]);

        q8 += sizeof(block_q8_1)/4;
        sumi[1] = ggml_cuda_dp4a(v2.x, q8[i], sumi[1]);

        q8 += sizeof(block_q8_1)/4;
        sumi[2] = ggml_cuda_dp4a(v1.y, q8[i], sumi[2]);

        q8 += sizeof(block_q8_1)/4;
        sumi[3] = ggml_cuda_dp4a(v2.y, q8[i], sumi[3]);
    }
    const uint16_t * sl16 = (const uint16_t *)bq3->scales;
    int32_t aux32 = __vsub4(((sl16[0] | (sl16[1] << 16)) >> 4*ib128) & 0x0f0f0f0f, 0x10101010);
    const int8_t * a8 = (const int8_t *)&aux32;
    *result += d * (__low2float(bq8_1[4*ib128+0].ds) * (a8[0] + ((extra << 4) & 0x10)) * sumi[0] +
                    __low2float(bq8_1[4*ib128+1].ds) * (a8[1] + ((extra << 3) & 0x10)) * sumi[1] +
                    __low2float(bq8_1[4*ib128+2].ds) * (a8[2] + ((extra << 2) & 0x10)) * sumi[2] +
                    __low2float(bq8_1[4*ib128+3].ds) * (a8[3] + ((extra << 1) & 0x10)) * sumi[3]);

}

__device__ __forceinline__ void vec_dot_iq1_bn_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    half d16; memcpy(&d16, vbq, sizeof(d16));
    float scale = d16;
    const block_iq1_bn * bq1 = (const block_iq1_bn *)((const char *)vbq + sizeof(d16)) + kbx;

    // iqs is 0 or 1

    int sumi = 0;
#if __CUDA_ARCH__ >= MIN_CC_DP4A // lowest compute capability for integer intrinsics
    uint16_t mult[2];
    mult[1] = iqs == 0 ? 27 : 3;
    mult[0] = mult[1] + (mult[1] << 1);
    const int * q8 = (const int *)bq8_1[iqs].qs;
    int val[4];
    for (int l = 0; l < 2; ++l) {
        int8_t * a = (int8_t *)val;
        const int i16 = 2*iqs + l;
        for (int k = 0; k < 3; ++k) {
            uint16_t q = bq1->ql[3*i16+k];
            for (int j = 4; j >= 0; --j) {
                uint16_t v = q & 0xff;
                v += v << 1;
                a[j] = v >> 8;
                q += q << 1;
            }
            a += 5;
        }
        uint16_t v = (mult[l]*bq1->extra) & 0xff;
        v += v << 1;
        *a = v >> 8;
        sumi = __dp4a(val[0], q8[4*l+0], __dp4a(val[1], q8[4*l+1], __dp4a(val[2], q8[4*l+2], __dp4a(val[3], q8[4*l+3], sumi))));
    }
    float2 d8 = __half22float2(bq8_1[iqs].ds);
    *result += scale * (d8.x * sumi - d8.y);
#else
    static const uint16_t k_mult[5] = {81, 27, 9, 3, 1};
    const int8_t * q8 = bq8_1[iqs].qs;
    for (int l = 0; l < 2; ++l) {
        const int i16 = 2*iqs + l;
        for (int k = 0; k < 3; ++k) {
            uint8_t q = bq1->ql[3*i16+k];
            for (int j = 0; j < 5; ++j) {
                uint8_t v = k_mult[j]*q;
                int8_t vs = (v + (v >> 1)) >> 7;
                sumi += q8[j]*(vs - 1);
            }
            q8 += 5;
        }
        uint8_t v = k_mult[i16]*bq1->extra;
        int8_t vs = (v + (v >> 1)) >> 7;
        sumi += q8[0]*(vs - 1);
        q8++;
    }
    *result += scale * __low2float(bq8_1[iqs].ds) * sumi;
#endif
}

__device__ __forceinline__ void vec_dot_iq2_bn_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    float scale = *(const float *)vbq;
    const block_iq2_bn * bq2 = (const block_iq2_bn *)((const char *)vbq + sizeof(float)) + kbx;

    // iqs is 0 or 1

#if __CUDA_ARCH__ >= MIN_CC_DP4A // lowest compute capability for integer intrinsics
    auto qs  = (const int *)bq2->qs + 2*iqs;
    auto q8l = (const int *)bq8_1[0].qs + 2*iqs;
    auto q8h = (const int *)bq8_1[1].qs + 2*iqs;
    int sumi1 = 0, sumi2 = 0, sumi3 = 0, sumi4 = 0;
    for (int j = 0; j < 2; ++j) {
        int vl = qs[j];
        int vh = qs[j] >> 4;
        sumi1 = __dp4a(vl & 0x03030303, q8l[j+0], sumi1);
        sumi2 = __dp4a(vl & 0x0c0c0c0c, q8l[j+4], sumi2);
        sumi3 = __dp4a(vh & 0x03030303, q8h[j+0], sumi3);
        sumi4 = __dp4a(vh & 0x0c0c0c0c, q8h[j+4], sumi4);
    }
    auto d8l = __half22float2(bq8_1[0].ds);
    auto d8h = __half22float2(bq8_1[1].ds);
    *result += scale * (d8l.x * (sumi1 + 0.25f*sumi2) + d8h.x * (sumi3 + 0.25f * sumi4) - 0.5f*d8l.y - 0.5f*d8h.y);
#else
    int sumi1 = 0, sumi2 = 0, sumi3 = 0, sumi4 = 0;
    auto q8l = bq8_1[0].qs + 8*iqs;
    auto q8h = bq8_1[1].qs + 8*iqs;
    auto qs  = bq2->qs + 8*iqs;
    for (int j = 0; j < 8; ++j) {
        sumi1 += q8l[j+ 0] * (qs[j] & 0x03);
        sumi2 += q8l[j+16] * (qs[j] & 0x0c);
        sumi3 += q8h[j+ 0] * (qs[j] & 0x30);
        sumi4 += q8h[j+16] * (qs[j] & 0xc0);
    }
    auto d8l = __half22float2(bq8_1[0].ds);
    auto d8h = __half22float2(bq8_1[1].ds);
    *result += scale * (d8l.x * (sumi1 + 0.25f*sumi2) + 0.0625f * d8h.x*(sumi3 + 0.25f*sumi4) - 0.5f*d8l.y - 0.5f*d8h.y);
#endif
}

} // namespace

static void mul_mat_vec_iq2_k_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_K, VDR_IQ2_K_Q8_1_MMVQ, vec_dot_iq2_k_q8_1>(args, stream);
}

static void mul_mat_vec_iq3_k_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ3_K, VDR_IQ3_K_Q8_1_MMVQ, vec_dot_iq3_k_q8_1>(args, stream);
}

static void mul_mat_vec_iq4_k_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_K, VDR_IQ4_K_Q8_1_MMVQ, vec_dot_iq4_k_q8_1>(args, stream);
}

static void mul_mat_vec_iq4_k_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_K_R4, 2, vec_dot_iq4_k_r4_q8_1, 4>(args, stream);
}

static void mul_mat_vec_iq4_ks_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_KS_R4, 2, vec_dot_iq4_ks_r4_q8_1, 4>(args, stream);
}

static void mul_mat_vec_iq1_s_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ1_S_R4, 2, vec_dot_iq1_s_r4_q8_1, 4>(args, stream);
}

static void mul_mat_vec_iq1_m_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ1_M_R4, 2, vec_dot_iq1_m_r4_q8_1, 4>(args, stream);
}

static void mul_mat_vec_iq5_k_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ5_K_R4, 2, vec_dot_iq5_k_r4_q8_1, 4>(args, stream);
}

static void mul_mat_vec_iq5_ks_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ5_KS_R4, 2, vec_dot_iq5_ks_r4_q8_1, 4>(args, stream);
}

static void mul_mat_vec_iq2_k_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_K_R4, 2, vec_dot_iq2_k_r4_q8_1, 4>(args, stream);
}

static void mul_mat_vec_iq3_k_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ3_K_R4, 2, vec_dot_iq3_k_r4_q8_1, 4>(args, stream);
}

static void mul_mat_vec_iq4_ks_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_KS, VDR_IQ4_KS_Q8_1_MMVQ, vec_dot_iq4_ks_q8_1>(args, stream);
}

static void mul_mat_vec_iq2_kl_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_KL, VDR_IQ3_K_Q8_1_MMVQ, vec_dot_iq2_kl_q8_1>(args, stream);
}

static void mul_mat_vec_iq3_ks_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ3_KS, VDR_IQ3_K_Q8_1_MMVQ, vec_dot_iq3_ks_q8_1>(args, stream);
}

static void mul_mat_vec_iq4_kt_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_KT, VDR_IQ4_KS_Q8_1_MMVQ, vec_dot_iq4_kt_q8_1>(args, stream);
}

static void mul_mat_vec_iq1_kt_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ1_KT, VDR_IQ4_KS_Q8_1_MMVQ, vec_dot_iq1_kt_q8_1>(args, stream);
}

static void mul_mat_vec_iq2_kt_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_KT, VDR_IQ4_KS_Q8_1_MMVQ, vec_dot_iq2_kt_q8_1>(args, stream);
}

static void mul_mat_vec_iq3_kt_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ3_KT, VDR_IQ4_KS_Q8_1_MMVQ, vec_dot_iq3_kt_q8_1>(args, stream);
}

static void mul_mat_vec_iq4_kss_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_KSS, VDR_IQ4_KSS_Q8_1_MMVQ, vec_dot_iq4_kss_q8_1>(args, stream);
}

static void mul_mat_vec_iq2_ks_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_KS, VDR_IQ2_KS_Q8_1_MMVQ, vec_dot_iq2_ks_q8_1>(args, stream);
}

static void mul_mat_vec_iq5_k_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ5_K, VDR_IQ5_K_Q8_1_MMVQ, vec_dot_iq5_k_q8_1>(args, stream);
}

static void mul_mat_vec_iq5_ks_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ5_KS, VDR_IQ5_K_Q8_1_MMVQ, vec_dot_iq5_ks_q8_1>(args, stream);
}

static void mul_mat_vec_iq6_k_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ6_K, VDR_IQ6_K_Q8_1_MMVQ, vec_dot_iq6_k_q8_1>(args, stream);
}

static void mul_mat_vec_iq1_bn_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ1_BN, 1, vec_dot_iq1_bn_q8_1>(args, stream);
}

static void mul_mat_vec_iq2_bn_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_BN, 1, vec_dot_iq2_bn_q8_1>(args, stream);
}

void iqk_mul_mat_vec_q(ggml_type type, const mmvq_args & args, hipStream_t stream) {
    switch (type) {
        case GGML_TYPE_IQ1_BN:
            mul_mat_vec_iq1_bn_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_BN:
            mul_mat_vec_iq2_bn_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_K:
            mul_mat_vec_iq2_k_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ3_K:
            mul_mat_vec_iq3_k_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_KL:
            mul_mat_vec_iq2_kl_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ3_KS:
            mul_mat_vec_iq3_ks_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ4_K:
            mul_mat_vec_iq4_k_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ4_KS:
            mul_mat_vec_iq4_ks_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ4_KSS:
            mul_mat_vec_iq4_kss_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ1_KT:
            mul_mat_vec_iq1_kt_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_KT:
            mul_mat_vec_iq2_kt_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ3_KT:
            mul_mat_vec_iq3_kt_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ4_KT:
            mul_mat_vec_iq4_kt_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_KS:
            mul_mat_vec_iq2_ks_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ5_K:
            mul_mat_vec_iq5_k_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ5_KS:
            mul_mat_vec_iq5_ks_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ6_K:
            mul_mat_vec_iq6_k_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_K_R4:
            mul_mat_vec_iq2_k_r4_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ3_K_R4:
            mul_mat_vec_iq3_k_r4_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ4_K_R4:
            mul_mat_vec_iq4_k_r4_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ4_KS_R4:
            mul_mat_vec_iq4_ks_r4_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ5_K_R4:
            mul_mat_vec_iq5_k_r4_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ5_KS_R4:
            mul_mat_vec_iq5_ks_r4_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ1_S_R4:
            mul_mat_vec_iq1_s_r4_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ1_M_R4:
            mul_mat_vec_iq1_m_r4_q8_1_cuda(args, stream);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
}
