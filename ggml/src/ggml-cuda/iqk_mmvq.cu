#include "hip/hip_runtime.h"
//
// Copyright (C) 2024 Iwan Kawrakow
// MIT license
// SPDX-License-Identifier: MIT
//

#include "iqk_mmvq.cuh"

typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs);

//  Reminder:
//    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
//    constexpr int qi  = ggml_cuda_type_traits<type>::qi;
//    constexpr int vdr = get_vdr_mmvq(type);

namespace {
template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda, int ncols_y>
__device__ void iqk_mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst, const int64_t row_size) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int nwarps              = 1;
    constexpr int rows_per_cuda_block = 1;
#else
    constexpr int nwarps              = ncols_y <= 4 ? 4 : 2;
    constexpr int rows_per_cuda_block = ncols_y == 1 ? 1 : 2;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp[j][i] += vec_dot_q_cuda((const void *)((const char *)vx + (row0 + i)*row_size),
                    &y[j*blocks_per_col_y + kby], kbx, kqs);
            }
        }
    }

    __shared__ float tmp_shared[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared[threadIdx.y-1][j][i][threadIdx.x] = tmp[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp[j][i] += tmp_shared[l][j][i][threadIdx.x];
            }
            tmp[j][i] = warp_reduce_sum(tmp[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + threadIdx.x < nrows_dst)) {
            dst[j*nrows_dst + row0 + threadIdx.x] = tmp[j][threadIdx.x];
        }
    }
}

template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda, int ncols_y>
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
// tell the compiler to use as many registers as it wants, see nwarps definition below
__launch_bounds__((ncols_y <= 4 ? 4 : 2)*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__global__ void iqk_mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst, const char * __restrict__ ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst, const int64_t row_size,
    const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, const int64_t ids_nb0) {
    int i2 = blockIdx.y;
    int i02 = ids_data ? *(const int *)(ids_data + i2*ids_nb0) : i2;
    if (i02 < 0) return;
    const char * cx = (const char *)vx + i02*nb02;
    const char * cy = (const char *)vy + i2*nb12;
    char * cdst = (char *)dst + i2*nb2;
    iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, ncols_y>(cx, cy, (float *)cdst, ncols_x, nrows_x, nrows_y, nrows_dst, row_size);
}

template <ggml_type type, int vdr, vec_dot_q_cuda_t vec_dot_q_cuda>
void iqk_mul_mat_vec_q_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    GGML_ASSERT(ncols_x % ggml_blck_size(type) == 0);
    //GGML_ASSERT(ncols_y <= MMVQ_MAX_BATCH_SIZE);

    int id = ggml_cuda_get_device();

    int64_t nwarps = 1;
    int64_t rows_per_cuda_block = 1;

    if (ggml_cuda_info().devices[id].cc < CC_RDNA2) { // NVIDIA and AMD older than RDNA2
        switch(ncols_y) {
            case 1:
                nwarps = 4;
                rows_per_cuda_block = 1;
                break;
            case 2:
            case 3:
            case 4:
                nwarps = 4;
                rows_per_cuda_block = 2;
                break;
            case 5:
            case 6:
            case 7:
            case 8:
                nwarps = 2;
                rows_per_cuda_block = 2;
                break;
            default:
                GGML_ASSERT(false);
                break;
        }
    }
    const int64_t nblocks = (nrows_x + rows_per_cuda_block - 1) / rows_per_cuda_block;
    const dim3 block_nums(nblocks, ne2, 1);
    const dim3 block_dims(WARP_SIZE, nwarps, 1);

    const int64_t row_size = ggml_row_size(type, ncols_x);

    switch (ncols_y) {
        case 1:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 1><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, nrows_dst, row_size, nb02, nb12, nb2, ids_nb0);
            break;
        case 2:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 2><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, nrows_dst, row_size, nb02, nb12, nb2, ids_nb0);
            break;
        case 3:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 3><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, nrows_dst, row_size, nb02, nb12, nb2, ids_nb0);
            break;
        case 4:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 4><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, nrows_dst, row_size, nb02, nb12, nb2, ids_nb0);
            break;
        case 5:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 5><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, nrows_dst, row_size, nb02, nb12, nb2, ids_nb0);
            break;
        case 6:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 6><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, nrows_dst, row_size, nb02, nb12, nb2, ids_nb0);
            break;
        case 7:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 7><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, nrows_dst, row_size, nb02, nb12, nb2, ids_nb0);
            break;
        case 8:
            iqk_mul_mat_vec_q<type, vdr, vec_dot_q_cuda, 8><<<block_nums, block_dims, 0, stream>>>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, nrows_dst, row_size, nb02, nb12, nb2, ids_nb0);
            break;
        default:
            GGML_ASSERT(false);
            break;
    }
}

__device__ __forceinline__ void get_int_from_table_16_shift(const uint32_t & q4, uint16_t shift, const uint8_t * all_values,
        int & val1, int & val2) {

    uint32_t aux32; const uint8_t * q8 = (const uint8_t *)&aux32;
    aux32 = q4 & 0x0f0f0f0f;
    const uint8_t * values = all_values + 16*(shift & 1);
    uint16_t v1 = values[q8[0]] | (values[q8[1]] << 8);
    uint16_t v2 = values[q8[2]] | (values[q8[3]] << 8);
    val1 = v1 | (v2 << 16);
    aux32 = (q4 >> 4) & 0x0f0f0f0f;
    values = all_values + 8*(shift & 2);
    v1 = values[q8[0]] | (values[q8[1]] << 8);
    v2 = values[q8[2]] | (values[q8[3]] << 8);
    val2 = v1 | (v2 << 16);
}

#define VDR_IQ4_K_Q8_1_MMVQ 4
#define VDR_IQ4_K_Q8_1_MMQ  4

__device__ __forceinline__ float vec_dot_iq4_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {

    const block_iq4_k * bq4 = (const block_iq4_k *) vbq + kbx;
    const uint8_t * all_values = (const uint8_t *)iq4k_values;

    // iqs is 0...28
    const int ib32 = iqs/4;
    // Why iqs/4 ?
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const uint16_t * q4 = (const uint16_t *)bq4->qs + 8*ib32;
    const uint16_t extra = bq4->extra >> 2*ib32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        const uint32_t aux32 = q4[2*j+0] | (q4[2*j+1] << 16);
        get_int_from_table_16_shift(aux32, extra, all_values, v1, v2);
        sumi1 = ggml_cuda_dp4a(v1, q8[j+0], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8[j+4], sumi2);
    }
    const float d = __half2float(bq4->d) * __low2float(bq8_1[ib32].ds);
    const uint8_t sh = bq4->scales_h[ib32/2] >> 4*(ib32%2);
    const int ls1 = ((bq4->scales_l[ib32] & 0xf) | ((sh << 4) & 0x30)) - 32;
    const int ls2 = ((bq4->scales_l[ib32] >>  4) | ((sh << 2) & 0x30)) - 32;
    return d * (sumi1 * ls1 + sumi2 * ls2);
}

#define VDR_IQ4_KS_Q8_1_MMVQ 4
#define VDR_IQ4_KS_Q8_1_MMQ  4

__device__ __forceinline__ float vec_dot_iq4_ks_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {

    float scale = *(const float *)vbq;
    const block_iq4_ks * bq4 = (const block_iq4_ks *)((const char *)vbq + sizeof(float)) + kbx;
    const uint8_t * all_values = (const uint8_t *)iq4k_values;

    // iqs is 0...28
    const int ib32 = iqs/4; // Why iqs/4 ?
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const uint32_t * q4 = (const uint32_t *)bq4->qs + 4*ib32;
    const float dl = scale * ((bq4->scales[ib32] & 254) - 127);
    int v1, v2;
    int sumi = 0;
    for (int j = 0; j < 4; ++j) {
        get_int_from_table_16_shift(q4[j], bq4->scales[ib32] & 1, all_values, v1, v2);
        sumi = ggml_cuda_dp4a(v1, q8[j+0], sumi);
        sumi = ggml_cuda_dp4a(v2, q8[j+4], sumi);
    }
    return dl * __low2float(bq8_1[ib32].ds) * sumi;
}

#define VDR_IQ4_KSS_Q8_1_MMVQ 4
#define VDR_IQ4_KSS_Q8_1_MMQ  4

__device__ __forceinline__ float vec_dot_iq4_kss_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {

    float scale = *(const float *)vbq;
    const block_iq4_kss * bq4 = (const block_iq4_kss *)((const char *)vbq + sizeof(float)) + kbx;
    const uint8_t * all_values = (const uint8_t *)iq4k_values;

    // iqs is 0...28
    const int ib32 = iqs/4; // Why iqs/4 ?
    const int32_t  * q8 = (const int *)bq8_1[ib32].qs;
    const uint32_t * q4 = (const uint32_t *)bq4->qs + 4*ib32;
    uint32_t s32 = (q4[0] & 0x00010001) | ((q4[1] & 0x00010001) << 2) | ((q4[2] & 0x00010001) << 4) | ((q4[3] & 0x00010001) << 6);
    uint8_t ls = (s32 | (s32 >> 15)) & 0xff;
    const float dl = scale * ((ls & 254) - 127);
    int v1, v2;
    int sumi = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t aux32 = q4[j] & 0xfffefffe;
        aux32 ^= (aux32 >> 1);
        get_int_from_table_16_shift(aux32, ls & 1, all_values, v1, v2);
        sumi = ggml_cuda_dp4a(v1, q8[j+0], sumi);
        sumi = ggml_cuda_dp4a(v2, q8[j+4], sumi);
    }
    return dl * __low2float(bq8_1[ib32].ds) * sumi;
}

#define VDR_IQ5_K_Q8_1_MMVQ 4
#define VDR_IQ5_K_Q8_1_MMQ  4

__device__ __forceinline__ int int_from_table(const uint8_t * a8, const uint8_t * values) {
    uint16_t v1 = values[a8[0]] | (values[a8[1]] << 8);
    uint16_t v2 = values[a8[2]] | (values[a8[3]] << 8);
    return v1 | (v2 << 16);
}

__device__ __forceinline__ float vec_dot_iq5_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {


    const block_iq5_k * bq5 = (const block_iq5_k *) vbq + kbx;
    const uint8_t * all_values = (const uint8_t *)iq5nl_values;

    int i4 = iqs/4;  // 0...7.  Blocks of 16 index is 4*(i4/2) + (i4%2) + (0 and 2)

    const int32_t  * q8_1 = (const int *)bq8_1[2*(i4/2)+0].qs + 4*(i4%2);
    const int32_t  * q8_2 = (const int *)bq8_1[2*(i4/2)+1].qs + 4*(i4%2);
    const uint32_t * q4 = (const uint32_t *)bq5->qs + 8*(i4/2) + 4*(i4%2);
    const uint32_t * qh = (const uint32_t *)bq5->qh + 4*(i4%2);
    const uint16_t extra = bq5->extra >> (4*(i4/2) + (i4%2));
    const uint8_t * values1 = all_values + 32*(extra & 1);
    const uint8_t * values2 = all_values +  8*(extra & 4);
    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)aux32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t h = qh[j] >> 2*(i4/2);
        aux32[0] = ((q4[j] >> 0) & 0x0f0f0f0f) | ((h << 4) & 0x10101010);
        aux32[1] = ((q4[j] >> 4) & 0x0f0f0f0f) | ((h << 3) & 0x10101010);
        v1 = int_from_table(a8+0, values1);
        v2 = int_from_table(a8+4, values2);
        sumi1 = ggml_cuda_dp4a(v1, q8_1[j], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8_2[j], sumi2);
    }
    const float d5 = __half2float(bq5->d);
    const uint8_t sh = bq5->scales_h[i4/2] >> 2*(i4%2);
    const int ls1 = (((bq5->scales_l[2*(i4/2)+0] >> 4*(i4%2)) & 0xf) | ((sh << 4) & 0x30)) - 32;
    const int ls2 = (((bq5->scales_l[2*(i4/2)+1] >> 4*(i4%2)) & 0xf) | ((sh << 0) & 0x30)) - 32;
    return d5 * (__low2float(bq8_1[2*(i4/2)+0].ds) * sumi1 * ls1 + __low2float(bq8_1[2*(i4/2)+1].ds) * sumi2 * ls2);
}

#define VDR_IQ6_K_Q8_1_MMVQ 4
#define VDR_IQ6_K_Q8_1_MMQ  4

__device__ __forceinline__ float vec_dot_iq6_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {


    const block_iq6_k * bq6 = (const block_iq6_k *) vbq + kbx;
    const uint8_t * all_values = (const uint8_t *)iq6nl_values;

    int i4 = iqs/4;  // 0...7.  Blocks of 16 index is 4*(i4/2) + (i4%2) + (0 and 2)
                     //         Blocks of 32 index is 2*(i4/2) + 0 or 1

    const int32_t  * q8_1 = (const int *)bq8_1[2*(i4/2)+0].qs + 4*(i4%2);
    const int32_t  * q8_2 = (const int *)bq8_1[2*(i4/2)+1].qs + 4*(i4%2);
    const uint32_t * q4 = (const uint32_t *)bq6->qs + 8*(i4/2) + 4*(i4%2);
    const uint32_t * qh = (const uint32_t *)bq6->qh + 8*(i4/4) + 4*(i4%2);
    const uint16_t extra = bq6->extra >> (4*(i4/2) + (i4%2));
    const uint8_t * values1 = all_values + 64*(extra & 1);
    const uint8_t * values2 = all_values + 16*(extra & 4);
    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)aux32;
    int v1, v2;
    int sumi1 = 0, sumi2 = 0;
    for (int j = 0; j < 4; ++j) {
        uint32_t h = qh[j] >> 4*((i4/2)%2);
        aux32[0] = ((q4[j] >> 0) & 0x0f0f0f0f) | ((h << 4) & 0x30303030);
        aux32[1] = ((q4[j] >> 4) & 0x0f0f0f0f) | ((h << 2) & 0x30303030);
        v1 = int_from_table(a8+0, values1);
        v2 = int_from_table(a8+4, values2);
        sumi1 = ggml_cuda_dp4a(v1, q8_1[j], sumi1);
        sumi2 = ggml_cuda_dp4a(v2, q8_2[j], sumi2);
    }
    const float d6 = __half2float(bq6->d);
    return d6 * (__low2float(bq8_1[2*(i4/2)+0].ds) * sumi1 * bq6->scales[4*(i4/2)+(i4%2)] + __low2float(bq8_1[2*(i4/2)+1].ds) * sumi2 * bq6->scales[4*(i4/2)+(i4%2)+2]);
}

static const __device__ uint32_t iq2k_table[512] = {
    0xe1e1e1e1, 0xe1e1e1f3, 0xe1e1e101, 0xe1e1e111, 0xe1e1f3e1, 0xe1e1f3f3, 0xe1e1f301, 0xe1e1f311,
    0xe1e101e1, 0xe1e101f3, 0xe1e10101, 0xe1e10111, 0xe1e111e1, 0xe1e111f3, 0xe1e11101, 0xe1e11111,
    0xe1f3e1e1, 0xe1f3e1f3, 0xe1f3e101, 0xe1f3e111, 0xe1f3f3e1, 0xe1f3f3f3, 0xe1f3f301, 0xe1f3f311,
    0xe1f301e1, 0xe1f301f3, 0xe1f30101, 0xe1f30111, 0xe1f311e1, 0xe1f311f3, 0xe1f31101, 0xe1f31111,
    0xe101e1e1, 0xe101e1f3, 0xe101e101, 0xe101e111, 0xe101f3e1, 0xe101f3f3, 0xe101f301, 0xe101f311,
    0xe10101e1, 0xe10101f3, 0xe1010101, 0xe1010111, 0xe10111e1, 0xe10111f3, 0xe1011101, 0xe1011111,
    0xe111e1e1, 0xe111e1f3, 0xe111e101, 0xe111e111, 0xe111f3e1, 0xe111f3f3, 0xe111f301, 0xe111f311,
    0xe11101e1, 0xe11101f3, 0xe1110101, 0xe1110111, 0xe11111e1, 0xe11111f3, 0xe1111101, 0xe1111111,
    0xf3e1e1e1, 0xf3e1e1f3, 0xf3e1e101, 0xf3e1e111, 0xf3e1f3e1, 0xf3e1f3f3, 0xf3e1f301, 0xf3e1f311,
    0xf3e101e1, 0xf3e101f3, 0xf3e10101, 0xf3e10111, 0xf3e111e1, 0xf3e111f3, 0xf3e11101, 0xf3e11111,
    0xf3f3e1e1, 0xf3f3e1f3, 0xf3f3e101, 0xf3f3e111, 0xf3f3f3e1, 0xf3f3f3f3, 0xf3f3f301, 0xf3f3f311,
    0xf3f301e1, 0xf3f301f3, 0xf3f30101, 0xf3f30111, 0xf3f311e1, 0xf3f311f3, 0xf3f31101, 0xf3f31111,
    0xf301e1e1, 0xf301e1f3, 0xf301e101, 0xf301e111, 0xf301f3e1, 0xf301f3f3, 0xf301f301, 0xf301f311,
    0xf30101e1, 0xf30101f3, 0xf3010101, 0xf3010111, 0xf30111e1, 0xf30111f3, 0xf3011101, 0xf3011111,
    0xf311e1e1, 0xf311e1f3, 0xf311e101, 0xf311e111, 0xf311f3e1, 0xf311f3f3, 0xf311f301, 0xf311f311,
    0xf31101e1, 0xf31101f3, 0xf3110101, 0xf3110111, 0xf31111e1, 0xf31111f3, 0xf3111101, 0xf3111111,
    0x01e1e1e1, 0x01e1e1f3, 0x01e1e101, 0x01e1e111, 0x01e1f3e1, 0x01e1f3f3, 0x01e1f301, 0x01e1f311,
    0x01e101e1, 0x01e101f3, 0x01e10101, 0x01e10111, 0x01e111e1, 0x01e111f3, 0x01e11101, 0x01e11111,
    0x01f3e1e1, 0x01f3e1f3, 0x01f3e101, 0x01f3e111, 0x01f3f3e1, 0x01f3f3f3, 0x01f3f301, 0x01f3f311,
    0x01f301e1, 0x01f301f3, 0x01f30101, 0x01f30111, 0x01f311e1, 0x01f311f3, 0x01f31101, 0x01f31111,
    0x0101e1e1, 0x0101e1f3, 0x0101e101, 0x0101e111, 0x0101f3e1, 0x0101f3f3, 0x0101f301, 0x0101f311,
    0x010101e1, 0x010101f3, 0x01010101, 0x01010111, 0x010111e1, 0x010111f3, 0x01011101, 0x01011111,
    0x0111e1e1, 0x0111e1f3, 0x0111e101, 0x0111e111, 0x0111f3e1, 0x0111f3f3, 0x0111f301, 0x0111f311,
    0x011101e1, 0x011101f3, 0x01110101, 0x01110111, 0x011111e1, 0x011111f3, 0x01111101, 0x01111111,
    0x11e1e1e1, 0x11e1e1f3, 0x11e1e101, 0x11e1e111, 0x11e1f3e1, 0x11e1f3f3, 0x11e1f301, 0x11e1f311,
    0x11e101e1, 0x11e101f3, 0x11e10101, 0x11e10111, 0x11e111e1, 0x11e111f3, 0x11e11101, 0x11e11111,
    0x11f3e1e1, 0x11f3e1f3, 0x11f3e101, 0x11f3e111, 0x11f3f3e1, 0x11f3f3f3, 0x11f3f301, 0x11f3f311,
    0x11f301e1, 0x11f301f3, 0x11f30101, 0x11f30111, 0x11f311e1, 0x11f311f3, 0x11f31101, 0x11f31111,
    0x1101e1e1, 0x1101e1f3, 0x1101e101, 0x1101e111, 0x1101f3e1, 0x1101f3f3, 0x1101f301, 0x1101f311,
    0x110101e1, 0x110101f3, 0x11010101, 0x11010111, 0x110111e1, 0x110111f3, 0x11011101, 0x11011111,
    0x1111e1e1, 0x1111e1f3, 0x1111e101, 0x1111e111, 0x1111f3e1, 0x1111f3f3, 0x1111f301, 0x1111f311,
    0x111101e1, 0x111101f3, 0x11110101, 0x11110111, 0x111111e1, 0x111111f3, 0x11111101, 0x11111111,
    0xe6e6e6e6, 0xe6e6e6f8, 0xe6e6e606, 0xe6e6e616, 0xe6e6f8e6, 0xe6e6f8f8, 0xe6e6f806, 0xe6e6f816,
    0xe6e606e6, 0xe6e606f8, 0xe6e60606, 0xe6e60616, 0xe6e616e6, 0xe6e616f8, 0xe6e61606, 0xe6e61616,
    0xe6f8e6e6, 0xe6f8e6f8, 0xe6f8e606, 0xe6f8e616, 0xe6f8f8e6, 0xe6f8f8f8, 0xe6f8f806, 0xe6f8f816,
    0xe6f806e6, 0xe6f806f8, 0xe6f80606, 0xe6f80616, 0xe6f816e6, 0xe6f816f8, 0xe6f81606, 0xe6f81616,
    0xe606e6e6, 0xe606e6f8, 0xe606e606, 0xe606e616, 0xe606f8e6, 0xe606f8f8, 0xe606f806, 0xe606f816,
    0xe60606e6, 0xe60606f8, 0xe6060606, 0xe6060616, 0xe60616e6, 0xe60616f8, 0xe6061606, 0xe6061616,
    0xe616e6e6, 0xe616e6f8, 0xe616e606, 0xe616e616, 0xe616f8e6, 0xe616f8f8, 0xe616f806, 0xe616f816,
    0xe61606e6, 0xe61606f8, 0xe6160606, 0xe6160616, 0xe61616e6, 0xe61616f8, 0xe6161606, 0xe6161616,
    0xf8e6e6e6, 0xf8e6e6f8, 0xf8e6e606, 0xf8e6e616, 0xf8e6f8e6, 0xf8e6f8f8, 0xf8e6f806, 0xf8e6f816,
    0xf8e606e6, 0xf8e606f8, 0xf8e60606, 0xf8e60616, 0xf8e616e6, 0xf8e616f8, 0xf8e61606, 0xf8e61616,
    0xf8f8e6e6, 0xf8f8e6f8, 0xf8f8e606, 0xf8f8e616, 0xf8f8f8e6, 0xf8f8f8f8, 0xf8f8f806, 0xf8f8f816,
    0xf8f806e6, 0xf8f806f8, 0xf8f80606, 0xf8f80616, 0xf8f816e6, 0xf8f816f8, 0xf8f81606, 0xf8f81616,
    0xf806e6e6, 0xf806e6f8, 0xf806e606, 0xf806e616, 0xf806f8e6, 0xf806f8f8, 0xf806f806, 0xf806f816,
    0xf80606e6, 0xf80606f8, 0xf8060606, 0xf8060616, 0xf80616e6, 0xf80616f8, 0xf8061606, 0xf8061616,
    0xf816e6e6, 0xf816e6f8, 0xf816e606, 0xf816e616, 0xf816f8e6, 0xf816f8f8, 0xf816f806, 0xf816f816,
    0xf81606e6, 0xf81606f8, 0xf8160606, 0xf8160616, 0xf81616e6, 0xf81616f8, 0xf8161606, 0xf8161616,
    0x06e6e6e6, 0x06e6e6f8, 0x06e6e606, 0x06e6e616, 0x06e6f8e6, 0x06e6f8f8, 0x06e6f806, 0x06e6f816,
    0x06e606e6, 0x06e606f8, 0x06e60606, 0x06e60616, 0x06e616e6, 0x06e616f8, 0x06e61606, 0x06e61616,
    0x06f8e6e6, 0x06f8e6f8, 0x06f8e606, 0x06f8e616, 0x06f8f8e6, 0x06f8f8f8, 0x06f8f806, 0x06f8f816,
    0x06f806e6, 0x06f806f8, 0x06f80606, 0x06f80616, 0x06f816e6, 0x06f816f8, 0x06f81606, 0x06f81616,
    0x0606e6e6, 0x0606e6f8, 0x0606e606, 0x0606e616, 0x0606f8e6, 0x0606f8f8, 0x0606f806, 0x0606f816,
    0x060606e6, 0x060606f8, 0x06060606, 0x06060616, 0x060616e6, 0x060616f8, 0x06061606, 0x06061616,
    0x0616e6e6, 0x0616e6f8, 0x0616e606, 0x0616e616, 0x0616f8e6, 0x0616f8f8, 0x0616f806, 0x0616f816,
    0x061606e6, 0x061606f8, 0x06160606, 0x06160616, 0x061616e6, 0x061616f8, 0x06161606, 0x06161616,
    0x16e6e6e6, 0x16e6e6f8, 0x16e6e606, 0x16e6e616, 0x16e6f8e6, 0x16e6f8f8, 0x16e6f806, 0x16e6f816,
    0x16e606e6, 0x16e606f8, 0x16e60606, 0x16e60616, 0x16e616e6, 0x16e616f8, 0x16e61606, 0x16e61616,
    0x16f8e6e6, 0x16f8e6f8, 0x16f8e606, 0x16f8e616, 0x16f8f8e6, 0x16f8f8f8, 0x16f8f806, 0x16f8f816,
    0x16f806e6, 0x16f806f8, 0x16f80606, 0x16f80616, 0x16f816e6, 0x16f816f8, 0x16f81606, 0x16f81616,
    0x1606e6e6, 0x1606e6f8, 0x1606e606, 0x1606e616, 0x1606f8e6, 0x1606f8f8, 0x1606f806, 0x1606f816,
    0x160606e6, 0x160606f8, 0x16060606, 0x16060616, 0x160616e6, 0x160616f8, 0x16061606, 0x16061616,
    0x1616e6e6, 0x1616e6f8, 0x1616e606, 0x1616e616, 0x1616f8e6, 0x1616f8f8, 0x1616f806, 0x1616f816,
    0x161606e6, 0x161606f8, 0x16160606, 0x16160616, 0x161616e6, 0x161616f8, 0x16161606, 0x16161616,
};

__device__ __forceinline__ int int_from_table_4(const uint8_t * a8, const int * values) {
    return values[a8[0] | (a8[1] << 2) | (a8[2] << 4) | (a8[3] << 6)];
}

#define VDR_IQ2_K_Q8_1_MMVQ 4
#define VDR_IQ2_K_Q8_1_MMQ  4

__device__ __forceinline__ float vec_dot_iq2_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {

    // iqs is 0, 4, 8, 12, 16, 20, 24, 28
    // we have 16 packed quants (when cast to int)

    int i4 = iqs/4;  // 0...7. We will process q8 blocks 4*(i4/4), 4*(i4/4)+1, 4*(i4/4)+2, 4*(i4/4)+3
    const int32_t  * q8_1 = (const int *)bq8_1[4*(i4/4)+0].qs + 2*(i4%4);
    const int32_t  * q8_2 = (const int *)bq8_1[4*(i4/4)+1].qs + 2*(i4%4);
    const int32_t  * q8_3 = (const int *)bq8_1[4*(i4/4)+2].qs + 2*(i4%4);
    const int32_t  * q8_4 = (const int *)bq8_1[4*(i4/4)+3].qs + 2*(i4%4);

    const block_iq2_k * bq2 = (const block_iq2_k *) vbq + kbx;
    const uint32_t * q2 = (const uint32_t *)bq2->qs + 8*(i4/4) + 2*(i4%4);
    const uint16_t extra = bq2->extra >> (8*(i4/4) + (i4%4)/2);

    const int * all_values = (const int *)iq2k_table;
    const int * values;

    uint32_t val1 = q2[0], val2 = q2[1];

    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)&aux32;
    int v1, v2;

    // Block of 16: (32*(4*(i4/4)+k)+8*(i4%4))/16 = 8*(i4/4) + 2*k + (i4%4)/2
    // -> scales_l[4*(i4/4) + k] >> 4*(((i4%4)/2)%2)

    const uint32_t * scales = (const uint32_t *)bq2->scales;
    uint32_t s32 = __vsub4((scales[i4/4] >> 4*(((i4%4)/2)%2)) & 0x0f0f0f0f, 0x08080808);
    const int8_t * s8 = (const int8_t *)&s32;

    aux32[0] = ((val1 >> 0) & 0x03030303); aux32[1] = ((val2 >> 0) & 0x03030303); values = all_values + ((extra & 0x01) << 8);
    v1 = int_from_table_4(a8 + 0, values);
    v2 = int_from_table_4(a8 + 4, values);
    int sumi1 = ggml_cuda_dp4a(v2, q8_1[1], ggml_cuda_dp4a(v1, q8_1[0], 0)) * s8[0];

    aux32[0] = ((val1 >> 2) & 0x03030303); aux32[1] = ((val2 >> 2) & 0x03030303); values = all_values + ((extra & 0x04) << 6);
    v1 = int_from_table_4(a8 + 0, values);
    v2 = int_from_table_4(a8 + 4, values);
    int sumi2 = ggml_cuda_dp4a(v2, q8_2[1], ggml_cuda_dp4a(v1, q8_2[0], 0)) * s8[1];

    aux32[0] = ((val1 >> 4) & 0x03030303); aux32[1] = ((val2 >> 4) & 0x03030303); values = all_values + ((extra & 0x10) << 4);
    v1 = int_from_table_4(a8 + 0, values);
    v2 = int_from_table_4(a8 + 4, values);
    int sumi3 = ggml_cuda_dp4a(v2, q8_3[1], ggml_cuda_dp4a(v1, q8_3[0], 0)) * s8[2];

    aux32[0] = ((val1 >> 6) & 0x03030303); aux32[1] = ((val2 >> 6) & 0x03030303); values = all_values + ((extra & 0x40) << 2);
    v1 = int_from_table_4(a8 + 0, values);
    v2 = int_from_table_4(a8 + 4, values);
    int sumi4 = ggml_cuda_dp4a(v2, q8_4[1], ggml_cuda_dp4a(v1, q8_4[0], 0)) * s8[3];

    return __half2float(bq2->d) * (__low2float(bq8_1[4*(i4/4)+0].ds) * sumi1
                                +  __low2float(bq8_1[4*(i4/4)+1].ds) * sumi2
                                +  __low2float(bq8_1[4*(i4/4)+2].ds) * sumi3
                                +  __low2float(bq8_1[4*(i4/4)+3].ds) * sumi4);

}

#define VDR_IQ2_KS_Q8_1_MMVQ 4
#define VDR_IQ2_KS_Q8_1_MMQ  4

__device__ __forceinline__ float vec_dot_iq2_ks_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {

    float scale = *(const half *)vbq;
    const block_iq2_ks * bq2 = (const block_iq2_ks *)((const char *)vbq + sizeof(half)) + kbx;

    int i4 = iqs/4;  // 0...7. We will process q8 blocks 4*(i4/4), 4*(i4/4)+1, 4*(i4/4)+2, 4*(i4/4)+3
    const int32_t  * q8_1 = (const int *)bq8_1[4*(i4/4)+0].qs + 2*(i4%4);
    const int32_t  * q8_2 = (const int *)bq8_1[4*(i4/4)+1].qs + 2*(i4%4);
    const int32_t  * q8_3 = (const int *)bq8_1[4*(i4/4)+2].qs + 2*(i4%4);
    const int32_t  * q8_4 = (const int *)bq8_1[4*(i4/4)+3].qs + 2*(i4%4);

    const uint16_t * q2 = (const uint16_t *)bq2->qs + 16*(i4/4) + 4*(i4%4);
    const uint16_t extra = bq2->extra >> 4*(i4/4);

    const int * all_values = (const int *)iq2k_table;
    const int * values;

    uint32_t val1 = q2[0] | (q2[1] << 16), val2 = q2[2] | (q2[3] << 16);

    uint32_t aux32[2];
    const uint8_t * a8 = (const uint8_t *)&aux32;
    int v1, v2;

    int8_t s8[4];
    s8[0] = ((bq2->scales[2*(i4/4)+0] & 0xf) | ((extra >> 4) & 0x10)) - 16;
    s8[1] = ((bq2->scales[2*(i4/4)+0] >>  4) | ((extra >> 5) & 0x10)) - 16;
    s8[2] = ((bq2->scales[2*(i4/4)+1] & 0xf) | ((extra >> 6) & 0x10)) - 16;
    s8[3] = ((bq2->scales[2*(i4/4)+1] >>  4) | ((extra >> 7) & 0x10)) - 16;

    aux32[0] = ((val1 >> 0) & 0x03030303); aux32[1] = ((val2 >> 0) & 0x03030303); values = all_values + ((extra & 0x01) << 8);
    v1 = int_from_table_4(a8 + 0, values);
    v2 = int_from_table_4(a8 + 4, values);
    int sumi1 = ggml_cuda_dp4a(v2, q8_1[1], ggml_cuda_dp4a(v1, q8_1[0], 0)) * s8[0];

    aux32[0] = ((val1 >> 2) & 0x03030303); aux32[1] = ((val2 >> 2) & 0x03030303); values = all_values + ((extra & 0x02) << 7);
    v1 = int_from_table_4(a8 + 0, values);
    v2 = int_from_table_4(a8 + 4, values);
    int sumi2 = ggml_cuda_dp4a(v2, q8_2[1], ggml_cuda_dp4a(v1, q8_2[0], 0)) * s8[1];

    aux32[0] = ((val1 >> 4) & 0x03030303); aux32[1] = ((val2 >> 4) & 0x03030303); values = all_values + ((extra & 0x04) << 6);
    v1 = int_from_table_4(a8 + 0, values);
    v2 = int_from_table_4(a8 + 4, values);
    int sumi3 = ggml_cuda_dp4a(v2, q8_3[1], ggml_cuda_dp4a(v1, q8_3[0], 0)) * s8[2];

    aux32[0] = ((val1 >> 6) & 0x03030303); aux32[1] = ((val2 >> 6) & 0x03030303); values = all_values + ((extra & 0x08) << 5);
    v1 = int_from_table_4(a8 + 0, values);
    v2 = int_from_table_4(a8 + 4, values);
    int sumi4 = ggml_cuda_dp4a(v2, q8_4[1], ggml_cuda_dp4a(v1, q8_4[0], 0)) * s8[3];

    return scale * (__low2float(bq8_1[4*(i4/4)+0].ds) * sumi1
                 +  __low2float(bq8_1[4*(i4/4)+1].ds) * sumi2
                 +  __low2float(bq8_1[4*(i4/4)+2].ds) * sumi3
                 +  __low2float(bq8_1[4*(i4/4)+3].ds) * sumi4);
}

#define VDR_IQ3_K_Q8_1_MMVQ 4
#define VDR_IQ3_K_Q8_1_MMQ  4

static const __device__ uint16_t iq3k_table[128] = {
    0xc1c1, 0xc1d8, 0xc1e9, 0xc1f6, 0xc101, 0xc10d, 0xc11c, 0xc12f, 0xd8c1, 0xd8d8, 0xd8e9, 0xd8f6, 0xd801, 0xd80d, 0xd81c, 0xd82f,
    0xe9c1, 0xe9d8, 0xe9e9, 0xe9f6, 0xe901, 0xe90d, 0xe91c, 0xe92f, 0xf6c1, 0xf6d8, 0xf6e9, 0xf6f6, 0xf601, 0xf60d, 0xf61c, 0xf62f,
    0x01c1, 0x01d8, 0x01e9, 0x01f6, 0x0101, 0x010d, 0x011c, 0x012f, 0x0dc1, 0x0dd8, 0x0de9, 0x0df6, 0x0d01, 0x0d0d, 0x0d1c, 0x0d2f,
    0x1cc1, 0x1cd8, 0x1ce9, 0x1cf6, 0x1c01, 0x1c0d, 0x1c1c, 0x1c2f, 0x2fc1, 0x2fd8, 0x2fe9, 0x2ff6, 0x2f01, 0x2f0d, 0x2f1c, 0x2f2f,
    0xc5c5, 0xc5dc, 0xc5ed, 0xc5fa, 0xc505, 0xc511, 0xc520, 0xc533, 0xdcc5, 0xdcdc, 0xdced, 0xdcfa, 0xdc05, 0xdc11, 0xdc20, 0xdc33,
    0xedc5, 0xeddc, 0xeded, 0xedfa, 0xed05, 0xed11, 0xed20, 0xed33, 0xfac5, 0xfadc, 0xfaed, 0xfafa, 0xfa05, 0xfa11, 0xfa20, 0xfa33,
    0x05c5, 0x05dc, 0x05ed, 0x05fa, 0x0505, 0x0511, 0x0520, 0x0533, 0x11c5, 0x11dc, 0x11ed, 0x11fa, 0x1105, 0x1111, 0x1120, 0x1133,
    0x20c5, 0x20dc, 0x20ed, 0x20fa, 0x2005, 0x2011, 0x2020, 0x2033, 0x33c5, 0x33dc, 0x33ed, 0x33fa, 0x3305, 0x3311, 0x3320, 0x3333,
};

__device__ __forceinline__ int int_from_table_2(const uint8_t * a8, const uint16_t * values) {
    return values[a8[0] | (a8[1] << 3)] | (values[a8[2] | (a8[3] << 3)] << 16);
}

__device__ __forceinline__ float vec_dot_iq3_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iiqs) {
    const block_iq3_k * bq3 = (const block_iq3_k *) vbq + kbx;

    int iqs = iiqs/4;
    const int ib128 = iqs/4;  // 0 or 1. 0 works on quants 0...127, 1 on quants 128...255
                              // Each thread processes 8 quants in each of the 4 32-blocks
    const int il8   = iqs%4;  // 0...3. 0 works on quants 0...7, 1 on quants 8...15, 2 on 16...23, 3 on 24...31
    const int shift = 4*(il8/2);

    const uint16_t * ql = (const uint16_t *)bq3->qs + 16*ib128 + 4*il8;
    const uint16_t * qh = (const uint16_t *)bq3->qh + 4*il8;

    uint32_t aux32;
    const uint8_t * aux8 = (const uint8_t *)&aux32;

    const int hshift = 4*(1-ib128);
    const uint16_t sh = bq3->scales_h >> (8*ib128 + il8/2);

    const uint8_t extra = bq3->extra >> (8*ib128 + il8/2);
    const uint16_t * values1 = iq3k_table + ((extra << 6) & 0x40);
    const uint16_t * values2 = iq3k_table + ((extra << 5) & 0x40);
    const uint16_t * values3 = iq3k_table + ((extra << 4) & 0x40);
    const uint16_t * values4 = iq3k_table + ((extra << 3) & 0x40);

    const int * q8;
    int sumi[4] = {0, 0, 0, 0};
    int v;
    for (int i = 0; i < 2; ++i) {
        uint32_t vl = ql[2*i+0] | (ql[2*i+1] << 16);
        uint32_t vh = ((qh[2*i+0] | (qh[2*i+1] << 16)) << hshift) >> 2;

        q8 = (const int *)bq8_1[4*ib128+0].qs + 2*il8;
        aux32 = (vl & 0x03030303) | (vh & 0x04040404);
        v = int_from_table_2(aux8, values1);
        sumi[0] = ggml_cuda_dp4a(v, q8[i], sumi[0]);
        vl >>= 2; vh >>= 1;

        q8 += sizeof(block_q8_1)/4;
        aux32 = (vl & 0x03030303) | (vh & 0x04040404);
        v = int_from_table_2(aux8, values2);
        sumi[1] = ggml_cuda_dp4a(v, q8[i], sumi[1]);
        vl >>= 2; vh >>= 1;

        q8 += sizeof(block_q8_1)/4;
        aux32 = (vl & 0x03030303) | (vh & 0x04040404);
        v = int_from_table_2(aux8, values3);
        sumi[2] = ggml_cuda_dp4a(v, q8[i], sumi[2]);
        vl >>= 2; vh >>= 1;

        q8 += sizeof(block_q8_1)/4;
        aux32 = (vl & 0x03030303) | (vh & 0x04040404);
        v = int_from_table_2(aux8, values4);
        sumi[3] = ggml_cuda_dp4a(v, q8[i], sumi[3]);

    }
    const float d = __half2float(bq3->d);
    const uint16_t * sl16 = (const uint16_t *)bq3->scales_l + 2*ib128;
    aux32 = ((((sl16[0] | (sl16[1] << 16)) >> shift) & 0x0f0f0f0f) << 1) | 0x01010101;
    return d * (__low2float(bq8_1[4*ib128+0].ds) * aux8[0] * (sh & 0x01 ? -1 : 1) * sumi[0] +
                __low2float(bq8_1[4*ib128+1].ds) * aux8[1] * (sh & 0x04 ? -1 : 1) * sumi[1] +
                __low2float(bq8_1[4*ib128+2].ds) * aux8[2] * (sh & 0x10 ? -1 : 1) * sumi[2] +
                __low2float(bq8_1[4*ib128+3].ds) * aux8[3] * (sh & 0x40 ? -1 : 1) * sumi[3]);

}

__device__ __forceinline__ float vec_dot_iq1_bn_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {

    half d16; memcpy(&d16, vbq, sizeof(d16));
    float scale = d16;
    const block_iq1_bn * bq1 = (const block_iq1_bn *)((const char *)vbq + sizeof(d16)) + kbx;

    // iqs is 0 or 1

    int sumi = 0;
#if __CUDA_ARCH__ >= MIN_CC_DP4A // lowest compute capability for integer intrinsics
    uint16_t mult[2];
    mult[1] = iqs == 0 ? 27 : 3;
    mult[0] = mult[1] + (mult[1] << 1);
    const int * q8 = (const int *)bq8_1[iqs].qs;
    int val[4];
    for (int l = 0; l < 2; ++l) {
        int8_t * a = (int8_t *)val;
        const int i16 = 2*iqs + l;
        for (int k = 0; k < 3; ++k) {
            uint16_t q = bq1->ql[3*i16+k];
            for (int j = 4; j >= 0; --j) {
                uint16_t v = q & 0xff;
                v += v << 1;
                a[j] = v >> 8;
                q += q << 1;
            }
            a += 5;
        }
        uint16_t v = (mult[l]*bq1->extra) & 0xff;
        v += v << 1;
        *a = v >> 8;
        sumi = __dp4a(val[0], q8[4*l+0], __dp4a(val[1], q8[4*l+1], __dp4a(val[2], q8[4*l+2], __dp4a(val[3], q8[4*l+3], sumi))));
    }
    float2 d8 = __half22float2(bq8_1[iqs].ds);
    return scale * (d8.x * sumi - d8.y);
#else
    static const uint16_t k_mult[5] = {81, 27, 9, 3, 1};
    const int8_t * q8 = bq8_1[iqs].qs;
    for (int l = 0; l < 2; ++l) {
        const int i16 = 2*iqs + l;
        for (int k = 0; k < 3; ++k) {
            uint8_t q = bq1->ql[3*i16+k];
            for (int j = 0; j < 5; ++j) {
                uint8_t v = k_mult[j]*q;
                int8_t vs = (v + (v >> 1)) >> 7;
                sumi += q8[j]*(vs - 1);
            }
            q8 += 5;
        }
        uint8_t v = k_mult[i16]*bq1->extra;
        int8_t vs = (v + (v >> 1)) >> 7;
        sumi += q8[0]*(vs - 1);
        q8++;
    }
    return scale * __low2float(bq8_1[iqs].ds) * sumi;
#endif
}

__device__ __forceinline__ float vec_dot_iq2_bn_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs) {

    float scale = *(const float *)vbq;
    const block_iq2_bn * bq2 = (const block_iq2_bn *)((const char *)vbq + sizeof(float)) + kbx;

    // iqs is 0 or 1

#if __CUDA_ARCH__ >= MIN_CC_DP4A // lowest compute capability for integer intrinsics
    auto qs  = (const int *)bq2->qs + 2*iqs;
    auto q8l = (const int *)bq8_1[0].qs + 2*iqs;
    auto q8h = (const int *)bq8_1[1].qs + 2*iqs;
    int sumi1 = 0, sumi2 = 0, sumi3 = 0, sumi4 = 0;
    for (int j = 0; j < 2; ++j) {
        int vl = qs[j];
        int vh = qs[j] >> 4;
        sumi1 = __dp4a(vl & 0x03030303, q8l[j+0], sumi1);
        sumi2 = __dp4a(vl & 0x0c0c0c0c, q8l[j+4], sumi2);
        sumi3 = __dp4a(vh & 0x03030303, q8h[j+0], sumi3);
        sumi4 = __dp4a(vh & 0x0c0c0c0c, q8h[j+4], sumi4);
    }
    auto d8l = __half22float2(bq8_1[0].ds);
    auto d8h = __half22float2(bq8_1[1].ds);
    return scale * (d8l.x * (sumi1 + 0.25f*sumi2) + d8h.x * (sumi3 + 0.25f * sumi4) - 0.5f*d8l.y - 0.5f*d8h.y);
#else
    int sumi1 = 0, sumi2 = 0, sumi3 = 0, sumi4 = 0;
    auto q8l = bq8_1[0].qs + 8*iqs;
    auto q8h = bq8_1[1].qs + 8*iqs;
    auto qs  = bq2->qs + 8*iqs;
    for (int j = 0; j < 8; ++j) {
        sumi1 += q8l[j+ 0] * (qs[j] & 0x03);
        sumi2 += q8l[j+16] * (qs[j] & 0x0c);
        sumi3 += q8h[j+ 0] * (qs[j] & 0x30);
        sumi4 += q8h[j+16] * (qs[j] & 0xc0);
    }
    auto d8l = __half22float2(bq8_1[0].ds);
    auto d8h = __half22float2(bq8_1[1].ds);
    return scale * (d8l.x * (sumi1 + 0.25f*sumi2) + 0.0625f * d8h.x*(sumi3 + 0.25f*sumi4) - 0.5f*d8l.y - 0.5f*d8h.y);
#endif
}

} // namespace

void mul_mat_vec_iq2_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_K, VDR_IQ2_K_Q8_1_MMVQ, vec_dot_iq2_k_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq3_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ3_K, VDR_IQ3_K_Q8_1_MMVQ, vec_dot_iq3_k_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq4_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_K, VDR_IQ4_K_Q8_1_MMVQ, vec_dot_iq4_k_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq4_ks_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_KS, VDR_IQ4_KS_Q8_1_MMVQ, vec_dot_iq4_ks_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq4_kss_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_KSS, VDR_IQ4_KSS_Q8_1_MMVQ, vec_dot_iq4_kss_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq2_ks_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_KS, VDR_IQ2_KS_Q8_1_MMVQ, vec_dot_iq2_ks_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq5_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ5_K, VDR_IQ5_K_Q8_1_MMVQ, vec_dot_iq5_k_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq6_k_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {

    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ6_K, VDR_IQ6_K_Q8_1_MMVQ, vec_dot_iq6_k_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq1_bn_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ1_BN, 1, vec_dot_iq1_bn_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}

void mul_mat_vec_iq2_bn_q8_1_cuda(
    const void * vx, const void * vy, float * dst, const char * ids_data,
    const int ncols_x, const int nrows_x, const int nrows_y, const int ncols_y, const int nrows_dst,
    const int ne2, const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, int64_t ids_nb0, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_BN, 1, vec_dot_iq2_bn_q8_1>(vx, vy, dst, ids_data, ncols_x, nrows_x, nrows_y, ncols_y, nrows_dst, ne2, nb02, nb12, nb2, ids_nb0, stream);
}
