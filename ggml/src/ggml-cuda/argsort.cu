#include "hip/hip_runtime.h"
//
// Copyright (C) 2023-2024 The ggml authors
// Copyright (C) 2024 Iwan Kawrakow
// MIT license
// SPDX-License-Identifier: MIT
//
#include "argsort.cuh"
#include "sumrows.cuh"

template<typename T>
static inline __device__ void ggml_cuda_swap(T & a, T & b) {
    T tmp = a;
    a = b;
    b = tmp;
}

struct store_ser {
    constexpr static bool has_thresh = true;
    int   min_experts;
    float thresh_experts;
    store_ser(int min, float thresh) : min_experts(min), thresh_experts(thresh) {}
};

struct store {
    constexpr static bool has_thresh = false;
};

template<ggml_sort_order order, typename Store, typename dst_t>
static __global__ void k_argsort_f32_T(const float * x, dst_t * dst, const int ncols, int ncols_pad, int ntop, Store s) {
//        int min_experts, float thresh_experts) {
    // bitonic sort
    int col = threadIdx.x;
    int row = blockIdx.y;

    if (col >= ncols_pad) {
        return;
    }

    const float * x_row = x + row * ncols;
    extern __shared__ int dst_row[];

    // initialize indices
    dst_row[col] = col;

    __syncthreads();

    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            int ixj = col ^ j;
            if (ixj > col) {
                if ((col & k) == 0) {
                    if (dst_row[col] >= ncols ||
                        (dst_row[ixj] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] > x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] < x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                } else {
                    if (dst_row[ixj] >= ncols ||
                        (dst_row[col] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] < x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] > x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }

    if constexpr (Store::has_thresh) {
        __syncthreads();
        float max_val = x_row[dst_row[0]];
        if (col < ntop) {
            if constexpr (std::is_same_v<dst_t, int>) {
                dst[row * ntop + col] = col < s.min_experts || x_row[dst_row[col]] >= s.thresh_experts*max_val ? dst_row[col] : -1;
            } else {
                dst[row * ntop + col] = col < s.min_experts || x_row[dst_row[col]] >= s.thresh_experts*max_val ? x_row[dst_row[col]] : 0.f;
            }
        }
    } else {
        if (col < ntop) {
            if constexpr (std::is_same_v<dst_t, int>) {
                dst[row * ntop + col] = dst_row[col];
            } else {
                dst[row * ntop + col] = x_row[dst_row[col]];
            }
        }
    }
}

#if 0
// Somehow this is not working. Someone sees the bug?
template<ggml_sort_order order>
static __global__ void k_topk_sum(const float * x, float * dst, const int ncols, int ncols_pad, int n_top_k) {
    // bitonic sort
    int col = threadIdx.x;
    int row = blockIdx.y;

    if (col >= ncols_pad) {
        return;
    }

    const float * x_row = x + row * ncols;
    extern __shared__ int dst_row[];

    // initialize indices
    dst_row[col] = col;

    __syncthreads();

    for (int k = 2; k <= ncols_pad; k *= 2) {
        for (int j = k / 2; j > 0; j /= 2) {
            int ixj = col ^ j;
            if (ixj > col) {
                if ((col & k) == 0) {
                    if (dst_row[col] >= ncols ||
                        (dst_row[ixj] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] > x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] < x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                } else {
                    if (dst_row[ixj] >= ncols ||
                        (dst_row[col] < ncols && (order == GGML_SORT_ORDER_ASC ?
                            x_row[dst_row[col]] < x_row[dst_row[ixj]] :
                            x_row[dst_row[col]] > x_row[dst_row[ixj]]))
                    ) {
                        ggml_cuda_swap(dst_row[col], dst_row[ixj]);
                    }
                }
            }
            __syncthreads();
        }
    }

    float val = col < n_top_k ? x[dst_row[col]] : 0;
    val = warp_reduce_sum(val);
    if (blockDim.x > WARP_SIZE) {
        __syncthreads();
        auto s_sum = dst_row;
        const int        warp_id = threadIdx.x / WARP_SIZE;
        const int        lane_id = threadIdx.x % WARP_SIZE;
        if (lane_id == 0) {
            s_sum[warp_id] = val;
        }
        __syncthreads();
        val = 0.0f;
        if (lane_id < (static_cast<int>(blockDim.x) / WARP_SIZE)) {
            val = s_sum[lane_id];
        }
        val = warp_reduce_sum(val);
    }

    if (col == 0) {
        dst[row] = val;
    }
}
#endif

static __global__ void k_apply_mask(float * dst, const int * groups,
        const int n_top_groups, const int n_per_group, const int ncols) {
    int row = blockIdx.y;
    for (int col = threadIdx.x; col < n_top_groups*n_per_group; col += blockDim.x) {
        int ig = groups[row*n_top_groups + col / n_per_group];
        int ic = col % n_per_group;
        dst[row*ncols + ig*n_per_group + ic] = -INFINITY;
    }
}

static int next_power_of_2(int x) {
    int n = 1;
    while (n < x) {
        n *= 2;
    }
    return n;
}

template <typename dst_t>
static void argsort_f32_T_cuda(const float * x, dst_t * dst, const int ncols, const int nrows, int ntop,
        ggml_sort_order order, int min_experts, float thresh_experts, hipStream_t stream) {
    // bitonic sort requires ncols to be power of 2
    const int ncols_pad = next_power_of_2(ncols);

    const dim3 block_dims(ncols_pad, 1, 1);
    const dim3 block_nums(1, nrows, 1);
    const size_t shared_mem = ncols_pad * sizeof(int);

    // FIXME: this limit could be raised by ~2-4x on Ampere or newer
    GGML_ASSERT(shared_mem <= ggml_cuda_info().devices[ggml_cuda_get_device()].smpb);

    if (order == GGML_SORT_ORDER_ASC) {
        if (min_experts >= 0 && min_experts < ncols && thresh_experts > 0) {
            k_argsort_f32_T<GGML_SORT_ORDER_ASC, store_ser><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad,
                    ntop, {min_experts, thresh_experts});
        } else {
            k_argsort_f32_T<GGML_SORT_ORDER_ASC, store><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad, ntop, {});
        }
    } else if (order == GGML_SORT_ORDER_DESC) {
        if (min_experts >= 0 && min_experts < ncols && thresh_experts > 0) {
            k_argsort_f32_T<GGML_SORT_ORDER_DESC, store_ser><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad,
                    ntop, {min_experts, thresh_experts});
        } else {
            k_argsort_f32_T<GGML_SORT_ORDER_DESC, store><<<block_nums, block_dims, shared_mem, stream>>>(x, dst, ncols, ncols_pad, ntop, {});
        }
    } else {
        GGML_ABORT("fatal error");
    }
}

void ggml_cuda_op_argsort(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    enum ggml_sort_order order = (enum ggml_sort_order) dst->op_params[0];

    argsort_f32_T_cuda(src0_d, (int *)dst_d, ncols, nrows, ncols, order, -1, 0.f, stream);
}

void ggml_cuda_op_argsort_thresh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_I32);
    GGML_ASSERT(ggml_is_contiguous(src0));

    const int64_t ncols = src0->ne[0];
    const int64_t nrows = ggml_nrows(src0);

    int min_experts = dst->op_params[0];
    float thresh;
    memcpy(&thresh, dst->op_params + 1, sizeof(float));

    argsort_f32_T_cuda(src0_d, (int *)dst_d, ncols, nrows, ncols, GGML_SORT_ORDER_DESC, min_experts, thresh, stream);
}

#if 0
static void ggml_cuda_op_topk_sum(ggml_backend_cuda_context & ctx, const float * src, float * dst, int ncols, int nrows, int n_top_k) {

    GGML_ASSERT(n_top_k <= ncols);

    const int ncols_pad = next_power_of_2(ncols);

    const dim3 block_dims(ncols_pad, 1, 1);
    const dim3 block_nums(1, nrows, 1);
    const size_t shared_mem = std::max(ncols_pad, WARP_SIZE) * sizeof(int);
    GGML_ASSERT(shared_mem <= ggml_cuda_info().devices[ggml_cuda_get_device()].smpb);

    k_topk_sum<GGML_SORT_ORDER_DESC><<<block_nums, block_dims, shared_mem, ctx.stream()>>>(src, dst, ncols, ncols_pad, n_top_k);
}
#endif

void ggml_cuda_op_grouped_topk(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    auto src = dst->src[0];
    GGML_ASSERT(dst->type == GGML_TYPE_I32);
    GGML_ASSERT(src->type == GGML_TYPE_F32);
    GGML_ASSERT(ggml_nrows(src) == ggml_nrows(dst));

    auto nrows = ggml_nrows(src);

    int n_groups     = dst->op_params[0];
    int n_top_groups = dst->op_params[1];
    int nk           = dst->op_params[2];

    int ne00 = src->ne[0];
    int ne0  = dst->ne[0];
    GGML_ASSERT(ne0 <= ne00);
    GGML_ASSERT(ne00%n_groups == 0);
    int n_per_group = ne00/n_groups;
    GGML_ASSERT(nk <= n_per_group);
    GGML_ASSERT(n_top_groups < n_groups);
    int n_discarded_groups = n_groups - n_top_groups;

    ggml_cuda_pool_alloc<float> sorted_group_scores(ctx.pool(), nk*nrows*n_groups);
    argsort_f32_T_cuda((const float *)src->data, sorted_group_scores.get(), n_per_group, nrows*n_groups, nk,
            GGML_SORT_ORDER_DESC, -1, 0.0f, ctx.stream());
    CUDA_CHECK(hipGetLastError());
    ggml_cuda_pool_alloc<float> group_scores(ctx.pool(), nrows*n_groups);
    sum_rows_f32_cuda((const float *)sorted_group_scores.get(), group_scores.get(), nk, nrows*n_groups, ctx.stream());
    CUDA_CHECK(hipGetLastError());

    // This is not working for some reason, so we resort to the slightly less efficient implementation above
    //ggml_cuda_pool_alloc<float> group_scores(ctx.pool(), nrows*n_groups);
    //ggml_cuda_op_topk_sum(ctx, (const float *)src->data, group_scores.get(), n_per_group, nrows*n_groups, nk);
    ////sum_rows_f32_cuda((const float *)src->data, group_scores.get(), n_per_group, nrows*n_groups, ctx.stream());
    //CUDA_CHECK(hipGetLastError());

    ggml_cuda_pool_alloc<int> discarded_groups(ctx.pool(), nrows*n_discarded_groups);
    argsort_f32_T_cuda(group_scores.get(), discarded_groups.get(), n_groups, nrows, n_discarded_groups, GGML_SORT_ORDER_ASC, -1, 0.0f, ctx.stream());
    CUDA_CHECK(hipGetLastError());

    {
        const dim3 block_dims(WARP_SIZE, 1, 1);
        const dim3 block_nums(1, nrows, 1);
        hipStream_t stream = ctx.stream();
        k_apply_mask<<<block_nums, block_dims, 0, ctx.stream()>>>((float *)src->data, discarded_groups.get(), n_discarded_groups, n_per_group, ne00);
        CUDA_CHECK(hipGetLastError());
    }

    argsort_f32_T_cuda((const float *)src->data, (int *)dst->data, ne00, nrows, ne0, GGML_SORT_ORDER_DESC, -1, 0.0f, ctx.stream());

}
