#include "hip/hip_runtime.h"
#include "binbcast.cuh"

static __device__ __forceinline__ float op_repeat(const float a, const float b) {
    return b;
    GGML_UNUSED(a);
}

static __device__ __forceinline__ float op_add(const float a, const float b) {
    return a + b;
}

static __device__ __forceinline__ float op_mul(const float a, const float b) {
    return a * b;
}

static __device__ __forceinline__ float op_div(const float a, const float b) {
    return a / b;
}

template<float (*bin_op)(const float, const float), typename src0_t, typename src1_t, typename dst_t>
static __global__ void k_bin_bcast(const src0_t * src0, const src1_t * src1, dst_t * dst,
        int ne0, int ne1, int ne2, int ne3,
        int ne10, int ne11, int ne12, int ne13,
        /*int s0, */ int s1,  int s2,  int s3,
        /*int s00,*/ int s01, int s02, int s03,
        /*int s10,*/ int s11, int s12, int s13) {
    const int i0s = blockDim.x*blockIdx.x + threadIdx.x;
    const int i1 = (blockDim.y*blockIdx.y + threadIdx.y);
    const int i2 = (blockDim.z*blockIdx.z + threadIdx.z) / ne3;
    const int i3 = (blockDim.z*blockIdx.z + threadIdx.z) % ne3;

    if (i0s >= ne0 || i1 >= ne1 || i2 >= ne2 || i3 >= ne3) {
        return;
    }

    const int i11 = i1 % ne11;
    const int i12 = i2 % ne12;
    const int i13 = i3 % ne13;

    const size_t i_src0 =  i3*s03 +  i2*s02 +  i1*s01;
    const size_t i_src1 = i13*s13 + i12*s12 + i11*s11;
    const size_t i_dst  =  i3*s3  +  i2*s2  +  i1*s1;

    const src0_t * src0_row = src0 + i_src0;
    const src1_t * src1_row = src1 + i_src1;
    dst_t * dst_row = dst + i_dst;

    for (int i0 = i0s; i0 < ne0; i0 += blockDim.x*gridDim.x) {
        const int i10 = i0 % ne10;
        dst_row[i0] = (dst_t)bin_op(src0 ? (float)src0_row[i0] : 0.0f, (float)src1_row[i10]);
    }
}

template<float (*bin_op)(const float, const float), typename src0_t, typename src1_t, typename dst_t>
static __global__ void k_bin_bcast_unravel(const src0_t * src0, const src1_t * src1, dst_t * dst,
        int ne0, int ne1, int ne2, int ne3,
        int ne10, int ne11, int ne12, int ne13,
        /*int s0, */ int s1,  int s2,  int s3,
        /*int s00,*/ int s01, int s02, int s03,
        /*int s10,*/ int s11, int s12, int s13) {

    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    const int i3 = i/(ne2*ne1*ne0);
    const int i2 = (i/(ne1*ne0)) % ne2;
    const int i1 = (i/ne0) % ne1;
    const int i0 = i % ne0;

    if (i0 >= ne0 || i1 >= ne1 || i2 >= ne2 || i3 >= ne3) {
        return;
    }

    const int i11 = i1 % ne11;
    const int i12 = i2 % ne12;
    const int i13 = i3 % ne13;

    const size_t i_src0 =  i3*s03 +  i2*s02 +  i1*s01;
    const size_t i_src1 = i13*s13 + i12*s12 + i11*s11;
    const size_t i_dst  =  i3*s3  +  i2*s2  +  i1*s1;

    const src0_t * src0_row = src0 + i_src0;
    const src1_t * src1_row = src1 + i_src1;
    dst_t * dst_row = dst + i_dst;

    const int i10 = i0 % ne10;
    dst_row[i0] = (dst_t)bin_op(src0 ? (float)src0_row[i0] : 0.0f, (float)src1_row[i10]);
}

template<float (*bin_op)(const float, const float)>
struct bin_bcast_cuda {
    template<typename src0_t, typename src1_t, typename dst_t>
    void operator()(const struct ggml_tensor * src0, const struct ggml_tensor * src1, struct ggml_tensor * dst,
            const src0_t * src0_dd, const src1_t * src1_dd, dst_t * dst_dd,
            hipStream_t stream) {

        GGML_TENSOR_BINARY_OP_LOCALS

        int nr0 = ne10/ne0;
        int nr1 = ne11/ne1;
        int nr2 = ne12/ne2;
        int nr3 = ne13/ne3;

        int nr[4] = { nr0, nr1, nr2, nr3 };

        // collapse dimensions until first broadcast dimension
        int64_t cne[] = {ne0, ne1, ne2, ne3};
        int64_t cne0[] = {ne00, ne01, ne02, ne03};
        int64_t cne1[] = {ne10, ne11, ne12, ne13};

        size_t cnb[] = {nb0, nb1, nb2, nb3};
        size_t cnb0[] = {nb00, nb01, nb02, nb03};
        size_t cnb1[] = {nb10, nb11, nb12, nb13};

        auto collapse = [](int64_t cne[]) {
            cne[0] *= cne[1];
            cne[1] = cne[2];
            cne[2] = cne[3];
            cne[3] = 1;
        };

        auto collapse_nb = [](size_t cnb[], const int64_t cne[]) {
            cnb[1] *= cne[1];
            cnb[2] *= cne[2];
            cnb[3] *= cne[3];
        };

        if (ggml_is_contiguous(src0) && ggml_is_contiguous(src1) && ggml_is_contiguous(dst)) {
            for (int i = 0; i < 4; i++) {
                if (nr[i] != 1) {
                    break;
                }
                if (i > 0) {
                    collapse_nb(cnb, cne);
                    collapse_nb(cnb0, cne0);
                    collapse_nb(cnb1, cne1);
                    collapse(cne);
                    collapse(cne0);
                    collapse(cne1);
                }
            }
        }

        {
            int64_t ne0 = cne[0];
            int64_t ne1 = cne[1];
            int64_t ne2 = cne[2];
            int64_t ne3 = cne[3];

            //int64_t ne00 = cne0[0]; GGML_UNUSED(ne00);
            //int64_t ne01 = cne0[1]; GGML_UNUSED(ne01);
            //int64_t ne02 = cne0[2]; GGML_UNUSED(ne02);
            //int64_t ne03 = cne0[3]; GGML_UNUSED(ne03);

            int64_t ne10 = cne1[0];
            int64_t ne11 = cne1[1];
            int64_t ne12 = cne1[2];
            int64_t ne13 = cne1[3];

            size_t nb0 = cnb[0];
            size_t nb1 = cnb[1];
            size_t nb2 = cnb[2];
            size_t nb3 = cnb[3];

            size_t nb00 = cnb0[0];
            size_t nb01 = cnb0[1];
            size_t nb02 = cnb0[2];
            size_t nb03 = cnb0[3];

            size_t nb10 = cnb1[0];
            size_t nb11 = cnb1[1];
            size_t nb12 = cnb1[2];
            size_t nb13 = cnb1[3];

            size_t s0 = nb0 / sizeof(dst_t);
            size_t s1 = nb1 / sizeof(dst_t);
            size_t s2 = nb2 / sizeof(dst_t);
            size_t s3 = nb3 / sizeof(dst_t);

            size_t s10 = nb10 / sizeof(src1_t);
            size_t s11 = nb11 / sizeof(src1_t);
            size_t s12 = nb12 / sizeof(src1_t);
            size_t s13 = nb13 / sizeof(src1_t);

            size_t s00 = nb00 / sizeof(src0_t);
            size_t s01 = nb01 / sizeof(src0_t);
            size_t s02 = nb02 / sizeof(src0_t);
            size_t s03 = nb03 / sizeof(src0_t);

            GGML_ASSERT(nb0 % sizeof(dst_t) == 0);
            GGML_ASSERT(nb1 % sizeof(dst_t) == 0);
            GGML_ASSERT(nb2 % sizeof(dst_t) == 0);
            GGML_ASSERT(nb3 % sizeof(dst_t) == 0);

            GGML_ASSERT(nb00 % sizeof(src0_t) == 0);
            GGML_ASSERT(nb01 % sizeof(src0_t) == 0);
            GGML_ASSERT(nb02 % sizeof(src0_t) == 0);
            GGML_ASSERT(nb03 % sizeof(src0_t) == 0);

            GGML_ASSERT(nb10 % sizeof(src1_t) == 0);
            GGML_ASSERT(nb11 % sizeof(src1_t) == 0);
            GGML_ASSERT(nb12 % sizeof(src1_t) == 0);
            GGML_ASSERT(nb13 % sizeof(src1_t) == 0);

            GGML_ASSERT(s0 == 1);
            GGML_ASSERT(s00 == 1);
            GGML_ASSERT(s10 == 1);

            const int block_size = 128;

            int64_t hne0 = std::max(ne0/2LL, 1LL);

            dim3 block_dims;
            block_dims.x = std::min<unsigned int>(hne0, block_size);
            block_dims.y = std::min<unsigned int>(ne1, block_size / block_dims.x);
            block_dims.z = std::min(std::min<unsigned int>(ne2*ne3, block_size / block_dims.x / block_dims.y), 64U);

            dim3 block_nums(
                (hne0 + block_dims.x - 1) / block_dims.x,
                (ne1 + block_dims.y - 1) / block_dims.y,
                (ne2*ne3 + block_dims.z - 1) / block_dims.z
            );

            if (block_nums.z > 65535) {
                // this is the maximum number of blocks in z dimension, fallback to 1D grid kernel
                int block_num = (ne0*ne1*ne2*ne3 + block_size - 1) / block_size;
                k_bin_bcast_unravel<bin_op><<<block_num, block_size, 0, stream>>>(
                    src0_dd, src1_dd, dst_dd,
                    ne0, ne1, ne2, ne3,
                    ne10, ne11, ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00, */ s01, s02, s03,
                    /* s10, */ s11, s12, s13);
            } else {
                k_bin_bcast<bin_op><<<block_nums, block_dims, 0, stream>>>(
                    src0_dd, src1_dd, dst_dd,
                    ne0, ne1, ne2, ne3,
                    ne10, ne11, ne12, ne13,
                    /* s0, */ s1, s2, s3,
                    /* s00, */ s01, s02, s03,
                    /* s10, */ s11, s12, s13);
            }
        }
    }
};

template<class op>
static void ggml_cuda_op_bin_bcast(
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst,
    const void * src0_dd, const void * src1_dd, void * dst_dd, hipStream_t stream) {

    //GGML_ASSERT(src1->type == GGML_TYPE_F32);

    if (src1->type == GGML_TYPE_F32) {
        if (src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32) {
            op()(src0, src1, dst, (const float *)src0_dd, (const float *)src1_dd, (float *)dst_dd, stream);
        } else if (src0->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F16) {
            op()(src0, src1, dst, (const half *) src0_dd, (const float *)src1_dd, (half *) dst_dd, stream);
        } else if (src0->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F32) {
            op()(src0, src1, dst, (const half *) src0_dd, (const float *)src1_dd, (float *)dst_dd, stream);
        } else {
            fprintf(stderr, "%s: unsupported types: dst: %s, src0: %s, src1: %s\n", __func__,
                    ggml_type_name(dst->type), ggml_type_name(src0->type), ggml_type_name(src1->type));
            GGML_ABORT("fatal error");
        }
    }
    else if (src1->type == GGML_TYPE_F16) {
        if (src0->type == GGML_TYPE_F32 && dst->type == GGML_TYPE_F32) {
            op()(src0, src1, dst, (const float *)src0_dd, (const half *)src1_dd, (float *)dst_dd, stream);
        } else if (src0->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F16) {
            op()(src0, src1, dst, (const half *) src0_dd, (const half *)src1_dd, (half *) dst_dd, stream);
        } else if (src0->type == GGML_TYPE_F16 && dst->type == GGML_TYPE_F32) {
            op()(src0, src1, dst, (const half *) src0_dd, (const half *)src1_dd, (float *)dst_dd, stream);
        } else {
            fprintf(stderr, "%s: unsupported types: dst: %s, src0: %s, src1: %s\n", __func__,
                    ggml_type_name(dst->type), ggml_type_name(src0->type), ggml_type_name(src1->type));
            GGML_ABORT("fatal error");
        }
    }
    else {
        GGML_ABORT("fatal error");
    }
}

void ggml_cuda_op_repeat(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    GGML_ASSERT(dst->type == dst->src[0]->type);
    if (dst->type == GGML_TYPE_F32 || dst->type == GGML_TYPE_F16) {
        ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_repeat>>(dst, dst->src[0], dst, nullptr, dst->src[0]->data, dst->data, ctx.stream());
        return;
    }
    auto src = dst->src[0];
    auto bs = ggml_blck_size(src->type);
    auto ts = ggml_type_size(src->type);
    if (src->nb[0] != ts || ts*(src->ne[0]/bs) % 2 != 0) {
        fprintf(stderr, "%s: unsupported case type = %s, nb[0] = %zu, type_size = %zu\n", __func__, ggml_type_name(src->type), src->nb[0], ts);
        GGML_ABORT("fatal error");
    }
    auto aux_src = *src;
    aux_src.type = GGML_TYPE_F16;
    aux_src.ne[0] = ts*(src->ne[0]/bs)/2;
    aux_src.nb[0] = 2;
    auto aux_dst = *dst;
    aux_dst.type = GGML_TYPE_F16;
    aux_dst.ne[0] = ts*(dst->ne[0]/bs)/2;
    aux_dst.nb[0] = 2;
    aux_dst.src[0] = &aux_src;
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_repeat>>(&aux_dst, &aux_src, &aux_dst, nullptr, dst->src[0]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_add(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_add>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

static __global__ void scale_f32_l(const float * x, float * dst, const void * data, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    const float * scale = (const float *)data;
    dst[i] = scale[0] * x[i];
}

static void scale_f32_cuda_l(const float * x, float * dst, const void * data, const int k, hipStream_t stream) {
    constexpr int CUDA_SCALE_BLOCK_SIZE = 512; //256;
    const int num_blocks = (k + CUDA_SCALE_BLOCK_SIZE - 1) / CUDA_SCALE_BLOCK_SIZE;
    scale_f32_l<<<num_blocks, CUDA_SCALE_BLOCK_SIZE, 0, stream>>>(x, dst, data, k);
}

static void ggml_cuda_op_scale_tensor(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float scale;
    memcpy(&scale, dst->src[1]->data, sizeof(float));

    scale_f32_cuda_l(src0_d, dst_d, dst->src[1]->data, ggml_nelements(src0), stream);
}

void ggml_cuda_op_mul(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    if (ggml_nelements(dst->src[1]) == 1 && dst->src[1]->type == GGML_TYPE_F32 && dst->src[0]->type == GGML_TYPE_F32) {
        ggml_cuda_op_scale_tensor(ctx, dst);
        return;
    }
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_mul>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}

void ggml_cuda_op_div(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_bin_bcast<bin_bcast_cuda<op_div>>(dst->src[0], dst->src[1], dst, dst->src[0]->data, dst->src[1]->data, dst->data, ctx.stream());
}
