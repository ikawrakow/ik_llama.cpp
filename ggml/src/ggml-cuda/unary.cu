#include "hip/hip_runtime.h"
//
// Copyright (C) 2023-2024 The ggml authors
// Copyright (C) 2024 Iwan Kawrakow
// MIT license
// SPDX-License-Identifier: MIT
//

#include "unary.cuh"

static __global__ void gelu_f32(const float * x, float * dst, const int k) {
    const float GELU_COEF_A    = 0.044715f;
    const float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    float xi = x[i];
    dst[i] = 0.5f*xi*(1.0f + tanhf(SQRT_2_OVER_PI*xi*(1.0f + GELU_COEF_A*xi*xi)));
}

static __global__ void gelu_quick_f32(const float * x, float * dst, int k) {
    const float GELU_QUICK_COEF = -1.702f;
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = x[i] * (1.0f / (1.0f + expf(GELU_QUICK_COEF * x[i])));
}

static __global__ void silu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / (1.0f + expf(-x[i]));
}

static __global__ void swiglu_f32(const float * x, float * dst, const int k, const int ne0, const int64_t nb1) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    const int row = i/ne0;
    const int idx = i%ne0;
    const int j   = row*nb1 + idx;
    dst[i] = x[j] * x[j + ne0] / (1.0f + expf(-x[j]));
}

static __global__ void fused_mul_silu_f32(const float * x, const float * y, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * y[i] / (1.0f + expf(-x[i]));
}

static __global__ void fused_mul_relu_f32(const float * x, const float * y, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0) * y[i];
}

static __global__ void fused_mul_gelu_f32(const float * x, const float * y, float * dst, const int k) {
    constexpr float GELU_COEF_A    = 0.044715f;
    constexpr float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    float xi = x[i];
    dst[i] = 0.5f*xi*y[i]*(1.0f + tanhf(SQRT_2_OVER_PI*xi*(1.0f + GELU_COEF_A*xi*xi)));
}

static __global__ void tanh_f32(const float * x, float * dst, int k) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = tanhf(x[i]);
}

static __global__ void relu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0);
}

static __global__ void sigmoid_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = 1.0f / (1.0f + expf(-x[i]));
}

static __global__ void biased_sigmoid_f32(const float * x, const float * bias, float * dst, float * dst_biased, const int k, const int ncols) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = 1.0f / (1.0f + expf(-x[i]));
    dst_biased[i] = dst[i] + bias[i % ncols];
}

static __global__ void hardsigmoid_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fminf(1.0f, fmaxf(0.0f, (x[i] + 3.0f) / 6.0f));
}

static __global__ void hardswish_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * fminf(1.0f, fmaxf(0.0f, (x[i] + 3.0f) / 6.0f));
}

static __global__ void leaky_relu_f32(const float * x, float * dst, const int k, const float negative_slope) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0) + fminf(x[i], 0.0f) * negative_slope;
}

static __global__ void sqr_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * x[i];
}

static __global__ void sqrt_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sqrtf(x[i]);
}

static void gelu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu_f32<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void gelu_quick_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu_quick_f32<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void silu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

#if 0
static void swiglu_f32_cuda(const float * x, float * dst, const int k, const int64_t ne0, const int64_t nb1, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    swiglu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k, ne0, nb1);
}
#endif

static void fused_mul_silu_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    fused_mul_silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void fused_mul_relu_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    fused_mul_relu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void fused_mul_gelu_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    fused_mul_gelu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void tanh_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_TANH_BLOCK_SIZE - 1) / CUDA_TANH_BLOCK_SIZE;
    tanh_f32<<<num_blocks, CUDA_TANH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void relu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    relu_f32<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void sigmoid_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIGMOID_BLOCK_SIZE - 1) / CUDA_SIGMOID_BLOCK_SIZE;
    sigmoid_f32<<<num_blocks, CUDA_SIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void biased_sigmoid_f32_cuda(const float * x, const float * bias, float * dst, float * dst_biased, const int k, const int ncols, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIGMOID_BLOCK_SIZE - 1) / CUDA_SIGMOID_BLOCK_SIZE;
    biased_sigmoid_f32<<<num_blocks, CUDA_SIGMOID_BLOCK_SIZE, 0, stream>>>(x, bias, dst, dst_biased, k, ncols);
}

static void hardsigmoid_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSIGMOID_BLOCK_SIZE - 1) / CUDA_HARDSIGMOID_BLOCK_SIZE;
    hardsigmoid_f32<<<num_blocks, CUDA_HARDSIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void hardswish_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSWISH_BLOCK_SIZE - 1) / CUDA_HARDSWISH_BLOCK_SIZE;
    hardswish_f32<<<num_blocks, CUDA_HARDSWISH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void leaky_relu_f32_cuda(const float * x, float * dst, const int k, const float negative_slope, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    leaky_relu_f32<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k, negative_slope);
}

static void sqr_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQR_BLOCK_SIZE - 1) / CUDA_SQR_BLOCK_SIZE;
    sqr_f32<<<num_blocks, CUDA_SQR_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void sqrt_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQRT_BLOCK_SIZE - 1) / CUDA_SQRT_BLOCK_SIZE;
    sqrt_f32<<<num_blocks, CUDA_SQRT_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

void ggml_cuda_op_gelu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    gelu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_silu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    silu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

#if 0
void ggml_cuda_op_swiglu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(dst));
    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->ne[0] == src0->ne[0]/2);

    swiglu_f32_cuda(src0_d, dst_d, ggml_nelements(dst), dst->ne[0], src0->nb[1]/sizeof(float), stream);
}
#endif

void ggml_fused_mul_unary(ggml_backend_cuda_context & ctx, ggml_unary_op op,
        int64_t nelements, const float * src0_d, const float * src1_d, float * dst_d) {

    hipStream_t stream = ctx.stream();

    switch (op) {
        case GGML_UNARY_OP_SILU: fused_mul_silu_f32_cuda(src0_d, src1_d, dst_d, nelements, stream); break;
        case GGML_UNARY_OP_RELU: fused_mul_relu_f32_cuda(src0_d, src1_d, dst_d, nelements, stream); break;
        case GGML_UNARY_OP_GELU: fused_mul_gelu_f32_cuda(src0_d, src1_d, dst_d, nelements, stream); break;
        default: GGML_ASSERT(false);
    }
}

void ggml_cuda_op_fused_mul_unary(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_are_same_shape(src0, dst));
    GGML_ASSERT(ggml_are_same_shape(src0, src1));

    ggml_unary_op op = (ggml_unary_op)dst->op_params[0];

    ggml_fused_mul_unary(ctx, op, ggml_nelements(dst), (const float *)src0->data, (const float *)src1->data, (float *)dst->data);

    //hipStream_t stream = ctx.stream();

    //const float * src0_d = (const float *)src0->data;
    //const float * src1_d = (const float *)src1->data;
    //float * dst_d = (float *)dst->data;

    //switch (op) {
    //    case GGML_UNARY_OP_SILU: fused_mul_silu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
    //    case GGML_UNARY_OP_RELU: fused_mul_relu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
    //    case GGML_UNARY_OP_GELU: fused_mul_gelu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
    //    default: GGML_ASSERT(false);
    //}
}

void ggml_cuda_op_gelu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    gelu_quick_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_tanh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    tanh_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    relu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sigmoid_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_biased_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    GGML_ASSERT(dst->op == GGML_OP_ADD);
    GGML_ASSERT(dst->src[0]->op == GGML_OP_UNARY);
    const ggml_tensor * src0 = dst->src[0]->src[0];
    const ggml_tensor * bias = dst->src[1];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(bias->type == GGML_TYPE_F32);
    GGML_ASSERT(bias->ne[0] == src0->ne[0]);
    GGML_ASSERT(ggml_nrows(bias) == 1);

    biased_sigmoid_f32_cuda(src0_d, (const float *)bias->data, (float *)dst->src[0]->data, dst_d, ggml_nelements(src0), src0->ne[0], stream);
}

void ggml_cuda_op_hardsigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    hardsigmoid_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_hardswish(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    hardswish_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_leaky_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float negative_slope;
    memcpy(&negative_slope, dst->op_params, sizeof(float));

    leaky_relu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), negative_slope, stream);
}

void ggml_cuda_op_sqr(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sqr_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_sqrt(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sqrt_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

template <typename T>
static __global__ void swiglu_oai_kernel(const T * x, const T * g, T * dst, const int64_t k, const int64_t n, const int64_t o0, const int64_t o1, float alpha, float limit) {
    const int64_t i = int64_t(blockDim.x)*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    // perform base op and multiply with gate (either offset in same tensor or a separate one)
    const int64_t j0 = (i / n) * o0 + (i % n);
    const int64_t j1 = o0 == o1 ? j0 : (i / n) * o1 + (i % n);

    float xi = x[j0];
    float gi = g[j1];
    xi = fminf(xi, limit);
    gi = fmaxf(fminf(gi, limit), -limit);

    float out_glu = xi / (1.0f + expf(-xi * alpha));
    out_glu = out_glu * (1.0f + gi);

    dst[i] = out_glu;
}

template <typename T>
static void swiglu_oai_cuda(const T * x, const T * g, T * dst, const int64_t k, const int64_t n, const int64_t o0, const int64_t o1, const float alpha, const float limit, hipStream_t stream) {
    const int64_t num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    swiglu_oai_kernel<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, g, dst, k, n, o0, o1, alpha, limit);
}

void ggml_cuda_op_swiglu_oai(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    void * src0_d = src0->data;
    void * src1_d = src1 ? src1->data : src0->data;
    const int64_t src0_o = src0->nb[1];
    const int64_t src1_o = src1 ? src1->nb[1] : src0->nb[1];
    void * dst_d = dst->data;
    const int64_t nc = src1 ? src0->ne[0] : src0->ne[0] / 2;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous_1(src0));
    GGML_ASSERT(src0->nb[0] == ggml_element_size(src0));
    GGML_ASSERT(ggml_is_contiguous(dst));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(src0->type == dst->type);
    GGML_ASSERT(dst->ne[0] == nc);
    GGML_ASSERT(ggml_nrows(dst) == ggml_nrows(src0));

    if (src1) {
        GGML_ASSERT(ggml_is_contiguous_1(src1));
        GGML_ASSERT(src1->nb[0] == ggml_element_size(src1));
        GGML_ASSERT(src1->ne[0] == nc);
        GGML_ASSERT(src0->type == src1->type);
    }

    //const int32_t swapped = ((const int32_t *) dst->op_params)[1];
    const int32_t swapped = false; //ggml_get_op_params_i32(dst, 1);
    const float * op_params = (const float *)dst->op_params;
    const float alpha = op_params[2];
    const float limit = op_params[3];

    float * src0_p = (float *) src0_d;
    float * src1_p = (float *) src1_d;

    if (!src1) {
        src0_p += swapped ? nc : 0;
        src1_p += swapped ? 0 : nc;
    }

    swiglu_oai_cuda(src0_p, src1_p, (float *)dst_d, ggml_nelements(dst), nc,
            src0_o / sizeof(float), src1_o / sizeof(float), alpha, limit, stream);
}

void ggml_swiglu_oai_cuda_f32(const float * x, const float * g, float * dst, const int64_t k, const int64_t n,
        const int64_t o0, const int64_t o1, const float alpha, const float limit, hipStream_t stream) {
    swiglu_oai_cuda(x, g, dst, k, n, o0, o1, alpha, limit, stream);
}

// =========================================

static __device__ __forceinline__ float op_abs(float x) {
    return fabsf(x);
}

static __device__ __forceinline__ float op_sgn(float x) {
    return (x > 0.f ? 1.f : ((x < 0.f ? -1.f : 0.f)));
}

static __device__ __forceinline__ float op_neg(float x) {
    return -x;
}

static __device__ __forceinline__ float op_step(float x) {
    return x > 0.0f;
}

static __device__ __forceinline__ float op_gelu_erf(float x) {
    const float SQRT_2_INV = 0.70710678118654752440084436210484f;
    return 0.5f*x*(1.0f + erff(x*SQRT_2_INV));
}

static __device__ __forceinline__ float op_exp(float x) {
    return expf(x);
}

static __device__ __forceinline__ float op_sin(float x) {
    return sinf(x);
}

static __device__ __forceinline__ float op_cos(float x) {
    return cosf(x);
}

static __device__ __forceinline__ float op_log(float x) {
    return logf(x);
}

static __device__ __forceinline__ float op_elu(float x) {
    return (x > 0.f) ? x : expm1f(x);
}

static __device__ __forceinline__ float op_relu(float x) {
    return fmaxf(x, 0);
}

static __device__ __forceinline__ float op_gelu(float x) {
    const float GELU_COEF_A    = 0.044715f;
    const float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    return 0.5f*x*(1.0f + tanhf(SQRT_2_OVER_PI*x*(1.0f + GELU_COEF_A*x*x)));
}

static __device__ __forceinline__ float op_silu(float x) {
    return x / (1.0f + expf(-x));
}

static __device__ __forceinline__ float op_gelu_quick(float x) {
    const float GELU_QUICK_COEF = -1.702f;
    return x * (1.0f / (1.0f + expf(GELU_QUICK_COEF * x)));
}

template <float (*op)(float), typename T>
static __global__ void unary_op_kernel(const T * x, T * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    dst[i] = (T)op((float)x[i]);
}

template <float (*op)(float), typename T>
static void unary_cuda(const T * x, T * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_NEG_BLOCK_SIZE - 1) / CUDA_NEG_BLOCK_SIZE;
    unary_op_kernel<op><<<num_blocks, CUDA_NEG_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

template <float (*op)(float)>
void ggml_cuda_op_unary(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const void * src0_d = src0->data;
    void * dst_d = dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);

    if (src0->type == GGML_TYPE_F16) {
        unary_cuda<op>((const half *)src0_d, (half *)dst_d, ggml_nelements(src0), stream);
    } else {
        unary_cuda<op>((const float *)src0_d, (float *)dst_d, ggml_nelements(src0), stream);
    }
}

void ggml_cuda_op_abs(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_abs>(ctx, dst);
}

void ggml_cuda_op_sgn(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sgn>(ctx, dst);
}

void ggml_cuda_op_neg(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_neg>(ctx, dst);
}

void ggml_cuda_op_step(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_step>(ctx, dst);
}

void ggml_cuda_op_gelu_erf(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_gelu_erf>(ctx, dst);
}

void ggml_cuda_op_exp(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_exp>(ctx, dst);
}

// === gated ops

template <float (*op)(float), typename T>
static __global__ void unary_gated_op_kernel(const T * x, const T * g, T * dst, const int64_t k, const int64_t n, const int64_t o0, const int64_t o1) {
    const int64_t i = int64_t(blockDim.x)*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    // perform base op and multiply with gate (either offset in same tensor or a separate one)
    const int64_t j0 = (i / n) * o0 + (i % n);
    const int64_t j1 = o0 == o1 ? j0 : (i / n) * o1 + (i % n);

    dst[i] = (T)(op((float)x[j0]) * (float)g[j1]);
}

template <float (*op)(float), typename T>
static void unary_gated_cuda(const T * x, const T * g, T * dst, const int64_t k, const int64_t n, const int64_t o0, const int64_t o1, hipStream_t stream) {
    const int64_t num_blocks = (k + CUDA_GLU_BLOCK_SIZE - 1) / CUDA_GLU_BLOCK_SIZE;
    unary_gated_op_kernel<op><<<num_blocks, CUDA_GLU_BLOCK_SIZE, 0, stream>>>(x, g, dst, k, n, o0, o1);
}
template <float (*op)(float)>
void ggml_cuda_op_unary_gated(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    void * src0_d = src0->data;
    void * src1_d = src1 ? src1->data : src0->data;
    const int64_t src0_o = src0->nb[1];
    const int64_t src1_o = src1 ? src1->nb[1] : src0->nb[1];
    void * dst_d = dst->data;
    const int64_t nc = src1 ? src0->ne[0] : src0->ne[0] / 2;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous_1(src0));
    GGML_ASSERT(src0->nb[0] == ggml_element_size(src0));
    GGML_ASSERT(ggml_is_contiguous(dst));

    GGML_ASSERT(src0->type == GGML_TYPE_F32 || src0->type == GGML_TYPE_F16);
    GGML_ASSERT( dst->type == GGML_TYPE_F32 ||  dst->type == GGML_TYPE_F16);
    GGML_ASSERT(src0->type == dst->type);
    GGML_ASSERT(dst->ne[0] == nc);
    GGML_ASSERT(ggml_nrows(dst) == ggml_nrows(src0));

    if (src1) {
        GGML_ASSERT(ggml_is_contiguous_1(src1));
        GGML_ASSERT(src1->nb[0] == ggml_element_size(src1));
        GGML_ASSERT(src1->ne[0] == nc);
        GGML_ASSERT(src0->type == src1->type);
    }

    const int32_t swapped = ((const int32_t *) dst->op_params)[1];

    if (src0->type == GGML_TYPE_F16) {
        half * src0_p = (half *) src0_d;
        half * src1_p = (half *) src1_d;

        if (!src1) {
            src0_p += swapped ? nc : 0;
            src1_p += swapped ? 0 : nc;
        }

        unary_gated_cuda<op>(src0_p, src1_p, (half *)dst_d, ggml_nelements(dst), nc, src0_o / sizeof(half), src1_o / sizeof(half), stream);
    } else {
        float * src0_p = (float *) src0_d;
        float * src1_p = (float *) src1_d;

        if (!src1) {
            src0_p += swapped ? nc : 0;
            src1_p += swapped ? 0 : nc;
        }

        unary_gated_cuda<op>(src0_p, src1_p, (float *)dst_d, ggml_nelements(dst), nc, src0_o / sizeof(float), src1_o / sizeof(float), stream);
    }
}

void ggml_cuda_op_reglu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_relu>(ctx, dst);
}

void ggml_cuda_op_geglu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_gelu>(ctx, dst);
}

void ggml_cuda_op_swiglu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_silu>(ctx, dst);
}

void ggml_cuda_op_geglu_erf(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_gelu_erf>(ctx, dst);
}

void ggml_cuda_op_geglu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary_gated<op_gelu_quick>(ctx, dst);
}

void ggml_cuda_op_sin(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_sin>(ctx, dst);
}

void ggml_cuda_op_cos(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_cos>(ctx, dst);
}

void ggml_cuda_op_log(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_log>(ctx, dst);
}

void ggml_cuda_op_elu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    ggml_cuda_op_unary<op_elu>(ctx, dst);
}

