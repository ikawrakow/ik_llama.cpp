#include "hip/hip_runtime.h"
//
// Copyright (C) 2023-2024 The ggml authors
// Copyright (C) 2024 Iwan Kawrakow
// MIT license
// SPDX-License-Identifier: MIT
//

#include "unary.cuh"

static __global__ void gelu_f32(const float * x, float * dst, const int k) {
    const float GELU_COEF_A    = 0.044715f;
    const float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    float xi = x[i];
    dst[i] = 0.5f*xi*(1.0f + tanhf(SQRT_2_OVER_PI*xi*(1.0f + GELU_COEF_A*xi*xi)));
}

static __global__ void gelu_quick_f32(const float * x, float * dst, int k) {
    const float GELU_QUICK_COEF = -1.702f;
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = x[i] * (1.0f / (1.0f + expf(GELU_QUICK_COEF * x[i])));
}

static __global__ void silu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] / (1.0f + expf(-x[i]));
}

static __global__ void swiglu_f32(const float * x, float * dst, const int k, const int ne0, const int64_t nb1) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    const int row = i/ne0;
    const int idx = i%ne0;
    const int j   = row*nb1 + idx;
    dst[i] = x[j] * x[j + ne0] / (1.0f + expf(-x[j]));
}

static __global__ void fused_mul_silu_f32(const float * x, const float * y, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * y[i] / (1.0f + expf(-x[i]));
}

static __global__ void multi_add_f32(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, const char * src0, char * dst) {
    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;
    int64_t k = ne0*ne1;
    if (i >= k) {
        return;
    }
    int i1 = i / ne0;
    int i0 = i % ne0;
    float * result = (float *)(dst + i1*nb1);
    const float * s = (const float *)(src0 + i1*nb01) + i0;
    if (nused == 1) {
        result[i0] = s[0];
    } else {
        float sum = s[0] + s[ne0];
        for (int j = 2; j < nused; ++j) sum += s[j*ne0];
        result[i0] = sum;
    }
}

static __global__ void fused_mul_relu_f32(const float * x, const float * y, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0) * y[i];
}

static __global__ void fused_mul_gelu_f32(const float * x, const float * y, float * dst, const int k) {
    constexpr float GELU_COEF_A    = 0.044715f;
    constexpr float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    float xi = x[i];
    dst[i] = 0.5f*xi*y[i]*(1.0f + tanhf(SQRT_2_OVER_PI*xi*(1.0f + GELU_COEF_A*xi*xi)));
}

static __global__ void tanh_f32(const float * x, float * dst, int k) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = tanhf(x[i]);
}

static __global__ void relu_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0);
}

static __global__ void sigmoid_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = 1.0f / (1.0f + expf(-x[i]));
}

static __global__ void hardsigmoid_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = fminf(1.0f, fmaxf(0.0f, (x[i] + 3.0f) / 6.0f));
}

static __global__ void hardswish_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * fminf(1.0f, fmaxf(0.0f, (x[i] + 3.0f) / 6.0f));
}

static __global__ void leaky_relu_f32(const float * x, float * dst, const int k, const float negative_slope) {
    const int i  = blockDim.x*blockIdx.x + threadIdx.x;
    if (i >= k) {
        return;
    }
    dst[i] = fmaxf(x[i], 0) + fminf(x[i], 0.0f) * negative_slope;
}

static __global__ void sqr_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = x[i] * x[i];
}

static __global__ void sqrt_f32(const float * x, float * dst, const int k) {
    const int i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }
    dst[i] = sqrtf(x[i]);
}

static void gelu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu_f32<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void gelu_quick_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    gelu_quick_f32<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void silu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void swiglu_f32_cuda(const float * x, float * dst, const int k, const int64_t ne0, const int64_t nb1, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    swiglu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, dst, k, ne0, nb1);
}

static void fused_mul_silu_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SILU_BLOCK_SIZE - 1) / CUDA_SILU_BLOCK_SIZE;
    fused_mul_silu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void fused_mul_relu_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    fused_mul_relu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void fused_mul_gelu_f32_cuda(const float * x, const float * y, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    fused_mul_gelu_f32<<<num_blocks, CUDA_SILU_BLOCK_SIZE, 0, stream>>>(x, y, dst, k);
}

static void tanh_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_TANH_BLOCK_SIZE - 1) / CUDA_TANH_BLOCK_SIZE;
    tanh_f32<<<num_blocks, CUDA_TANH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void relu_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    relu_f32<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void sigmoid_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SIGMOID_BLOCK_SIZE - 1) / CUDA_SIGMOID_BLOCK_SIZE;
    sigmoid_f32<<<num_blocks, CUDA_SIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void hardsigmoid_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSIGMOID_BLOCK_SIZE - 1) / CUDA_HARDSIGMOID_BLOCK_SIZE;
    hardsigmoid_f32<<<num_blocks, CUDA_HARDSIGMOID_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void hardswish_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_HARDSWISH_BLOCK_SIZE - 1) / CUDA_HARDSWISH_BLOCK_SIZE;
    hardswish_f32<<<num_blocks, CUDA_HARDSWISH_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void leaky_relu_f32_cuda(const float * x, float * dst, const int k, const float negative_slope, hipStream_t stream) {
    const int num_blocks = (k + CUDA_RELU_BLOCK_SIZE - 1) / CUDA_RELU_BLOCK_SIZE;
    leaky_relu_f32<<<num_blocks, CUDA_RELU_BLOCK_SIZE, 0, stream>>>(x, dst, k, negative_slope);
}

static void sqr_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQR_BLOCK_SIZE - 1) / CUDA_SQR_BLOCK_SIZE;
    sqr_f32<<<num_blocks, CUDA_SQR_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void sqrt_f32_cuda(const float * x, float * dst, const int k, hipStream_t stream) {
    const int num_blocks = (k + CUDA_SQRT_BLOCK_SIZE - 1) / CUDA_SQRT_BLOCK_SIZE;
    sqrt_f32<<<num_blocks, CUDA_SQRT_BLOCK_SIZE, 0, stream>>>(x, dst, k);
}

static void multi_add_f32_cuda(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, const char * src0, char * dst, hipStream_t stream) {
    int64_t k = ne0 * ne1;
    const int num_blocks = (k + CUDA_MULTI_ADD_BLOCK_SIZE - 1) / CUDA_MULTI_ADD_BLOCK_SIZE;
    multi_add_f32<<<num_blocks, CUDA_MULTI_ADD_BLOCK_SIZE, 0, stream>>>(nused, ne0, ne1, nb1, nb01, src0, dst);
}

void ggml_cuda_op_multi_add(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->ne[2] == 1 && dst->ne[3] == 1);
    GGML_ASSERT(dst->nb[0] == sizeof(float));
    int nused = dst->op_params[0];
    GGML_ASSERT(nused >= 1);
    const char * src0 = (const char *)dst->src[0]->data;
    hipStream_t stream = ctx.stream();
    multi_add_f32_cuda(nused, dst->ne[0], dst->ne[1], dst->nb[1], dst->src[0]->nb[1], src0, (char *)dst->data, stream);
}

void ggml_cuda_op_gelu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    gelu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_silu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    silu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_swiglu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_is_contiguous(dst));
    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->ne[0] == src0->ne[0]/2);

    swiglu_f32_cuda(src0_d, dst_d, ggml_nelements(dst), dst->ne[0], src0->nb[1]/sizeof(float), stream);
}

void ggml_fused_mul_unary(ggml_backend_cuda_context & ctx, ggml_unary_op op,
        int64_t nelements, const float * src0_d, const float * src1_d, float * dst_d) {

    hipStream_t stream = ctx.stream();

    switch (op) {
        case GGML_UNARY_OP_SILU: fused_mul_silu_f32_cuda(src0_d, src1_d, dst_d, nelements, stream); break;
        case GGML_UNARY_OP_RELU: fused_mul_relu_f32_cuda(src0_d, src1_d, dst_d, nelements, stream); break;
        case GGML_UNARY_OP_GELU: fused_mul_gelu_f32_cuda(src0_d, src1_d, dst_d, nelements, stream); break;
        default: GGML_ASSERT(false);
    }
}

void ggml_cuda_op_fused_mul_unary(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    GGML_ASSERT(ggml_is_contiguous(src0));
    GGML_ASSERT(ggml_are_same_shape(src0, dst));
    GGML_ASSERT(ggml_are_same_shape(src0, src1));

    ggml_unary_op op = (ggml_unary_op)dst->op_params[0];

    ggml_fused_mul_unary(ctx, op, ggml_nelements(dst), (const float *)src0->data, (const float *)src1->data, (float *)dst->data);

    //hipStream_t stream = ctx.stream();

    //const float * src0_d = (const float *)src0->data;
    //const float * src1_d = (const float *)src1->data;
    //float * dst_d = (float *)dst->data;

    //switch (op) {
    //    case GGML_UNARY_OP_SILU: fused_mul_silu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
    //    case GGML_UNARY_OP_RELU: fused_mul_relu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
    //    case GGML_UNARY_OP_GELU: fused_mul_gelu_f32_cuda(src0_d, src1_d, dst_d, ggml_nelements(dst), stream); break;
    //    default: GGML_ASSERT(false);
    //}
}

void ggml_cuda_op_gelu_quick(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    gelu_quick_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_tanh(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    tanh_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    relu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_sigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sigmoid_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_hardsigmoid(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    hardsigmoid_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_hardswish(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    hardswish_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_leaky_relu(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    float negative_slope;
    memcpy(&negative_slope, dst->op_params, sizeof(float));

    leaky_relu_f32_cuda(src0_d, dst_d, ggml_nelements(src0), negative_slope, stream);
}

void ggml_cuda_op_sqr(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sqr_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

void ggml_cuda_op_sqrt(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const float * src0_d = (const float *)src0->data;
    float * dst_d = (float *)dst->data;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous(src0));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);

    sqrt_f32_cuda(src0_d, dst_d, ggml_nelements(src0), stream);
}

template <typename T>
static __global__ void swiglu_oai_kernel(const T * x, const T * g, T * dst, const int64_t k, const int64_t n, const int64_t o0, const int64_t o1, float alpha, float limit) {
    const int64_t i = int64_t(blockDim.x)*blockIdx.x + threadIdx.x;

    if (i >= k) {
        return;
    }

    // perform base op and multiply with gate (either offset in same tensor or a separate one)
    const int64_t j0 = (i / n) * o0 + (i % n);
    const int64_t j1 = o0 == o1 ? j0 : (i / n) * o1 + (i % n);

    float xi = x[j0];
    float gi = g[j1];
    xi = fminf(xi, limit);
    gi = fmaxf(fminf(gi, limit), -limit);

    float out_glu = xi / (1.0f + expf(-xi * alpha));
    out_glu = out_glu * (1.0f + gi);

    dst[i] = out_glu;
}

template <typename T>
static void swiglu_oai_cuda(const T * x, const T * g, T * dst, const int64_t k, const int64_t n, const int64_t o0, const int64_t o1, const float alpha, const float limit, hipStream_t stream) {
    const int64_t num_blocks = (k + CUDA_GELU_BLOCK_SIZE - 1) / CUDA_GELU_BLOCK_SIZE;
    swiglu_oai_kernel<<<num_blocks, CUDA_GELU_BLOCK_SIZE, 0, stream>>>(x, g, dst, k, n, o0, o1, alpha, limit);
}

void ggml_cuda_op_swiglu_oai(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];
    void * src0_d = src0->data;
    void * src1_d = src1 ? src1->data : src0->data;
    const int64_t src0_o = src0->nb[1];
    const int64_t src1_o = src1 ? src1->nb[1] : src0->nb[1];
    void * dst_d = dst->data;
    const int64_t nc = src1 ? src0->ne[0] : src0->ne[0] / 2;
    hipStream_t stream = ctx.stream();

    GGML_ASSERT(ggml_is_contiguous_1(src0));
    GGML_ASSERT(src0->nb[0] == ggml_element_size(src0));
    GGML_ASSERT(ggml_is_contiguous(dst));

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(src0->type == dst->type);
    GGML_ASSERT(dst->ne[0] == nc);
    GGML_ASSERT(ggml_nrows(dst) == ggml_nrows(src0));

    if (src1) {
        GGML_ASSERT(ggml_is_contiguous_1(src1));
        GGML_ASSERT(src1->nb[0] == ggml_element_size(src1));
        GGML_ASSERT(src1->ne[0] == nc);
        GGML_ASSERT(src0->type == src1->type);
    }

    //const int32_t swapped = ((const int32_t *) dst->op_params)[1];
    const int32_t swapped = false; //ggml_get_op_params_i32(dst, 1);
    const float * op_params = (const float *)dst->op_params;
    const float alpha = op_params[2];
    const float limit = op_params[3];

    float * src0_p = (float *) src0_d;
    float * src1_p = (float *) src1_d;

    if (!src1) {
        src0_p += swapped ? nc : 0;
        src1_p += swapped ? 0 : nc;
    }

    swiglu_oai_cuda(src0_p, src1_p, (float *)dst_d, ggml_nelements(dst), nc,
            src0_o / sizeof(float), src1_o / sizeof(float), alpha, limit, stream);
}

void ggml_swiglu_oai_cuda_f32(const float * x, const float * g, float * dst, const int64_t k, const int64_t n,
        const int64_t o0, const int64_t o1, const float alpha, const float limit, hipStream_t stream) {
    swiglu_oai_cuda(x, g, dst, k, n, o0, o1, alpha, limit, stream);
}
