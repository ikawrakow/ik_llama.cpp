#include "hip/hip_runtime.h"
#include "concat.cuh"

// contiguous kernels
static __global__ void concat_f32_dim0(const float * x, const float * y, float * dst, const int64_t ne0, const int64_t ne00) {
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;

    if (nidx < ne00) { // src0
        int offset_src =
            nidx +
            blockIdx.y * ne00 +
            blockIdx.z * ne00 * gridDim.y;
        dst[offset_dst] = x[offset_src];
    } else {
        int offset_src =
            (nidx - ne00) +
            blockIdx.y * (ne0 - ne00) +
            blockIdx.z * (ne0 - ne00) * gridDim.y;
        dst[offset_dst] = y[offset_src];
    }
}

// contiguous kernels
static __global__ void concat_f32_dim0(const float * x, const float * y, float * dst, const int64_t ne0, const int64_t ne00,
        int64_t nb02, int64_t nb12, int64_t nb2) {
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * nb2;

    if (nidx < ne00) { // src0
        int offset_src =
            nidx +
            blockIdx.y * ne00 +
            blockIdx.z * nb02;
        dst[offset_dst] = x[offset_src];
    } else {
        int offset_src =
            (nidx - ne00) +
            blockIdx.y * (ne0 - ne00) +
            blockIdx.z * nb12;
        dst[offset_dst] = y[offset_src];
    }
}

static __global__ void concat_f32_dim1(const float * x, const float * y, float * dst, const int64_t ne0, const int64_t ne01) {
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;

    if (blockIdx.y < ne01) { // src0
        int offset_src =
            nidx +
            blockIdx.y * ne0 +
            blockIdx.z * ne0 * ne01;
        dst[offset_dst] = x[offset_src];
    } else {
        int offset_src =
            nidx +
            (blockIdx.y - ne01) * ne0 +
            blockIdx.z * ne0 * (gridDim.y - ne01);
        dst[offset_dst] = y[offset_src];
    }
}

static __global__ void concat_f32_dim2(const float * x, const float * y, float * dst, const int64_t ne0, const int64_t ne02) {
    int nidx = threadIdx.x + blockIdx.x * blockDim.x;
    if (nidx >= ne0) {
        return;
    }

    int offset_dst =
        nidx +
        blockIdx.y * ne0 +
        blockIdx.z * ne0 * gridDim.y;

    if (blockIdx.z < ne02) { // src0
        int offset_src =
            nidx +
            blockIdx.y * ne0 +
            blockIdx.z * ne0 * gridDim.y;
        dst[offset_dst] = x[offset_src];
    } else {
        int offset_src =
            nidx +
            blockIdx.y * ne0 +
            (blockIdx.z - ne02) * ne0 *  gridDim.y;
        dst[offset_dst] = y[offset_src];
    }
}

static void concat_f32_cuda(const float * x, const float * y, float * dst, int ne00, int ne01, int ne02, int ne0, int ne1, int ne2, int dim, hipStream_t stream) {
    int num_blocks = (ne0 + CUDA_CONCAT_BLOCK_SIZE - 1) / CUDA_CONCAT_BLOCK_SIZE;
    if (dim == 0 && ne1 >= 65536) {
        int64_t nstep = (ne1 + 32767)/32768;
        for (int64_t istep = 0; istep < nstep; ++istep) {
            int64_t i1 = 32768*istep;
            int64_t n1 = i1 + 32768 <= ne1 ? 32768 : ne1 - i1;
            dim3 gridDim(num_blocks, n1, ne2);
            const float * xi = x + i1*ne00;
            const float * yi = y + i1*(ne0 - ne00);
            float * dst_i = dst + i1*ne0;
            concat_f32_dim0<<<gridDim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(xi, yi, dst_i, ne0, ne00, ne00*ne01, (ne0-ne00)*ne01, ne0*ne1);
        }
        return;
    }
    dim3 gridDim(num_blocks, ne1, ne2);
    if (dim == 0) {
        concat_f32_dim0<<<gridDim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(x, y, dst, ne0, ne00);
        //concat_f32_dim0<<<gridDim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(x, y, dst, ne0, ne00, ne00*ne01, (ne0-ne00)*ne01, ne0*ne1);
        return;
    }
    if (dim == 1) {
        concat_f32_dim1<<<gridDim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(x, y, dst, ne0, ne01);
        return;
    }
    concat_f32_dim2<<<gridDim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(x, y, dst, ne0, ne02);
}

// non-contiguous kernel (slow)
static __global__ void concat_f32_non_cont(
        const char * src0,
        const char * src1,
              char * dst,
           int64_t   ne00,
           int64_t   ne01,
           int64_t   ne02,
           int64_t   ne03,
          uint64_t   nb00,
          uint64_t   nb01,
          uint64_t   nb02,
          uint64_t   nb03,
           int64_t /*ne10*/,
           int64_t /*ne11*/,
           int64_t /*ne12*/,
           int64_t /*ne13*/,
          uint64_t   nb10,
          uint64_t   nb11,
          uint64_t   nb12,
          uint64_t   nb13,
           int64_t   ne0,
           int64_t /*ne1*/,
           int64_t /*ne2*/,
           int64_t /*ne3*/,
          uint64_t   nb0,
          uint64_t   nb1,
          uint64_t   nb2,
          uint64_t   nb3,
          int32_t   dim) {
    const int64_t i3 = blockIdx.z;
    const int64_t i2 = blockIdx.y;
    const int64_t i1 = blockIdx.x;

    int64_t o[4] = {0, 0, 0, 0};
    o[dim] = dim == 0 ? ne00 : (dim == 1 ? ne01 : (dim == 2 ? ne02 : ne03));

    const float * x;

    for (int i0 = threadIdx.x; i0 < ne0; i0 += blockDim.x) {
        if (i0 < ne00 && i1 < ne01 && i2 < ne02 && i3 < ne03) {
            x = (const float *)(src0 + (i3       )*nb03 + (i2       )*nb02 + (i1       )*nb01 + (i0       )*nb00);
        } else {
            x = (const float *)(src1 + (i3 - o[3])*nb13 + (i2 - o[2])*nb12 + (i1 - o[1])*nb11 + (i0 - o[0])*nb10);
        }

        float * y = (float *)(dst + i3*nb3 + i2*nb2 + i1*nb1 + i0*nb0);

        *y = *x;
    }
}


void ggml_cuda_op_concat(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    const ggml_tensor * src1 = dst->src[1];

    GGML_ASSERT(src0->type == src1->type && src0->type == dst->type);

    hipStream_t stream = ctx.stream();

    const int32_t dim = ((int32_t *) dst->op_params)[0];

    if (ggml_is_contiguous(src0) && ggml_is_contiguous(src1) &&
        (dim == 3 || (dim == 2 && dst->ne[3] == 1) || (dim == 1 && dst->ne[2]*dst->ne[3] == 1))) {
        const size_t size0 = ggml_nbytes(src0);
        const size_t size1 = ggml_nbytes(src1);
        CUDA_CHECK(hipMemcpyAsync((char *)dst->data,         src0->data, size0, hipMemcpyDeviceToDevice, stream));
        CUDA_CHECK(hipMemcpyAsync((char *)dst->data + size0, src1->data, size1, hipMemcpyDeviceToDevice, stream));
        return;
    }

    if (dim == 0 && src0->nb[0] == ggml_type_size(src0->type) && src1->nb[0] == ggml_type_size(src1->type) &&
            src0->nb[1] % sizeof(float) == 0 && src1->nb[1] % sizeof(float) == 0) {
        auto bs = ggml_blck_size(dst->type);
        auto ts = ggml_type_size(dst->type);
        auto ne00_eff = (src0->ne[0]/bs)*ts/sizeof(float);
        auto ne0_eff  = (dst->ne[0]/bs)*ts/sizeof(float);
        if (ggml_is_contiguous(src0) && ggml_is_contiguous(src1)) {
            //if (dst->ne[1] >= 65536 || dst->ne[2] >= 65536) {
            //    fprintf(stderr, "%s: ne1 = %ld, ne2 = %ld exceed max. blocks when computing %s\n", __func__, dst->ne[1], dst->ne[2], dst->name);
            //    GGML_ABORT("fatal error");
            //}
            const float * src0_d = (const float *)src0->data;
            const float * src1_d = (const float *)src1->data;
            float * dst_d = (float *)dst->data;
            //printf("%s(%s, %s): %ld %zu %zu  %ld %zu %zu\n", __func__, src0->name, src1->name, src0->ne[0], src0->nb[0], src0->nb[1], dst->ne[0], dst->nb[0], dst->nb[1]);
            for (int i3 = 0; i3 < dst->ne[3]; i3++) {
                concat_f32_cuda(
                        src0_d + i3 * (src0->nb[3] / 4),
                        src1_d + i3 * (src1->nb[3] / 4),
                        dst_d + i3 * ( dst->nb[3] / 4),
                        ne00_eff, src0->ne[1], src0->ne[2],
                        ne0_eff, dst->ne[1], dst->ne[2], dim, stream);
                        //src0->nb[1]/sizeof(float), src0->ne[1], src0->ne[2],
                        //dst->nb[1]/sizeof(float), dst->ne[1], dst->ne[2], dim, stream);
                        //src0->ne[0]*src0->nb[0]/sizeof(float), src0->ne[1], src0->ne[2],
                        //dst->ne[0]*dst->nb[0]/sizeof(float),  dst->ne[1],  dst->ne[2], dim, stream);
            }
        } else {
            //printf("%s(not contiguous): %s(%s) and %s(%s)\n", __func__, src0->name, ggml_type_name(src0->type), src1->name, ggml_type_name(src1->type));
            auto ne10_eff = (src1->ne[0]/bs)*ts/sizeof(float);
            dim3 grid_dim(dst->ne[1], dst->ne[2], dst->ne[3]);
            concat_f32_non_cont<<<grid_dim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(
                    (const char *)src0->data,
                    (const char *)src1->data,
                    (      char *)dst->data,
                    ne00_eff, src0->ne[1], src0->ne[2], src0->ne[3],
                    //src0->ne[0]*src0->nb[0]/sizeof(float), src0->ne[1], src0->ne[2], src0->ne[3],
                    sizeof(float), src0->nb[1], src0->nb[2], src0->nb[3],
                    ne10_eff, src1->ne[1], src1->ne[2], src1->ne[3],
                    //src1->ne[0]*src1->nb[0]/sizeof(float), src1->ne[1], src1->ne[2], src1->ne[3],
                    sizeof(float), src1->nb[1], src1->nb[2], src1->nb[3],
                    ne0_eff,  dst->ne[1],  dst->ne[2],  dst->ne[3],
                    //dst->ne[0]*dst->nb[0]/sizeof(float),  dst->ne[1],  dst->ne[2],  dst->ne[3],
                    sizeof(float),  dst->nb[1],  dst->nb[2],  dst->nb[3], dim);
        }
        return;
    }

    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->type  == GGML_TYPE_F32);

    if (ggml_is_contiguous(src0) && ggml_is_contiguous(src1)) {
        //if (dst->ne[1] >= 65536 || dst->ne[2] >= 65536) {
        //    fprintf(stderr, "%s: ne1 = %ld, ne2 = %ld exceed max. blocks when computing %s\n", __func__, dst->ne[1], dst->ne[2], dst->name);
        //    GGML_ABORT("fatal error");
        //}
        const float * src0_d = (const float *)src0->data;
        const float * src1_d = (const float *)src1->data;

        float * dst_d = (float *)dst->data;

        for (int i3 = 0; i3 < dst->ne[3]; i3++) {
            concat_f32_cuda(
                    src0_d + i3 * (src0->nb[3] / 4),
                    src1_d + i3 * (src1->nb[3] / 4),
                    dst_d + i3 * ( dst->nb[3] / 4),
                    src0->ne[0], src0->ne[1], src0->ne[2],
                    dst->ne[0],  dst->ne[1],  dst->ne[2], dim, stream);
        }
    } else {
        dim3 grid_dim(dst->ne[1], dst->ne[2], dst->ne[3]);
        concat_f32_non_cont<<<grid_dim, CUDA_CONCAT_BLOCK_SIZE, 0, stream>>>(
                (const char *)src0->data,
                (const char *)src1->data,
                (      char *)dst->data,
                src0->ne[0], src0->ne[1], src0->ne[2], src0->ne[3],
                src0->nb[0], src0->nb[1], src0->nb[2], src0->nb[3],
                src1->ne[0], src1->ne[1], src1->ne[2], src1->ne[3],
                src1->nb[0], src1->nb[1], src1->nb[2], src1->nb[3],
                dst->ne[0],  dst->ne[1],  dst->ne[2],  dst->ne[3],
                dst->nb[0],  dst->nb[1],  dst->nb[2],  dst->nb[3], dim);
    }
}
