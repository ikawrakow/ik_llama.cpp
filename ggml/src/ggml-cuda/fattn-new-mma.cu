#include "hip/hip_runtime.h"
// Adapted from https://github.com/ggml-org/llama.cpp/pull/13435
//
// Copyright (C) 2025 The ggml authors
// Copyright (C) 2025 Iwan Kawrakow
// MIT license
// SPDX-License-Identifier: MIT
//

#include "common.cuh"
#include "cp-async.cuh"
#include "mma_new.cuh"
#include "fattn-common.cuh"
#include "fattn-new-mma.cuh"

using namespace ggml_cuda_mma;

typedef tile<16,  8, half2> tile_A;
typedef tile< 8,  8, half2> tile_B;
typedef tile<16,  8, half2> tile_B_16;
typedef tile<16,  8, float> tile_C_KQ;
typedef tile<16, 16, float> tile_C_KQ_16;
typedef tile<16,  4, half2> tile_C_VKQ;
typedef tile<16,  8, half2> tile_C_VKQ_16;

// Config options for specific head sizes.
// Should not affect results, only speed/register pressure/shared memory use.
//
// nbatch_fa:      number of KV rows per softmax rescaling of KQ rowsums and VKQ accumulators.
// nwarps_max:     maximum number of warps per CUDA block, up to 8 warps in total can run per SM (given enough shared memory).
// Q_in_reg:       whether the Q values should be kept permanently in registers.
// nstages_target: targeted number of pipeline stages for cp_async (if available), 0 means synchronous data loading.
// nbatch_K2:      number of K half2 values in direction of DKQ to load in parallel.
// nbatch_V2:      number of V half2 values in direction of DV to load in parallel.
// nbatch_combine: number of VKQ half2 values in direction of DV to combine in parallel.

template <int DKQ, int DV>
struct fattn_mma_f16_config;

//
// The previous MMA version is better (faster)
// I'm keeping these around commented out for now,
// and only using the 576, 512 case.
// Perhaps the 256 head size needs a closer look
// to see if this implementation is better.
//
//template <>
//struct fattn_mma_f16_config< 64,  64> {
//    static constexpr int  nbatch_fa      = 64;
//    static constexpr int  nwarps_max     = 4;
//    static constexpr bool Q_in_reg       = true;
//    static constexpr int  nstages_target = 2;
//
//    static int get_nbatch_K2_host(const int /*cc*/, const int /*ncols*/) {
//        return 32;
//    }
//
//    static constexpr __device__ int get_nbatch_K2_device(int /*ncols*/) {
//        return 32;
//    }
//
//    static int get_nbatch_V2_host(const int /*cc*/, const int /*ncols*/) {
//        return 32;
//    }
//
//    static constexpr __device__ int get_nbatch_V2_device(int /*ncols*/) {
//        return 32;
//    }
//
//    static int get_nbatch_combine_host(const int /*cc*/, const int /*ncols*/) {
//        return 32;
//    }
//
//    static constexpr __device__ int get_nbatch_combine_device(int /*ncols*/) {
//        return 32;
//    }
//};
//
//template <>
//struct fattn_mma_f16_config< 80,  80> {
//    static constexpr int  nbatch_fa      = 64;
//    static constexpr int  nwarps_max     = 4;
//    static constexpr bool Q_in_reg       = true;
//    static constexpr int  nstages_target = 2;
//
//    static int get_nbatch_K2_host(const int /*cc*/, const int /*ncols*/) {
//        return 40;
//    }
//
//    static constexpr __device__ int get_nbatch_K2_device(int /*ncols*/) {
//        return 40;
//    }
//
//    static int get_nbatch_V2_host(const int /*cc*/, const int /*ncols*/) {
//        return 40;
//    }
//
//    static constexpr __device__ int get_nbatch_V2_device(int /*ncols*/) {
//        return 40;
//    }
//
//    static int get_nbatch_combine_host(const int /*cc*/, const int /*ncols*/) {
//        return 40;
//    }
//
//    static constexpr __device__ int get_nbatch_combine_device(int /*ncols*/) {
//        return 40;
//    }
//};
//
//template <>
//struct fattn_mma_f16_config< 96,  96> {
//    static constexpr int  nbatch_fa      = 64;
//    static constexpr int  nwarps_max     = 4;
//    static constexpr bool Q_in_reg       = true;
//    static constexpr int  nstages_target = 2;
//
//    static int get_nbatch_K2_host(const int /*cc*/, const int /*ncols*/) {
//        return 48;
//    }
//
//    static constexpr __device__ int get_nbatch_K2_device(int /*ncols*/) {
//        return 48;
//    }
//
//    static int get_nbatch_V2_host(const int /*cc*/, const int /*ncols*/) {
//        return 48;
//    }
//
//    static constexpr __device__ int get_nbatch_V2_device(int /*ncols*/) {
//        return 48;
//    }
//
//    static int get_nbatch_combine_host(const int /*cc*/, const int /*ncols*/) {
//        return 48;
//    }
//
//    static constexpr __device__ int get_nbatch_combine_device(int /*ncols*/) {
//        return 48;
//    }
//};
//
//template <>
//struct fattn_mma_f16_config<112, 112> {
//    static constexpr int  nbatch_fa      = 64;
//    static constexpr int  nwarps_max     = 4;
//    static constexpr bool Q_in_reg       = true;
//    static constexpr int  nstages_target = 2;
//
//    static int get_nbatch_K2_host(const int /*cc*/, const int /*ncols*/) {
//        return 56;
//    }
//
//    static constexpr __device__ int get_nbatch_K2_device(int /*ncols*/) {
//        return 56;
//    }
//
//    static int get_nbatch_V2_host(const int /*cc*/, const int /*ncols*/) {
//        return 56;
//    }
//
//    static constexpr __device__ int get_nbatch_V2_device(int /*ncols*/) {
//        return 56;
//    }
//
//    static int get_nbatch_combine_host(const int /*cc*/, const int /*ncols*/) {
//        return 56;
//    }
//
//    static constexpr __device__ int get_nbatch_combine_device(int /*ncols*/) {
//        return 56;
//    }
//};
//
//template <>
//struct fattn_mma_f16_config<128, 128> {
//    static constexpr int  nbatch_fa      = 64;
//    static constexpr int  nwarps_max     = 4;
//    static constexpr bool Q_in_reg       = true;
//    static constexpr int  nstages_target = 2;
//
//    static int get_nbatch_K2_host(const int /*cc*/, const int /*ncols*/) {
//        return 64;
//    }
//
//    static constexpr __device__ int get_nbatch_K2_device(int /*ncols*/) {
//        return 64;
//    }
//
//    static int get_nbatch_V2_host(const int /*cc*/, const int /*ncols*/) {
//        return 64;
//    }
//
//    static constexpr __device__ int get_nbatch_V2_device(int /*ncols*/) {
//        return 64;
//    }
//
//    static int get_nbatch_combine_host(const int /*cc*/, const int /*ncols*/) {
//        return 64;
//    }
//
//    static constexpr __device__ int get_nbatch_combine_device(int /*ncols*/) {
//        return 64;
//    }
//};
//
//template <>
//struct fattn_mma_f16_config<256, 256> {
//    static constexpr int  nbatch_fa      = 32;
//    static constexpr int  nwarps_max     = 4;
//    static constexpr bool Q_in_reg       = true;
//    static constexpr int  nstages_target = 2;
//
//    static int get_nbatch_K2_host(const int /*cc*/, const int /*ncols*/) {
//        return 128;
//    }
//
//    static constexpr __device__ int get_nbatch_K2_device(int /*ncols*/) {
//        return 128;
//    }
//
//    static int get_nbatch_V2_host(const int /*cc*/, const int /*ncols*/) {
//        return 128;
//    }
//
//    static constexpr __device__ int get_nbatch_V2_device(int /*ncols*/) {
//        return 128;
//    }
//
//    static int get_nbatch_combine_host(const int cc, const int ncols) {
//        if (ggml_cuda_highest_compiled_arch(cc) == CC_TURING) {
//            return ncols <= 16 ? 128 : 64;
//        }
//        return 64;
//    }
//
//    static constexpr __device__ int get_nbatch_combine_device(int ncols) {
//#if __CUDA_ARCH__ == CC_TURING
//        return ncols <= 16 ? 128 : 64;
//#else
//        GGML_UNUSED(ncols);
//        return 128;
//#endif // __CUDA_ARCH__ == CC_TURING
//    }
//};

template <>
struct fattn_mma_f16_config<576, 512> {
    static constexpr int  nbatch_fa      = 32;
    static constexpr int  nwarps_max     = 8;
    static constexpr bool Q_in_reg       = false;
    static constexpr int  nstages_target = 1;

    static int get_nbatch_K2_host(const int cc, const int ncols) {
        if (ggml_cuda_highest_compiled_arch(cc) == CC_TURING) {
            return ncols <= 16 ? 96 : 160;
        }
        return ncols <= 16 ? 288 : 160;
    }

    static constexpr __device__ int get_nbatch_K2_device(int ncols) {
#if __CUDA_ARCH__ == CC_TURING
        return ncols <= 16 ? 96 : 160;
#else
        return ncols <= 16 ? 288 : 160;
#endif // __CUDA_ARCH__ == CC_TURING
    }

    static int get_nbatch_V2_host(const int cc, const int ncols) {
        if (ggml_cuda_highest_compiled_arch(cc) == CC_TURING) {
            return ncols <= 16 ? 64 : 128;
        }
        return ncols <= 16 ? 256 : 128;
    }

    static constexpr __device__ int get_nbatch_V2_device(int ncols) {
#if __CUDA_ARCH__ == CC_TURING
        return ncols <= 16 ? 64 : 128;
#else
        return ncols <= 16 ? 256 : 128;
#endif // __CUDA_ARCH__ == CC_TURING
    }

    static int get_nbatch_combine_host(const int /*cc*/, const int /*ncols*/) {
        return 128;
    }

    static constexpr __device__ int get_nbatch_combine_device(int /*ncols*/) {
        return 128;
    }
};

// ------------------------------------------------------------------------------------------------------------------

// The compiler is always able to unroll loops if they contain continue expressions.
// In such cases loop unrolling can still be achieved via recursion:
template <int n>
struct ggml_cuda_unroll {
    template <typename Func, typename... Args>
    __device__ void operator()(const Func & f, Args... args) const {
        f(n - 1, args...);
        ggml_cuda_unroll<n - 1>{}(f, args...);
    }
};

template <>
struct ggml_cuda_unroll<1> {
    template <typename Func, typename... Args>
    __device__ void operator()(const Func & f, Args... args) const {
        f(0, args...);
    }
};

template<int stride_tile, int nwarps, int nbatch_fa, bool use_cp_async>
static __device__ __forceinline__ void flash_attn_ext_f16_load_tile(
        const half2 * const __restrict__ KV, half2 * const __restrict__ tile_KV, const int D2, const int stride_KV) {

    // K/V data is loaded with decreasing granularity for D for better memory bandwidth.
    // The minimum granularity with cp.async is 16 bytes, with synchronous data loading it's 4 bytes.

    if constexpr (use_cp_async) {
        constexpr int preload = 64;
        constexpr int h2_per_chunk = 16/sizeof(half2);
        const int chunks_per_row = D2 / h2_per_chunk;

        const unsigned int tile_KV_32 = ggml_cuda_cvta_generic_to_shared(tile_KV);

        auto load = [&] __device__ (auto n) {
            const int stride_k = WARP_SIZE >> n;
            const int k0_start = stride_k == WARP_SIZE ? 0 : chunks_per_row - chunks_per_row % (2*stride_k);
            const int k0_stop  =                             chunks_per_row - chunks_per_row % (1*stride_k);
            const int stride_i = WARP_SIZE / stride_k;

            if (k0_start == k0_stop) {
                return;
            }

#pragma unroll
            for (int i0 = 0; i0 < nbatch_fa; i0 += nwarps*stride_i) {
                const int i = i0 + threadIdx.y*stride_i + (stride_k == WARP_SIZE ? 0 : threadIdx.x / stride_k);

                if (i0 + nwarps*stride_i > nbatch_fa && i >= nbatch_fa) {
                    break;
                }

#pragma unroll
                for (int k0 = k0_start; k0 < k0_stop; k0 += stride_k) {
                    const int k = k0 + (stride_k == WARP_SIZE ? threadIdx.x : threadIdx.x % stride_k);

                    cp_async_cg_16<preload>(tile_KV_32 + i*(stride_tile*sizeof(half2)) + k*16, KV + i*stride_KV + k*h2_per_chunk);
                }
            }
        };
        ggml_cuda_unroll<5>{}(load);
    } else {
        static_assert(nbatch_fa % (4*nwarps) == 0, "out of bounds");
        auto load = [&] __device__ (const int n) {
            const int stride_k = WARP_SIZE >> n;
            const int k0_start = stride_k == WARP_SIZE ? 0 : D2 - D2 % (2*stride_k);
            const int k0_stop  =                             D2 - D2 % (1*stride_k);
            const int stride_i = WARP_SIZE / stride_k;

            if (k0_start == k0_stop) {
                return;
            }

#pragma unroll
            for (int i0 = 0; i0 < nbatch_fa; i0 += nwarps*stride_i) {
                const int i = i0 + threadIdx.y*stride_i + (stride_k == WARP_SIZE ? 0 : threadIdx.x / stride_k);

                if (i0 + nwarps*stride_i > nbatch_fa && i >= nbatch_fa) {
                    break;
                }

#pragma unroll
                for (int k0 = k0_start; k0 < k0_stop; k0 += stride_k) {
                    const int k = k0 + (stride_k == WARP_SIZE ? threadIdx.x : threadIdx.x % stride_k);

                    tile_KV[i*stride_tile + k] = KV[i*stride_KV + k];
                }
            }
        };
        ggml_cuda_unroll<3>{}(load);
    }
}

template<int ncols1, int nwarps, int nbatch_fa, bool use_cp_async>
static __device__ __forceinline__ void flash_attn_ext_f16_load_mask(
        const half2 * const __restrict__ mask_h2, half2 * const __restrict__ tile_mask, const int stride_mask) {
    static_assert(nbatch_fa == 2*WARP_SIZE || WARP_SIZE % nbatch_fa == 0, "bad KQ_per_iter");

    if constexpr (use_cp_async) {
        constexpr int preload = nbatch_fa >= 32 ? nbatch_fa * sizeof(half) : 64;
        constexpr int cols_per_warp = 8*WARP_SIZE/nbatch_fa;
        constexpr int stride_j = nwarps * cols_per_warp;

        const unsigned int tile_mask_32 = ggml_cuda_cvta_generic_to_shared(tile_mask);

#pragma unroll
        for (int j0 = 0; j0 < ncols1; j0 += stride_j) {
            const int j = j0 + threadIdx.y*cols_per_warp +
                (nbatch_fa == 2*WARP_SIZE ? threadIdx.x / (WARP_SIZE/4) : threadIdx.x / (WARP_SIZE/cols_per_warp));

            if (j0 + stride_j > ncols1 && j >= ncols1) {
                break;
            }

            const int i = 4 * (threadIdx.x % (nbatch_fa/8));

            cp_async_cg_16<preload>(tile_mask_32 + j*(nbatch_fa*sizeof(half) + 16) + i*sizeof(half2), mask_h2 + j*stride_mask + i);
        }
        return;
    }

    constexpr int cols_per_warp = 2*WARP_SIZE/nbatch_fa;
    constexpr int stride_j = nwarps * cols_per_warp;
#pragma unroll
    for (int j0 = 0; j0 < ncols1; j0 += stride_j) {
        const int j = j0 + threadIdx.y*cols_per_warp + (nbatch_fa == 2*WARP_SIZE ? 0 : threadIdx.x / (WARP_SIZE/cols_per_warp));

        if (j0 + stride_j > ncols1 && j >= ncols1) {
            break;
        }

        const int i = nbatch_fa == 2*WARP_SIZE ? threadIdx.x : threadIdx.x % (WARP_SIZE/cols_per_warp);

        tile_mask[j*(nbatch_fa/2 + 4) + i] = mask_h2[j*stride_mask + i];
    }
}

template<int DKQ, int DV, int ncols1, int ncols2, int nwarps, int ntiles, bool use_logit_softcap, bool mla, bool needs_fixup, bool is_fixup, bool last_iter>
static __device__ __forceinline__ void flash_attn_ext_f16_iter(
        const float2 * const __restrict__ Q_f2,
        const half2  * const __restrict__ K_h2,
        const half2  * const __restrict__ V_h2,
        const half2  * const __restrict__ mask_h2,
        float2       * const __restrict__ dstk,
        float2       * const __restrict__ dstk_fixup,
        const float scale,
        const float slope,
        const float logit_softcap,
        const int ne01,
        const int ne02,
        const int stride_K,
        const int stride_V,
        const int stride_mask,
        const int jt,
        half2        * const __restrict__ tile_Q,
        half2        * const __restrict__ tile_K,
        half2        * const __restrict__ tile_V,
        half2        * const __restrict__ tile_mask,
        const tile_B * const __restrict__ Q_B,
        tile_C_VKQ   * const __restrict__ VKQ_C,
        float        * const __restrict__ KQ_max,
        float        * const __restrict__ KQ_rowsum,
        const int kb0) {
#ifdef INT8_MMA_AVAILABLE
    typedef fattn_mma_f16_config<DKQ, DV> c;

#ifdef CP_ASYNC_AVAILABLE
    constexpr int nstages = c::nstages_target;
#else
    constexpr int nstages = 0;
#endif // CP_ASYNC_AVAILABLE

    constexpr int cols_per_warp   = ntiles * tile_B::I;
    constexpr int cols_per_thread = ntiles == 1 ? 2 : ntiles;
    constexpr int np              = nwarps * (cols_per_warp/ncols2) / ncols1; // Number of parallel CUDA warps per Q column.
    constexpr int ncols           = ncols1 * ncols2;
    constexpr int nbatch_K2       = c::get_nbatch_K2_device(ncols);
    constexpr int nbatch_V2       = c::get_nbatch_V2_device(ncols);

    constexpr int stride_tile_Q = DKQ/2     + 4;
    constexpr int stride_tile_K = nbatch_K2 + 4;

    static_assert(!mla || nbatch_K2 >= nbatch_V2, "bad nbatch_K2, nbatch_V2 for MLA");
    constexpr int stride_tile_V = mla ? stride_tile_K : nbatch_V2 + 4;

    const int k_VKQ_0 = kb0 * c::nbatch_fa;
    tile_C_KQ KQ_C[c::nbatch_fa/(np*tile_C_KQ::I) * ntiles];

    // Use wide variants of tiles if ntiles >= 2.
    tile_B_16     * Q_B_16   = (tile_B_16     *) Q_B;
    tile_C_VKQ_16 * VKQ_C_16 = (tile_C_VKQ_16 *) VKQ_C;
    tile_C_KQ_16  * KQ_C_16  = (tile_C_KQ_16  *) KQ_C;

    if constexpr (nstages > 1) {
        static_assert(!mla, "multi-stage loading not implemented for MLA");
        static_assert(nbatch_K2 == DKQ/2, "batching not implemented for multi stage loading");
        constexpr bool use_cp_async = true;
        cp_async_wait_all();
        __syncthreads();
        flash_attn_ext_f16_load_tile<stride_tile_V, nwarps, c::nbatch_fa, use_cp_async>
            (V_h2 + k_VKQ_0*stride_V, tile_V, nbatch_V2, stride_V);
    } else {
        constexpr bool use_cp_async = nstages == 1;
        if constexpr (ncols2 > 1 || mask_h2) {
            flash_attn_ext_f16_load_mask<ncols1, nwarps, c::nbatch_fa, use_cp_async>(mask_h2 + k_VKQ_0/2, tile_mask, stride_mask);
        }
    }

#pragma unroll
    for (int k0_start = 0; k0_start < DKQ/2; k0_start += nbatch_K2) {
        const int k0_stop = k0_start + nbatch_K2 < DKQ/2 ? k0_start + nbatch_K2 : DKQ/2;
        const int k0_diff = k0_stop - k0_start;

        if constexpr (nstages <= 1) {
            constexpr bool use_cp_async = nstages == 1;
            flash_attn_ext_f16_load_tile<stride_tile_K, nwarps, c::nbatch_fa, use_cp_async>
                (K_h2 + k_VKQ_0*stride_K + k0_start, tile_K, k0_diff, stride_K);
            if constexpr (use_cp_async) {
                cp_async_wait_all();
            }
            __syncthreads();
        }

        // Calculate tile of KQ:
        if constexpr (c::Q_in_reg) {
#pragma unroll
            for (int i_KQ_00 = 0; i_KQ_00 < c::nbatch_fa; i_KQ_00 += np*tile_A::I) {
                const int i_KQ_0 = i_KQ_00 + (threadIdx.y % np)*tile_A::I;
#pragma unroll
                for (int k_KQ_0 = k0_start; k_KQ_0 < k0_stop; k_KQ_0 += tile_A::J) {
                    tile_A K_A;
                    load_ldmatrix(K_A, tile_K + i_KQ_0*stride_tile_K + (k_KQ_0 - k0_start), stride_tile_K);
                    if constexpr (ntiles == 1) {
                        mma(KQ_C[i_KQ_00/(np*tile_A::I)], K_A, Q_B[k_KQ_0/tile_A::J]);
                    } else {
#pragma unroll
                        for (int t = 0; t < ntiles/2; ++t) {
                            // Wide version of KQ_C is column-major => swap A and B.
                            mma(KQ_C_16[i_KQ_00/(np*tile_A::I) * ntiles/2 + t], Q_B_16[k_KQ_0/tile_A::J * ntiles/2 + t], K_A);
                        }
                    }
                }
            }
        } else {
            static_assert(ntiles == 2, "ntiles != 2 not implemented");
#pragma unroll
            for (int k_KQ_0 = k0_start; k_KQ_0 < k0_stop; k_KQ_0 += tile_A::J) {
                load_ldmatrix(Q_B_16[0], tile_Q + (threadIdx.y / np)*(tile_B_16::I*stride_tile_Q) + k_KQ_0, stride_tile_Q);

#pragma unroll
                for (int i_KQ_00 = 0; i_KQ_00 < c::nbatch_fa; i_KQ_00 += np*tile_A::I) {
                    const int i_KQ_0 = i_KQ_00 + (threadIdx.y % np)*tile_A::I;

                    tile_A K_A;
                    load_ldmatrix(K_A, tile_K + i_KQ_0*stride_tile_K + (k_KQ_0 - k0_start), stride_tile_K);

                    // Wide version of KQ_C is column-major => swap A and B.
                    mma(KQ_C_16[i_KQ_00/(np*tile_A::I)], Q_B_16[0], K_A);
                }
            }
        }

        if constexpr (nstages <= 1) {
            __syncthreads(); // Only needed if tile_K == tile_V.
        }
    }

    if constexpr (use_logit_softcap) {
        static_assert(c::nbatch_fa % (np*tile_C_KQ::I) == 0, "bad loop size");
#pragma unroll
        for (int i = 0; i < c::nbatch_fa/(np*tile_C_KQ::I) * ntiles; ++i) {
#pragma unroll
            for (int l = 0; l < tile_C_KQ::ne; ++l) {
                KQ_C[i].x[l] = logit_softcap*tanhf(KQ_C[i].x[l]);
            }
        }
    }

    float KQ_max_new[cols_per_thread];
#pragma unroll
    for (int col = 0; col < cols_per_thread; ++col) {
        KQ_max_new[col] = KQ_max[col];
    }
    float KQ_rowsum_add[cols_per_thread] = {0.0f};

    if constexpr (ntiles == 1) {
        if constexpr (ncols2 > 1 || mask_h2) {
#pragma unroll
            for (int i00 = 0; i00 < c::nbatch_fa; i00 += np*tile_C_KQ::I) {
                const int i0 = i00 + (threadIdx.y % np)*tile_C_KQ::I;
#pragma unroll
                for (int l = 0; l < tile_C_KQ::ne; ++l) {
                    const int i = i0 + tile_C_KQ::get_i(l);
                    const int j = ((threadIdx.y / np)*tile_C_KQ::J + tile_C_KQ::get_j(l)) / ncols2;

                    KQ_C[i00/(np*tile_C_KQ::I)].x[l] += slope *
                        __half2float(((const half *) tile_mask)[j*(c::nbatch_fa + 8) + i]);
                }
            }
        }

        // Calculate softmax for each KQ column using the current max. value.
        // The divisor is stored in KQ_rowsum and will be applied at the end.
        static_assert(c::nbatch_fa % (np*tile_C_KQ::I) == 0, "bad loop size");
#pragma unroll
        for (int k = 0; k < c::nbatch_fa/(np*tile_C_KQ::I); ++k) {
#pragma unroll
            for (int l = 0; l < tile_C_KQ::ne; ++l) {
                KQ_max_new[l % 2] = fmaxf(KQ_max_new[l % 2], KQ_C[k].x[l]);
            }
        }

        // Values per KQ column are spread across 8 threads, does not need full warp reduce:
#pragma unroll
        for (int col = 0; col < cols_per_thread; ++col) {
#pragma unroll
            for (int offset = 16; offset >= 4; offset >>= 1) {
                KQ_max_new[col] = fmaxf(KQ_max_new[col], __shfl_xor_sync(0xFFFFFFFF, KQ_max_new[col], offset, WARP_SIZE));
            }
        }

        static_assert(c::nbatch_fa % (np*tile_C_KQ::I) == 0, "bad loop size");
#pragma unroll
        for (int k = 0; k < c::nbatch_fa/(np*tile_C_KQ::I); ++k) {
#pragma unroll
            for (int l = 0; l < tile_C_KQ::ne; ++l) {
                KQ_C[k].x[l] = expf(KQ_C[k].x[l] - KQ_max_new[l % 2]);

                KQ_rowsum_add[l % 2] += KQ_C[k].x[l];
            }
        }
    } else { // ntiles > 1
        if (ncols2 > 1 || mask_h2) {
#pragma unroll
            for (int i00 = 0; i00 < c::nbatch_fa; i00 += np*tile_C_KQ_16::J) {
                const int i0 = i00 + (threadIdx.y % np)*tile_C_KQ_16::J;
#pragma unroll
                for (int t = 0; t < ntiles/2; ++t) {
#pragma unroll
                    for (int l0 = 0; l0 < tile_C_KQ_16::ne; l0 += 2) {
                        const int i = (i0 + tile_C_KQ_16::get_j(l0)) / 2;
                        const int j = ((threadIdx.y / np)*cols_per_warp + t*tile_C_KQ_16::I + tile_C_KQ_16::get_i(l0)) / ncols2;

                        const float2 tmp = __half22float2(tile_mask[j*(c::nbatch_fa/2 + 4) + i]);
                        const int KQ_index = i00/(np*tile_C_KQ_16::J) * ntiles/2 + t;
                        KQ_C_16[KQ_index].x[l0 + 0] += slope*tmp.x;
                        KQ_C_16[KQ_index].x[l0 + 1] += slope*tmp.y;
                    }
                }
            }
        }

        // Calculate softmax for each KQ column using the current max. value.
        // The divisor is stored in KQ_rowsum and will be applied at the end.
        static_assert(c::nbatch_fa % (np*tile_C_KQ::I) == 0, "bad loop size");
#pragma unroll
        for (int k = 0; k < c::nbatch_fa/(np*tile_C_KQ_16::J); ++k) {
#pragma unroll
            for (int t = 0; t < ntiles/2; ++t) {
#pragma unroll
                for (int l = 0; l < tile_C_KQ_16::ne; ++l) {
                    const int KQ_index = 2*t + (l/2) % 2;
                    KQ_max_new[KQ_index] = fmaxf(KQ_max_new[KQ_index], KQ_C_16[k*ntiles/2 + t].x[l]);
                }
            }
        }

        // Values per KQ column are spread across 4 threads, does not need full warp reduce:
#pragma unroll
        for (int col = 0; col < cols_per_thread; ++col) {
#pragma unroll
            for (int offset = 2; offset >= 1; offset >>= 1) {
                KQ_max_new[col] = fmaxf(KQ_max_new[col], __shfl_xor_sync(0xFFFFFFFF, KQ_max_new[col], offset, WARP_SIZE));
            }
        }

        static_assert(c::nbatch_fa % (np*tile_C_KQ_16::J) == 0, "bad loop size");
#pragma unroll
        for (int k = 0; k < c::nbatch_fa/(np*tile_C_KQ_16::J); ++k) {
#pragma unroll
            for (int t = 0; t < ntiles/2; ++t) {
#pragma unroll
                for (int l = 0; l < tile_C_KQ_16::ne; ++l) {
                    const int KQ_index = 2*t + (l/2) % 2;

                    KQ_C_16[k*ntiles/2 + t].x[l] = expf(KQ_C_16[k*ntiles/2 + t].x[l] - KQ_max_new[KQ_index]);

                    KQ_rowsum_add[KQ_index] += KQ_C_16[k*ntiles/2 + t].x[l];
                }
            }
        }
    }

    {
        float KQ_max_scale[cols_per_thread];
#pragma unroll
        for (int col = 0; col < cols_per_thread; ++col) {
            KQ_max_scale[col] = expf(KQ_max[col] - KQ_max_new[col]);
            KQ_max[col] = KQ_max_new[col];

            // Scale previous KQ_rowsum to account for a potential increase in KQ_max:
            KQ_rowsum[col] = KQ_max_scale[col]*KQ_rowsum[col] + KQ_rowsum_add[col];
        }

        if constexpr (ntiles == 1) {
            const half2 KQ_max_scale_h2 = make_half2(KQ_max_scale[0], KQ_max_scale[1]);
#pragma unroll
            for (int i = 0; i < DV/tile_C_VKQ::I; ++i) {
#pragma unroll
                for (int l = 0; l < tile_C_VKQ::ne; ++l) {
                    VKQ_C[i].x[l] *= KQ_max_scale_h2;
                }
            }
        } else {
#pragma unroll
            for (int col = 0; col < cols_per_thread; ++col) {
                const half2 KQ_max_scale_h2 = make_half2(KQ_max_scale[col], KQ_max_scale[col]);
#pragma unroll
                for (int i = 0; i < DV/tile_C_VKQ_16::J; ++i) {
#pragma unroll
                    for (int l0 = 0; l0 < tile_C_VKQ_16::ne; l0 += 2) {
                        VKQ_C_16[i*ntiles/2 + col/2].x[l0 + col % 2] *= KQ_max_scale_h2;
                    }
                }
            }
        }
    }

    // Convert KQ C tiles into B tiles for VKQ calculation:
    tile_B B[c::nbatch_fa/(np*2*tile_B::J) * ntiles];
    tile_B_16 * B_16 = (tile_B_16 *) B;
    static_assert(c::nbatch_fa % (np*2*tile_B::J) == 0, "bad loop size");
    if constexpr (ntiles == 1) {
#pragma unroll
        for (int k = 0; k < c::nbatch_fa/(np*2*tile_B::J); ++k) {
            B[k] = get_transposed(get_half2(KQ_C[k]));
        }
    } else {
        for (int k = 0; k < c::nbatch_fa/(np*2*tile_B_16::J); ++k) {
#pragma unroll
            for (int t = 0; t < ntiles/2; ++t) {
                B_16[k*ntiles/2 + t] = get_half2(KQ_C_16[k*ntiles/2 + t]);
            }
        }
    }

    if constexpr (nstages > 1) {
        // Preload K tile for next iteration:
        constexpr bool use_cp_async = true;
        cp_async_wait_all();
        __syncthreads();
        if (!last_iter) {
            if (ncols2 > 1 || mask_h2) {
                flash_attn_ext_f16_load_mask<ncols1, nwarps, c::nbatch_fa, use_cp_async>
                    (mask_h2 + (k_VKQ_0 + c::nbatch_fa)/2, tile_mask, stride_mask);
            }
            flash_attn_ext_f16_load_tile<stride_tile_K, nwarps, c::nbatch_fa, use_cp_async>
                (K_h2 + (k_VKQ_0 + c::nbatch_fa)*stride_K, tile_K, nbatch_K2, stride_K);
        }
    }


    // For MLA K and V have the same data.
    // Therefore, iterate over V in reverse and re-use the data if possible.
    static_assert(!mla || nstages <= 1, "combination of MLA and multi-stage loading not implemented");
    constexpr int reusable_cutoff = mla ? (DKQ - 1) - (DKQ - 1) % (2*nbatch_K2) - (DKQ - DV) : DV;
#pragma unroll
    for (int i0_stop = DV; i0_stop > 0; i0_stop -= 2*nbatch_V2) {
        const int i0_start = i0_stop - 2*nbatch_V2 > 0 ? i0_stop - 2*nbatch_V2 : 0;
        const int i0_diff  = i0_stop - i0_start;

        if (nstages <= 1 && i0_start < reusable_cutoff) {
            constexpr bool use_cp_async = nstages == 1;
            flash_attn_ext_f16_load_tile<stride_tile_V, nwarps, c::nbatch_fa, use_cp_async>
                (V_h2 + k_VKQ_0*stride_V + i0_start/2, tile_V, i0_diff/2, stride_V);
            if constexpr (use_cp_async) {
                cp_async_wait_all();
            }
            __syncthreads();
        }
        const half2 * tile_V_i = i0_start < reusable_cutoff ? tile_V : tile_V + (i0_start - reusable_cutoff)/2;

        // Calculate VKQ tile:
#pragma unroll
        for (int i_VKQ_0 = i0_start; i_VKQ_0 < i0_stop; i_VKQ_0 += tile_C_VKQ::I) {
            static_assert((c::nbatch_fa/2) % (np*tile_A::J) == 0, "bad loop size");
#pragma unroll
            for (int k00 = 0; k00 < c::nbatch_fa/2; k00 += np*tile_A::J) {
                const int k0 = k00 + (threadIdx.y % np)*tile_A::J;

                tile_A A;
                load_ldmatrix_trans(A, tile_V_i + 2*k0*stride_tile_V + (i_VKQ_0 - i0_start)/2, stride_tile_V);
                if constexpr (ntiles == 1) {
                    mma(VKQ_C[i_VKQ_0/tile_C_VKQ::I], A, B[k00/(np*tile_A::J)]);
                } else {
#pragma unroll
                    for (int t = 0; t < ntiles/2; ++t) {
                        // Wide version of VKQ_C is column-major => swap A and B.
                        mma(VKQ_C_16[i_VKQ_0/tile_C_VKQ::I * ntiles/2 + t], B_16[k00/(np*tile_A::J) * ntiles/2 + t], A);
                    }
                }
            }
        }

        if constexpr (nstages <= 1) {
            __syncthreads(); // Only needed if tile_K == tile_V.
        }
    }
#else
    GGML_UNUSED(Q_f2); GGML_UNUSED(K_h2); GGML_UNUSED(V_h2);
    GGML_UNUSED(mask_h2); GGML_UNUSED(dstk); GGML_UNUSED(dstk_fixup);
    GGML_UNUSED(scale); GGML_UNUSED(slope); GGML_UNUSED(logit_softcap);
    GGML_UNUSED(ne01); GGML_UNUSED(ne02); GGML_UNUSED(stride_K); GGML_UNUSED(stride_V);
    GGML_UNUSED(stride_mask); GGML_UNUSED(jt); GGML_UNUSED(tile_K);
    GGML_UNUSED(stride_mask); GGML_UNUSED(jt); GGML_UNUSED(tile_K);
    GGML_UNUSED(tile_V); GGML_UNUSED(tile_mask); GGML_UNUSED(Q_B);
    GGML_UNUSED(VKQ_C); GGML_UNUSED(KQ_max); GGML_UNUSED(KQ_rowsum);
    GGML_UNUSED(kb0);
    NO_DEVICE_CODE;
#endif // INT8_MMA_AVAILABLE
}

template<int DKQ, int DV, int ncols1, int ncols2, int nwarps, int ntiles, bool use_logit_softcap, bool mla, bool needs_fixup, bool is_fixup>
static __device__ __forceinline__ void flash_attn_ext_f16_process_tile(
        const float2 * const __restrict__ Q_f2,
        const half2  * const __restrict__ K_h2,
        const half2  * const __restrict__ V_h2,
        const half2  * const __restrict__ mask_h2,
        float2       * const __restrict__ dstk,
        float2       * const __restrict__ dstk_fixup,
        const float scale,
        const float slope,
        const float logit_softcap,
        const int ne01,
        const int ne02,
        const int stride_Q1,
        const int stride_Q2,
        const int stride_K,
        const int stride_V,
        const int stride_mask,
        const int jt,
        const int kb0_start,
        const int kb0_stop) {
#ifdef INT8_MMA_AVAILABLE
    //In this kernel Q, K, V are matrices while i, j, k are matrix indices.

    typedef fattn_mma_f16_config<DKQ, DV> c;

#ifdef CP_ASYNC_AVAILABLE
    constexpr int nstages = c::nstages_target;
#else
    constexpr int nstages = 0;
#endif // CP_ASYNC_AVAILABLE

    constexpr int ncols           = ncols1 * ncols2;
    constexpr int cols_per_warp   = ntiles * tile_B::I;
    constexpr int cols_per_thread = ntiles == 1 ? 2 : ntiles;
    constexpr int np              = nwarps * (cols_per_warp/ncols2) / ncols1; // Number of parallel CUDA warps per Q column.
    constexpr int nbatch_K2       = c::get_nbatch_K2_device(ncols);
    constexpr int nbatch_V2       = c::get_nbatch_V2_device(ncols);

    static_assert(nwarps * (cols_per_warp/ncols2) % ncols1 == 0, "bad nwarps");

    constexpr int stride_tile_Q = DKQ/2     + 4;
    constexpr int stride_tile_K = nbatch_K2 + 4;

    static_assert(!mla || nbatch_K2 >= nbatch_V2, "bad nbatch_K2, nbatch_V2 for MLA");
    constexpr int stride_tile_V = mla ? stride_tile_K : nbatch_V2 + 4;
    constexpr int stride_tile_KV_max = stride_tile_K > stride_tile_V ? stride_tile_K : stride_tile_V;

    extern __shared__ half2 tile_Q[];
    half2 * tile_K    = c::Q_in_reg ? tile_Q                                : tile_Q + ncols        * stride_tile_Q;
    half2 * tile_V    = nstages > 1 ? tile_K + c::nbatch_fa * stride_tile_K : tile_K;
    half2 * tile_mask = nstages > 1 ? tile_V + c::nbatch_fa * stride_tile_V : tile_V + c::nbatch_fa * stride_tile_KV_max;

    tile_B       Q_B[(c::Q_in_reg ? DKQ/(2*tile_B::J) : 1) * ntiles];
    tile_C_VKQ VKQ_C[DV/tile_C_VKQ::I  * ntiles];

    tile_B_16     * Q_B_16   = (tile_B_16     *) Q_B;
    tile_C_VKQ_16 * VKQ_C_16 = (tile_C_VKQ_16 *) VKQ_C;

    float KQ_rowsum[cols_per_thread] = {0.0f};
    float KQ_max[cols_per_thread];
#pragma unroll
    for (int col = 0; col < cols_per_thread; ++col) {
        KQ_max[col] = -FLT_MAX/2.0f;
    }

    // Load Q data into tile_Q, either temporarily or permanently.
    // Q in registers is faster, but register pressure is the biggest bottleneck.
    // The loading is done with decreasing granularity for D for better memory bandwidth.
    const half2 scale_h2 = make_half2(scale, scale);
#pragma unroll
    for (int stride_k : {WARP_SIZE, WARP_SIZE/2, WARP_SIZE/4}) {
        const int k0_start  = stride_k == WARP_SIZE ? 0 : DKQ/2 - (DKQ/2) % (2*stride_k);
        const int k0_stop   =                             DKQ/2 - (DKQ/2) % (1*stride_k);
        const int stride_jc = WARP_SIZE / stride_k;

        if (k0_start == k0_stop) {
            continue;
        }

#pragma unroll
        for (int jc0 = 0; jc0 < ncols; jc0 += nwarps*stride_jc) {
            const int jc = jc0 + threadIdx.y*stride_jc + (stride_k == WARP_SIZE ? 0 : threadIdx.x / stride_k);

            if (jc0 + nwarps*stride_jc > ncols && jc >= ncols) {
                break;
            }

            const int j = jc / ncols2;
            const int c = jc % ncols2;

            if (jt*ncols1 + j < ne01) {
#pragma unroll
                for (int k0 = k0_start; k0 < k0_stop; k0 += stride_k) {
                    const int k = k0 + (stride_k == WARP_SIZE ? threadIdx.x : threadIdx.x % stride_k);

                    const float2 tmp = Q_f2[(jt*ncols1 + j)*stride_Q1 + c*stride_Q2 + k];
                    tile_Q[jc*stride_tile_Q + k] = scale_h2 * make_half2(tmp.x, tmp.y);
                }
            } else {
#pragma unroll
                for (int k0 = k0_start; k0 < k0_stop; k0 += stride_k) {
                    const int k = k0 + (stride_k == WARP_SIZE ? threadIdx.x : threadIdx.x % stride_k);

                    tile_Q[jc*stride_tile_Q + k] = make_half2(0.0f, 0.0f);
                }
            }
        }
    }

    __syncthreads();

    if constexpr (c::Q_in_reg) {
        const int j0 = (threadIdx.y / np) * cols_per_warp;

#pragma unroll
        for (int k0 = 0; k0 < DKQ/2; k0 += tile_B::J) {
            if constexpr (ntiles == 1) {
                load_ldmatrix(Q_B[k0/tile_B::J], tile_Q + j0*stride_tile_Q + k0, stride_tile_Q);
            } else {
#pragma unroll
                for (int t = 0; t < ntiles/2; ++t) {
                    load_ldmatrix(Q_B_16[k0/tile_B_16::J * ntiles/2 + t],
                        tile_Q + (j0 + t*tile_B_16::I)*stride_tile_Q + k0, stride_tile_Q);
                }
            }
        }
    }

    __syncthreads();

    // Preload mask and K data for first iteration when using cp_async with multiple stages:
    if constexpr (nstages > 1) {
        static_assert(nbatch_K2 == DKQ/2, "batching not implemented for multi-stage pipeline");
        constexpr bool use_cp_async = true;
        if (ncols2 > 1 || mask_h2) {
            flash_attn_ext_f16_load_mask<ncols1, nwarps, c::nbatch_fa, use_cp_async>
                (mask_h2 + kb0_start*c::nbatch_fa/2, tile_mask, stride_mask);
        }
        flash_attn_ext_f16_load_tile<stride_tile_K, nwarps, c::nbatch_fa, use_cp_async>
            (K_h2 + kb0_start*c::nbatch_fa*stride_K, tile_K, nbatch_K2, stride_K);
    }

    // Iterate over ne11 == previous tokens:
    for (int kb0 = kb0_start; kb0 < kb0_stop-1; ++kb0) {
        constexpr bool last_iter = false;
        flash_attn_ext_f16_iter<DKQ, DV, ncols1, ncols2, nwarps, ntiles, use_logit_softcap, mla, needs_fixup, is_fixup, last_iter>
            (Q_f2, K_h2, V_h2, mask_h2, dstk, dstk_fixup, scale, slope, logit_softcap,
             ne01, ne02, stride_K, stride_V, stride_mask, jt, tile_Q, tile_K, tile_V, tile_mask, Q_B, VKQ_C, KQ_max, KQ_rowsum, kb0);
    }
    { // kb0_start is always < kb0_stop so the last iter can be executed unconditionally.
        constexpr bool last_iter = true;
        flash_attn_ext_f16_iter<DKQ, DV, ncols1, ncols2, nwarps, ntiles, use_logit_softcap, mla, needs_fixup, is_fixup, last_iter>
            (Q_f2, K_h2, V_h2, mask_h2, dstk, dstk_fixup, scale, slope, logit_softcap,
             ne01, ne02, stride_K, stride_V, stride_mask, jt, tile_Q, tile_K, tile_V, tile_mask, Q_B, VKQ_C, KQ_max, KQ_rowsum, kb0_stop-1);
    }

    // With multi-stage loading there is no __syncthreads at the end of the iter,
    //     there can be a race condition on shared memory access for combining/writing back results.
    if constexpr (nstages > 1 && nwarps*cols_per_warp > c::nbatch_fa) {
        __syncthreads();
    }

    // Finally, sum up partial KQ rowsums.
    // The partial sums are spread across 8/4 threads each, does not need full reduce.
    {
        constexpr int offset_first = ntiles == 1 ? 16 : 2;
        constexpr int offset_last  = ntiles == 1 ?  4 : 1;
#pragma unroll
        for (int col = 0; col < cols_per_thread; ++col) {
#pragma unroll
            for (int offset = offset_first; offset >= offset_last; offset >>= 1) {
                KQ_rowsum[col] += __shfl_xor_sync(0xFFFFFFFF, KQ_rowsum[col], offset, WARP_SIZE);
            }
        }
    }

    // Combine VKQ accumulator values if np > 1.
    // It's also faster to do small writes to shared memory, then large write to VRAM than to do small writes to VRAM.
    // So also write VKQ accumulators to shared memory in column-major format if np == 1.

    constexpr int nbatch_combine = c::get_nbatch_combine_device(ncols);
    constexpr int tile_stride    = nbatch_combine + 4;
    static_assert((DV/2) % nbatch_combine == 0, "bad nbatch_combine");

    if constexpr (ntiles == 1) {
        const int jc_cwmo = (threadIdx.x % (2*tile_C_VKQ::J)) / tile_C_VKQ::J; // jc combine write meta offset
        const int jc_cwm = threadIdx.y*(2*tile_C_VKQ::J) + 2*tile_C_VKQ::get_j(-1) + jc_cwmo; // jc combine write meta
        const float2 KQ_cmr = make_float2(KQ_max[jc_cwmo], KQ_rowsum[jc_cwmo]); // KQ combine max rowsum

        if (((!needs_fixup && !is_fixup) || np > 1) && threadIdx.x < 2*tile_C_VKQ::J) {
            // Use the 16 bytes of padding in each row to store the meta data: KQ max, KQ rowsum, KQ max scale.
            ((float2 *) tile_Q)[jc_cwm*(tile_stride/2) + nbatch_combine/2] = KQ_cmr;
        }

        __syncthreads();

        if (np == 1) {
            // No combination is needed, the meta data can be directly written from registers to VRAM.
            if (needs_fixup && threadIdx.x < tile_B::I) {
                float2 * dstk_fixup_meta = dstk_fixup + blockIdx.x*ncols;
                dstk_fixup_meta[jc_cwm] = KQ_cmr;
            }
            if (is_fixup && threadIdx.x < tile_B::I) {
                float2 * dstk_fixup_meta = dstk_fixup + (gridDim.x + blockIdx.x)*ncols;
                dstk_fixup_meta[jc_cwm] = KQ_cmr;
            }
        }
    } else {
        static_assert(ntiles == 2 || ntiles == 4, "bad ntiles");
        const int jc_cwm = threadIdx.y*cols_per_warp // jc combine write meta
            + (ntiles == 4 ? ((threadIdx.x % 4) / 2) * tile_C_VKQ_16::I : 0)
            + tile_C_VKQ_16::get_i(threadIdx.x % 4);
        const float2 KQ_cmr = make_float2(KQ_max[threadIdx.x % cols_per_thread], KQ_rowsum[threadIdx.x % cols_per_thread]); // KQ combine max rowsum

        if (((!needs_fixup && !is_fixup) || np > 1) && (ntiles == 4 || threadIdx.x % 4 < cols_per_thread)) {
            // Use the 16 bytes of padding in each row to store the meta data: KQ max, KQ rowsum, KQ max scale.
            ((float2 *) tile_Q)[jc_cwm*(tile_stride/2) + nbatch_combine/2] = KQ_cmr;
        }

        __syncthreads();

        if (np == 1) {
            // No combination is needed, the meta data can be directly written from registers to VRAM.
            if (needs_fixup && (ntiles == 4 || threadIdx.x % 4 < ntiles)) {
                float2 * dstk_fixup_meta = dstk_fixup + blockIdx.x*ncols;
                dstk_fixup_meta[jc_cwm] = KQ_cmr;
            }
            if (is_fixup && (ntiles == 4 || threadIdx.x % 4 < ntiles)) {
                float2 * dstk_fixup_meta = dstk_fixup + (gridDim.x + blockIdx.x)*ncols;
                dstk_fixup_meta[jc_cwm] = KQ_cmr;
            }
        }
    }

    static_assert(np == 1 || ntiles == 1 || ntiles == 2, "bad ntiles");
    if (np > 1 && threadIdx.y % np == 0) {
        // Combine the meta data for parallel warps via shared memory.
        // Warps with threadIdx.y % np != 0 must NOT return early.
        // All threads must return simultaneously to avoid race conditions with work on the next tile.

        constexpr int nmeta = np*cols_per_warp >= WARP_SIZE ? np*cols_per_warp/WARP_SIZE : 1;

        const int jc_meta = threadIdx.y*cols_per_warp + (np*cols_per_warp < WARP_SIZE ? threadIdx.x % (np*cols_per_warp) : threadIdx.x);
        float2 * const meta_ptr = ((float2 *) tile_Q) + jc_meta*(tile_stride/2) + nbatch_combine/2;
        float2 meta[nmeta];
#pragma unroll
        for (int imeta = 0; imeta < nmeta; ++imeta) {
            meta[imeta] = meta_ptr[imeta * WARP_SIZE * tile_stride/2];
        }

        float KQ_cmn = meta[0].x; // KQ combine max new, max between all parallel warps.
#pragma unroll
        for (int imeta = 1; imeta < nmeta; ++imeta) {
            KQ_cmn = fmaxf(KQ_cmn, meta[imeta].x);
        }
#pragma unroll
        for (int offset = np*cols_per_warp/2; offset >= cols_per_warp; offset >>= 1) {
            if (offset < WARP_SIZE) {
                KQ_cmn = fmaxf(KQ_cmn, __shfl_xor_sync(0xFFFFFFFF, KQ_cmn, offset, WARP_SIZE));
            }
        }

        float KQ_cms[nmeta]; // KQ combine max scale per warp.
#pragma unroll
        for (int imeta = 0; imeta < nmeta; ++imeta) {
            KQ_cms[imeta] = expf(meta[imeta].x - KQ_cmn);
        }

        float KQ_crs = KQ_cms[0]*meta[0].y; // KQ combine rowsum, scaled sum of all parallel warps.
#pragma unroll
        for (int imeta = 1; imeta < nmeta; ++imeta) {
            KQ_crs += KQ_cms[imeta]*meta[imeta].y;
        }
#pragma unroll
        for (int offset = np*cols_per_warp/2; offset >= cols_per_warp; offset >>= 1) {
            if (offset < WARP_SIZE) {
                KQ_crs += __shfl_xor_sync(0xFFFFFFFF, KQ_crs, offset, WARP_SIZE);
            }
        }

        // do we really need this?
        __syncthreads();

        // Write back combined meta data:
#pragma unroll
        for (int imeta = 0; imeta < nmeta; ++imeta) {
            if (np*cols_per_warp >= WARP_SIZE || threadIdx.x < np*cols_per_warp) {
                // Combined KQ max scale + rowsum.
                meta_ptr[imeta * WARP_SIZE * tile_stride/2] = make_float2(KQ_cms[imeta], KQ_crs);
            }
        }

        // Combined KQ max + rowsum.
        static_assert(cols_per_warp <= WARP_SIZE);
        if (needs_fixup && (cols_per_warp == WARP_SIZE || threadIdx.x < cols_per_warp)) {
            float2 * dstk_fixup_meta = dstk_fixup + blockIdx.x*ncols;
            dstk_fixup_meta[(threadIdx.y/np)*cols_per_warp + threadIdx.x] = make_float2(KQ_cmn, KQ_crs);
        }
        if (is_fixup && (cols_per_warp == WARP_SIZE || threadIdx.x < cols_per_warp)) {
            float2 * dstk_fixup_meta = dstk_fixup + (gridDim.x + blockIdx.x)*ncols;
            dstk_fixup_meta[(threadIdx.y/np)*cols_per_warp + threadIdx.x] = make_float2(KQ_cmn, KQ_crs);
        }
    } else if (np > 1) {
        // Warps with threadIdx.y % np == 0 execute a __syncthreads() in the if branch.
        // Therefore, all other warps also need to execute a __syncthreads().
        // Otherwise the points at which warps synchronize with each other would become misaligned.
        __syncthreads();
    }

#pragma unroll
    for (int k00 = 0; k00 < DV/2; k00 += nbatch_combine) {
        if constexpr (ntiles == 1) {
            const int jc_cwd = threadIdx.y*tile_B::I + tile_B::get_i(-1); // jc combine write data
#pragma unroll
            for (int k0 = 0; k0 < nbatch_combine; k0 += tile_B::J) {
                const tile_B B = get_transposed(VKQ_C[(k00 + k0)/tile_B::J]); // Conversion of C to B matrix puts it in column-major format.

#pragma unroll
                for (int l = 0; l < tile_B::ne; ++l) {
                    const int k = k0 + tile_B::get_j(l);

                    tile_Q[jc_cwd*tile_stride + k] = B.x[l];
                }
            }
        } else {
#pragma unroll
            for (int t = 0; t < ntiles/2; ++t) {
                const int j0 = threadIdx.y*cols_per_warp + t*tile_C_VKQ_16::I;
#pragma unroll
                for (int k0 = 0; k0 < nbatch_combine; k0 += tile_C_VKQ_16::J) {
#pragma unroll
                    for (int l = 0; l < tile_C_VKQ_16::ne; ++l) {
                        const int j = j0 + tile_C_VKQ_16::get_i(l);
                        const int k = k0 + tile_C_VKQ_16::get_j(l);

                        tile_Q[j*tile_stride + k] = VKQ_C_16[(k00 + k0)/tile_C_VKQ_16::J * ntiles/2 + t].x[l];
                    }
                }
            }
        }

        __syncthreads();

        if (np == 1 || threadIdx.y % np == 0) {
            // The first 2*2*gridDim.x*ncols floats in dstk_fixup are for storing max. values and row sums.
            // The values after that are for the partial results of the individual blocks.
            float2 * dstk_fixup_data = dstk_fixup + gridDim.x*(2*ncols) + blockIdx.x*(ncols*(DV/2));

#pragma unroll
            for (int stride_k : {WARP_SIZE, WARP_SIZE/2, WARP_SIZE/4}) {
                const int k0_start  = stride_k == WARP_SIZE ? 0 : nbatch_combine - nbatch_combine % (2*stride_k);
                const int k0_stop   =                             nbatch_combine - nbatch_combine % (1*stride_k);
                const int stride_jc = WARP_SIZE / stride_k;

                if (k0_start == k0_stop) {
                    continue;
                }

#pragma unroll
                for (int jc0_dst = 0; jc0_dst < ncols; jc0_dst += (nwarps/np)*stride_jc) {
                    const int jc_dst = jc0_dst + (threadIdx.y/np)*stride_jc + (stride_k == WARP_SIZE ? 0 : threadIdx.x / stride_k);

                    if (jc0_dst + (nwarps/np)*stride_jc > ncols && jc_dst >= ncols) {
                        break;
                    }

                    const int jc_tile_K = (jc_dst/cols_per_warp)*(np*cols_per_warp) + jc_dst % cols_per_warp;

                    const int j_dst = jc_dst / ncols2;
                    const int c_dst = jc_dst % ncols2;

                    if (!is_fixup && jt*ncols1 + j_dst >= ne01) {
                        continue;
                    }

                    const float * meta_j = (const float *) tile_Q + jc_tile_K*tile_stride + nbatch_combine;
#pragma unroll
                    for (int k0 = k0_start; k0 < k0_stop; k0 += stride_k) {
                        const int k = k0 + (stride_k == WARP_SIZE ? threadIdx.x : threadIdx.x % stride_k);

                        float2 dstk_val = make_float2(0.0f, 0.0f);
#pragma unroll
                        for (int ip = 0; ip < np; ++ip) {
                            const float KQ_crs = np == 1 ? 1.0f : meta_j[ip*cols_per_warp * tile_stride + 0];
                            const float2 dstk_val_add = __half22float2(tile_Q[(jc_tile_K + ip*cols_per_warp) * tile_stride + k]);
                            dstk_val.x += dstk_val_add.x*KQ_crs;
                            dstk_val.y += dstk_val_add.y*KQ_crs;
                        }

                        if (!needs_fixup && !is_fixup) {
                            const float KQ_rowsum_j = meta_j[1];
                            dstk_val.x /= KQ_rowsum_j;
                            dstk_val.y /= KQ_rowsum_j;
                        }

                        if (is_fixup) {
                            dstk_fixup_data[jc_dst*(DV/2) + k00 + k] = dstk_val;
                        } else {
                            dstk[((jt*ncols1 + j_dst)*ne02 + c_dst)*(DV/2) + k00 + k] = dstk_val;
                        }
                    }
                }
            }
        }
        if (np > 1) {
            __syncthreads();
        }
    }
#else
    GGML_UNUSED(Q_f2); GGML_UNUSED(K_h2); GGML_UNUSED(V_h2);
    GGML_UNUSED(mask_h2); GGML_UNUSED(dstk); GGML_UNUSED(dstk_fixup);
    GGML_UNUSED(scale); GGML_UNUSED(slope); GGML_UNUSED(logit_softcap);
    GGML_UNUSED(ne01); GGML_UNUSED(ne02); GGML_UNUSED(stride_Q1);
    GGML_UNUSED(stride_Q2); GGML_UNUSED(stride_K); GGML_UNUSED(stride_V); GGML_UNUSED(stride_mask);
    GGML_UNUSED(jt); GGML_UNUSED(kb0_start); GGML_UNUSED(kb0_stop);
    NO_DEVICE_CODE;
#endif // INT8_MMA_AVAILABLE
}

template<int DKQ, int DV, int ncols1, int ncols2, int nwarps, int ntiles, bool use_logit_softcap, bool mla>
__launch_bounds__(nwarps*WARP_SIZE, 1)
static __global__ void flash_attn_ext_f16(
        const char * __restrict__ Q,
        const char * __restrict__ K,
        const char * __restrict__ V,
        const char * __restrict__ mask,
        float      * __restrict__ dst,
        float2     * __restrict__ dst_meta,
        const float scale,
        const float max_bias,
        const float m0,
        const float m1,
        const float logit_softcap,
        const uint32_t n_head_log2,
        const int ne00,
        const int ne01,
        const int ne02,
        const int ne03,
        const int ne10,
        const int ne11,
        const int ne12,
        const int ne13,
        const int ne31,
        const int nb31,
        const int nb01,
        const int nb02,
        const int nb03,
        const int nb11,
        const int nb12,
        const int nb13,
        const int nb21,
        const int nb22,
        const int nb23,
        const int ne0,
        const int ne1,
        const int ne2,
        const int ne3) {
#if defined(INT8_MMA_AVAILABLE)

    // Skip unused kernel variants for faster compilation:
    if constexpr (use_logit_softcap && !(DKQ == 128 || DKQ == 256)) {
        NO_DEVICE_CODE;
        return;
    }
#if __CUDA_ARCH__ == CC_TURING
    if constexpr (ncols1*ncols2 > 32) {
        NO_DEVICE_CODE;
        return;
    }
#endif __CUDA_ARCH__ == CC_TURING

    static_assert(!mla || DKQ >= DV, "MLA needs DKQ >= DV");

    typedef fattn_mma_f16_config<DKQ, DV> c;

    static_assert(FATTN_KQ_STRIDE % fattn_mma_f16_config<DKQ, DV>::nbatch_fa == 0, "bad nbatch_fa");

    const int gqa_ratio = ne02 / ne12; // With grouped query attention there are > 1 Q matrices per K, V matrix.

    const int stride_Q1   = nb01 / sizeof(float2);
    const int stride_Q2   = nb02 / sizeof(float2);
    const int stride_K    = nb11 / sizeof(half2);
    const int stride_mask = nb31 / sizeof(half2);

    const int stride_V = mla ? stride_K : nb21 / sizeof(half2);

    const int iter_k = ne11 / FATTN_KQ_STRIDE;
    const int iter_j = (ne01 + (ncols1 - 1)) / ncols1;

    constexpr int kb_niter = FATTN_KQ_STRIDE / c::nbatch_fa; // Number of kernel iterations per assigned KQ slice.

    // kbc == k block continuous, current index in continuous ijk space.
    int       kbc      = (blockIdx.x + 0)*iter_k*iter_j*(ne02/ncols2) / gridDim.x;
    const int kbc_stop = (blockIdx.x + 1)*iter_k*iter_j*(ne02/ncols2) / gridDim.x;

    // If the seams of 2 CUDA blocks fall within an output tile their results need to be combined.
    // For this we need to track both the block that starts the tile (needs_fixup) and the block that finishes the tile (is_fixup).
    // In the most general case >2 seams can fall into the same tile.

    // kb0 == k start index when in the output tile.
    int kb0_start = kbc % iter_k;
    int kb0_stop  = min(iter_k, kb0_start + kbc_stop - kbc);
    while (kbc < kbc_stop && kb0_stop == iter_k) {
        const int channel = kbc / (iter_k*iter_j);
        const int jt      = (kbc - channel*iter_k*iter_j) / iter_k; // j index of current tile.

        const float2 * Q_f2    = (const float2 *) (Q + nb02* channel*ncols2);
        const half2  * K_h2    = (const half2  *) (K + nb12*(channel*ncols2 / gqa_ratio));
        const half2  * mask_h2 = ncols2 > 1 || mask ? (const half2  *) mask + (nb31/sizeof(half2))*jt*ncols1 : nullptr;
        float2       * dstk    = ((float2 *) dst) + channel*(ncols2 * DV/2);

        const half2 * V_h2 = mla ? K_h2 + (DKQ/2 - DV/2) : (const half2 *) (V + nb22*(channel*ncols2 / gqa_ratio));

        const float slope = ncols2 == 1 ? get_alibi_slope(max_bias, channel, n_head_log2, m0, m1) : 1.0f;

        const int kb0_start_kernel = kb0_start * kb_niter;
        const int kb0_stop_kernel  = kb0_stop  * kb_niter;

        constexpr bool is_fixup = false; // All but (potentially) the last iterations write their data to dst rather than the fixup buffer.
        if (kb0_start == 0) {
            constexpr bool needs_fixup = false; // CUDA block is working on an entire tile.
            flash_attn_ext_f16_process_tile<DKQ, DV, ncols1, ncols2, nwarps, ntiles, use_logit_softcap, mla, needs_fixup, is_fixup>
                (Q_f2, K_h2, V_h2, mask_h2, dstk, dst_meta, scale, slope, logit_softcap,
                 ne01, ne02, stride_Q1, stride_Q2, stride_K, stride_V, stride_mask, jt, kb0_start_kernel, kb0_stop_kernel);
        } else {
            constexpr bool needs_fixup = true; // CUDA block is working on the beginning of a tile.
            flash_attn_ext_f16_process_tile<DKQ, DV, ncols1, ncols2, nwarps, ntiles, use_logit_softcap, mla, needs_fixup, is_fixup>
                (Q_f2, K_h2, V_h2, mask_h2, dstk, dst_meta, scale, slope, logit_softcap,
                 ne01, ne02, stride_Q1, stride_Q2, stride_K, stride_V, stride_mask, jt, kb0_start_kernel, kb0_stop_kernel);
        }

        kbc += iter_k;
        kbc -= kbc % iter_k;

        kb0_start = 0;
        kb0_stop  = min(iter_k, kbc_stop - kbc);
    }

    if (kbc >= kbc_stop) {
        return;
    }

    const int channel = kbc / (iter_k*iter_j);
    const int jt      = (kbc - channel*iter_k*iter_j) / iter_k; // j index of current tile.

    const float2 * Q_f2    = (const float2 *) (Q + nb02* channel*ncols2);
    const half2  * K_h2    = (const half2  *) (K + nb12*(channel*ncols2 / gqa_ratio));
    const half2  * mask_h2 = ncols2 > 1 || mask ? (const half2  *) mask + (nb31/sizeof(half2))*jt*ncols1 : nullptr;
    float2       * dstk    = ((float2 *) dst) + channel*(ncols2 * DV/2);

    const half2 * V_h2 = mla ? K_h2 + (DKQ/2 - DV/2) : (const half2 *) (V + nb22*(channel*ncols2 / gqa_ratio));

    const float slope = ncols2 == 1 ? get_alibi_slope(max_bias, channel, n_head_log2, m0, m1) : 1.0f;

    const int kb0_start_kernel = kb0_start * kb_niter;
    const int kb0_stop_kernel  = kb0_stop  * kb_niter;

    constexpr bool is_fixup = true; // Last index writes its data to fixup buffer to avoid data races with other blocks.
    constexpr bool needs_fixup = false;
    flash_attn_ext_f16_process_tile<DKQ, DV, ncols1, ncols2, nwarps, ntiles, use_logit_softcap, mla, needs_fixup, is_fixup>
        (Q_f2, K_h2, V_h2, mask_h2, dstk, dst_meta, scale, slope, logit_softcap,
         ne01, ne02, stride_Q1, stride_Q2, stride_K, stride_V, stride_mask, jt, kb0_start_kernel, kb0_stop_kernel);
#else
    GGML_UNUSED(Q); GGML_UNUSED(K); GGML_UNUSED(V); GGML_UNUSED(mask);
    GGML_UNUSED(dst); GGML_UNUSED(dst_meta); GGML_UNUSED(scale);
    GGML_UNUSED(max_bias); GGML_UNUSED(m0); GGML_UNUSED(m1);
    GGML_UNUSED(n_head_log2); GGML_UNUSED(logit_softcap); GGML_UNUSED(ne00);
    GGML_UNUSED(ne01); GGML_UNUSED(ne02); GGML_UNUSED(ne03); GGML_UNUSED(ne10);
    GGML_UNUSED(ne11); GGML_UNUSED(ne12); GGML_UNUSED(ne13); GGML_UNUSED(ne31);
    GGML_UNUSED(nb31); GGML_UNUSED(nb01); GGML_UNUSED(nb02); GGML_UNUSED(nb03);
    GGML_UNUSED(nb11); GGML_UNUSED(nb12); GGML_UNUSED(nb13); GGML_UNUSED(nb21);
    GGML_UNUSED(nb22); GGML_UNUSED(nb23); GGML_UNUSED(ne0); GGML_UNUSED(ne1);
    GGML_UNUSED(ne2); GGML_UNUSED(ne3);
    NO_DEVICE_CODE;
#endif // defined(INT8_MMA_AVAILABLE)
}


template<int D, int ncols1, int ncols2> // D == head size
__launch_bounds__(D, 1)
static __global__ void flash_attn_stream_k_fixup(
        float * __restrict__ dst, const float2 * __restrict__ dst_fixup, const int ne01, const int ne02, const int ne11) {
    constexpr int ncols = ncols1*ncols2;

    const int bidx0 = blockIdx.x;
    const int j     = blockIdx.y;
    const int c     = blockIdx.z;
    const int jc    = j*ncols2 + c;
    const int tid   = threadIdx.x;

    const float * dst_fixup_data = ((const float *) dst_fixup) + gridDim.x*(2*2*ncols);

    const int iter_k = ne11 / FATTN_KQ_STRIDE;
    const int iter_j = (ne01 + (ncols1 - 1)) / ncols1;

    const int kbc0      = (bidx0 + 0)*iter_k*iter_j*(ne02/ncols2) / gridDim.x;
    const int kbc0_stop = (bidx0 + 1)*iter_k*iter_j*(ne02/ncols2) / gridDim.x;

    const bool did_not_have_any_data   = kbc0 == kbc0_stop;
    const bool wrote_beginning_of_tile = kbc0 % iter_k == 0;
    const bool did_not_write_last      = kbc0/iter_k == kbc0_stop/iter_k && kbc0_stop % iter_k != 0;
    if (did_not_have_any_data || wrote_beginning_of_tile || did_not_write_last) {
        return;
    }

    const int channel = kbc0 / (iter_k*iter_j);
    const int jt      = (kbc0 - channel*iter_k*iter_j) / iter_k;

    if (jt*ncols1 + j >= ne01) {
        return;
    }

    dst += jt*ne02*(ncols1*D) + channel*(ncols2*D) + (j*ne02 + c)*D + tid;

    // Load the partial result that needs a fixup:
    float dst_val = 0.0f;
    float max_val = 0.0f;
    float rowsum  = 0.0f;
    {
        dst_val = *dst;

        const float2 tmp = dst_fixup[bidx0*ncols + jc];
        max_val = tmp.x;
        rowsum  = tmp.y;
    }

    // Iterate over previous blocks and compute the combined results.
    // All CUDA blocks that get here must have a previous block that needs a fixup.
    int bidx = bidx0 - 1;
    int kbc_stop = kbc0;
    while(true) {
        const int kbc = bidx*iter_k*iter_j*(ne02/ncols2) / gridDim.x;
        if (kbc == kbc_stop) { // Did not have any data.
            bidx--;
            kbc_stop = kbc;
            continue;
        }

        const float dst_add = dst_fixup_data[bidx*ncols*D + jc*D + tid];

        const float2 tmp = dst_fixup[(gridDim.x + bidx)*ncols + jc];

        // Scale the current and new value accumulators depending on the max. values.
        const float max_val_new = fmaxf(max_val, tmp.x);

        const float diff_val = max_val - max_val_new;
        const float diff_add = tmp.x   - max_val_new;

        const float scale_val = diff_val >= SOFTMAX_FTZ_THRESHOLD ? expf(diff_val) : 0.0f;
        const float scale_add = diff_add >= SOFTMAX_FTZ_THRESHOLD ? expf(diff_add) : 0.0f;

        dst_val = scale_val*dst_val + scale_add*dst_add;
        rowsum  = scale_val*rowsum  + scale_add*tmp.y;

        max_val = max_val_new;

        // If this block started in a previous tile we are done and don't need to combine additional partial results.
        if (kbc % iter_k == 0 || kbc/iter_k < kbc0/iter_k) {
            break;
        }
        bidx--;
        kbc_stop = kbc;
    }

    // Write back final result:
    *dst = dst_val / rowsum;
}

template<int D> // D == head size
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
__launch_bounds__(D, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void flash_attn_combine_results_new(
        const float  * __restrict__ VKQ_parts,
        const float2 * __restrict__ VKQ_meta,
        float * __restrict__ dst,
        const int parallel_blocks) {
    VKQ_parts += parallel_blocks*D * gridDim.z*blockIdx.x;
    VKQ_meta  += parallel_blocks   * gridDim.z*blockIdx.x;
    dst       +=                 D * gridDim.z*blockIdx.x;

    const int tid = threadIdx.x;
    __builtin_assume(tid < D);

    extern __shared__ float2 meta[];
    if (tid < 2*parallel_blocks) {
        ((float *) meta)[threadIdx.x] = ((const float *)VKQ_meta) [blockIdx.z*(2*parallel_blocks) + tid];
    }

    __syncthreads();

    float kqmax = meta[0].x;
    for (int l = 1; l < parallel_blocks; ++l) {
        kqmax = max(kqmax, meta[l].x);
    }

    float VKQ_numerator   = 0.0f;
    float VKQ_denominator = 0.0f;
    for (int l = 0; l < parallel_blocks; ++l) {
        const float diff = meta[l].x - kqmax;
        float KQ_max_scale = expf(diff);
        const uint32_t ftz_mask = 0xFFFFFFFF * (diff > SOFTMAX_FTZ_THRESHOLD);
        *((uint32_t *) &KQ_max_scale) &= ftz_mask;

        VKQ_numerator   += KQ_max_scale * VKQ_parts[l*gridDim.z*D + blockIdx.z*D + tid];
        VKQ_denominator += KQ_max_scale * meta[l].y;
    }

    dst[blockIdx.z*D + tid] = VKQ_numerator / VKQ_denominator;
}

template <int DV, int ncols1, int ncols2>
static void launch_fattn_new_mma(
    ggml_backend_cuda_context & ctx, ggml_tensor * dst, fattn_kernel_t fattn_kernel, const int nwarps, const size_t nbytes_shared,
    const int KQ_row_granularity, const bool need_f16_K, const bool need_f16_V, const bool stream_k, const int warp_size = WARP_SIZE
) {
    constexpr int ncols = ncols1 * ncols2;

    const ggml_tensor * Q = dst->src[0];
    const ggml_tensor * K = dst->src[1];
    const ggml_tensor * V = dst->src[2];

    const ggml_tensor * mask = dst->src[3];

    ggml_tensor * KQV = dst;

    GGML_ASSERT(Q->type == GGML_TYPE_F32);
    GGML_ASSERT(KQV->type == GGML_TYPE_F32);

    GGML_ASSERT(!mask || mask->type == GGML_TYPE_F16);
    GGML_ASSERT(!mask || mask->ne[1] >= GGML_PAD(Q->ne[1], 16) &&
        "the Flash-Attention CUDA kernel requires the mask to be padded to 16 and at least n_queries big");

    GGML_ASSERT(K->ne[1] % FATTN_KQ_STRIDE == 0 && "Incorrect KV cache padding.");

    GGML_ASSERT(Q->ne[3] == 1);

    ggml_cuda_pool & pool = ctx.pool();
    hipStream_t main_stream = ctx.stream();
    const int id  = ggml_cuda_get_device();
    const int cc  = ggml_cuda_info().devices[id].cc;
    const int nsm = ggml_cuda_info().devices[id].nsm;

    ggml_cuda_pool_alloc<half>   K_f16(pool);
    ggml_cuda_pool_alloc<half>   V_f16(pool);
    ggml_cuda_pool_alloc<float>  dst_tmp(pool);
    ggml_cuda_pool_alloc<float2> dst_tmp_meta(pool);

    const char * K_data = (const char *) K->data;
    size_t nb11 = K->nb[1];
    size_t nb12 = K->nb[2];
    size_t nb13 = K->nb[3];

    const char * V_data = (const char *) V->data;
    size_t nb21 = V->nb[1];
    size_t nb22 = V->nb[2];
    size_t nb23 = V->nb[3];

    if (need_f16_K && K->type != GGML_TYPE_F16) {
        K_f16.alloc(ggml_nelements(K));
        to_fp16_cuda_t to_fp16 = ggml_get_to_fp16_cuda(K->type);
        to_fp16(K_data, K_f16.ptr, 1, ggml_nelements(K), main_stream);
        K_data = (char *) K_f16.ptr;

        nb11 = K->ne[0]*sizeof(half);
        nb12 = nb11*K->ne[1];
        nb13 = nb12*K->ne[2];

        // Original PR in llama.cpp. I don't think that can work when K is not contiguous (e.g., nb11 > nb12), there are
        //                           gaps between the rows, etc., as ggml_get_to_fp16_cuda stores into contiguous memory.
        //const size_t bs = ggml_blck_size(K->type);
        //const size_t ts = ggml_type_size(K->type);

        //nb11 = nb11*bs*sizeof(half)/ts;
        //nb12 = nb12*bs*sizeof(half)/ts;
        //nb13 = nb13*bs*sizeof(half)/ts;
    }

    if (need_f16_V && V->type != GGML_TYPE_F16) {
        if constexpr (DV == 512) {
            // DeepSeek. In this case the V cache is the same as the K cache, except that
            //           it has 512 elements per row instead of 576.
            nb21 = nb11;
            nb22 = nb12;
            nb23 = nb13;
            V_data = K_data;
        } else {
            V_f16.alloc(ggml_nelements(V));
            to_fp16_cuda_t to_fp16 = ggml_get_to_fp16_cuda(V->type);
            to_fp16(V_data, V_f16.ptr, 1, ggml_nelements(V), main_stream);
            V_data = (char *) V_f16.ptr;

            nb21 = K->ne[0]*sizeof(half);
            nb22 = nb21*V->ne[1];
            nb23 = nb22*V->ne[2];

            // Original PR in llama.cpp. Same comment as above for the K cache.
            //const size_t bs = ggml_blck_size(V->type);
            //const size_t ts = ggml_type_size(V->type);

            //nb21 = nb21*bs*sizeof(half)/ts;
            //nb22 = nb22*bs*sizeof(half)/ts;
            //nb23 = nb23*bs*sizeof(half)/ts;
        }
    }

    int parallel_blocks = 1;

    const int ntiles_x = ((Q->ne[1] + ncols1 - 1) / ncols1);
    const int ntiles_total = ntiles_x * (Q->ne[2] / ncols2) * Q->ne[3];

    const dim3 block_dim(warp_size, nwarps, 1);
    int max_blocks_per_sm = 1; // Max. number of active blocks limited by occupancy.
    CUDA_CHECK(hipOccupancyMaxActiveBlocksPerMultiprocessor(&max_blocks_per_sm, fattn_kernel, block_dim.x * block_dim.y * block_dim.z, nbytes_shared));

    dim3 blocks_num;
    if (stream_k) {
        // For short contexts it can be faster to have the SMs work on whole tiles because this lets us skip the fixup.
        const int max_blocks = max_blocks_per_sm*nsm;
        const int tiles_nwaves = (ntiles_total + max_blocks - 1) / max_blocks;
        const int tiles_efficiency_percent = 100 * ntiles_total / (max_blocks*tiles_nwaves);

        const int nblocks_stream_k = max_blocks;

        const bool use_stream_k = cc >= CC_ADA_LOVELACE || tiles_efficiency_percent < 75;

        blocks_num.x = use_stream_k ? nblocks_stream_k : ntiles_total;
        blocks_num.y = 1;
        blocks_num.z = 1;

        dst_tmp_meta.alloc(blocks_num.x*ncols * (2*2 + DV) * sizeof(float));
    } else {
        GGML_ASSERT(K->ne[1] % KQ_row_granularity == 0);
        const int ntiles_KQ = K->ne[1] / KQ_row_granularity; // Max. number of parallel blocks limited by tensor size.

        // parallel_blocks should be at least large enough to achieve max. occupancy for a single wave:
        parallel_blocks = std::max((nsm * max_blocks_per_sm) / ntiles_total, 1);

        // parallel_blocks must not be larger than what the tensor size allows:
        parallel_blocks = std::min(parallel_blocks, ntiles_KQ);

        // If ntiles_total % blocks_per_wave != 0 then some efficiency is lost due to tail effects.
        // Test whether parallel_blocks can be set to a higher value for better efficiency.
        const int blocks_per_wave = nsm * max_blocks_per_sm;
        int nwaves_best = 0;
        int efficiency_percent_best = 0;
        for (int parallel_blocks_test = parallel_blocks; parallel_blocks_test <= ntiles_KQ; ++parallel_blocks_test) {
            const int nblocks_total = ntiles_total * parallel_blocks_test;
            const int nwaves = (nblocks_total + blocks_per_wave - 1) / blocks_per_wave;
            const int efficiency_percent = 100 * nblocks_total / (nwaves*blocks_per_wave);

            // Stop trying configurations with more waves if we already have good efficiency to avoid excessive overhead.
            if (efficiency_percent_best >= 90 && nwaves > nwaves_best) {
                break;
            }

            if (efficiency_percent > efficiency_percent_best) {
                nwaves_best = nwaves;
                efficiency_percent_best = efficiency_percent;
                parallel_blocks = parallel_blocks_test;
            }
        }

        blocks_num.x = ntiles_x;
        blocks_num.y = parallel_blocks;
        blocks_num.z = Q->ne[2]*Q->ne[3];

        if (parallel_blocks > 1) {
            dst_tmp.alloc(parallel_blocks*ggml_nelements(KQV));
            dst_tmp_meta.alloc(parallel_blocks*ggml_nrows(KQV));
        }
    }
    float scale         = 1.0f;
    float max_bias      = 0.0f;
    float logit_softcap = 0.0f;

    memcpy(&scale,         (const float *) KQV->op_params + 0, sizeof(float));
    memcpy(&max_bias,      (const float *) KQV->op_params + 1, sizeof(float));
    memcpy(&logit_softcap, (const float *) KQV->op_params + 2, sizeof(float));

    if (logit_softcap != 0.0f) {
        scale /= logit_softcap;
    }

    const uint32_t n_head      = Q->ne[2];
    const uint32_t n_head_log2 = 1u << uint32_t(floorf(log2f(float(n_head))));

    const float m0 = powf(2.0f, -(max_bias       ) / n_head_log2);
    const float m1 = powf(2.0f, -(max_bias / 2.0f) / n_head_log2);

    GGML_ASSERT(block_dim.x % warp_size == 0);
    fattn_kernel<<<blocks_num, block_dim, nbytes_shared, main_stream>>>(
        (const char *) Q->data,
        K_data,
        V_data,
        mask ? ((const char *) mask->data) : nullptr,
        !stream_k && parallel_blocks > 1 ? dst_tmp.ptr : (float *) KQV->data, dst_tmp_meta.ptr,
        scale, max_bias, m0, m1, logit_softcap, n_head_log2,
        Q->ne[0], Q->ne[1], Q->ne[2], Q->ne[3],
        K->ne[0], K->ne[1], K->ne[2], K->ne[3],
        mask ? mask->ne[1] : 0, mask ?  mask->nb[1] : 0,
        Q->nb[1], Q->nb[2], Q->nb[3],
        nb11, nb12, nb13,
        nb21, nb22, nb23,
        KQV->ne[0], KQV->ne[1], KQV->ne[2], KQV->ne[3]
    );
    CUDA_CHECK(hipGetLastError());

    if (stream_k) {
        if (ntiles_total % blocks_num.x != 0) { // Fixup is only needed if the SMs work on fractional tiles.
            const dim3 block_dim_combine(DV, 1, 1);
            const dim3 blocks_num_combine = {blocks_num.x, ncols1, ncols2};

            flash_attn_stream_k_fixup<DV, ncols1, ncols2>
                <<<blocks_num_combine, block_dim_combine, 0, main_stream>>>
                ((float *) KQV->data, dst_tmp_meta.ptr, Q->ne[1], Q->ne[2], K->ne[1]);
        }
    } else if (parallel_blocks > 1) {
        const dim3 block_dim_combine(DV, 1, 1);
        const dim3 blocks_num_combine(Q->ne[1], 1, blocks_num.z);
        const size_t nbytes_shared_combine = parallel_blocks*sizeof(float2);

        flash_attn_combine_results_new<DV>
            <<<blocks_num_combine, block_dim_combine, nbytes_shared_combine, main_stream>>>
            (dst_tmp.ptr, dst_tmp_meta.ptr, (float *) KQV->data, parallel_blocks);
    }
    CUDA_CHECK(hipGetLastError());
}


template <int DKQ, int DV, int ncols1, int ncols2>
static void ggml_cuda_flash_attn_ext_mma_f16_case(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV = dst;
    const int id = ggml_cuda_get_device();
    const int cc = ggml_cuda_info().devices[id].cc;

    typedef fattn_mma_f16_config<DKQ, DV> c;

    const int nstages = cp_async_available(cc) ? c::nstages_target : 0;

    constexpr int ncols         = ncols1 * ncols2;
    constexpr int ntiles        = ncols <= 8 ? 1 : 2; // Number of tiles per warp.
    constexpr int cols_per_warp = ntiles * tile_B::I;
    constexpr int nwarps_max_x  = ncols / cols_per_warp;
    constexpr int nwarps_max_y  = c::nbatch_fa / tile_A::I;
    constexpr int nwarps        = nwarps_max_x*nwarps_max_y <= c::nwarps_max ? nwarps_max_x*nwarps_max_y : c::nwarps_max;

    constexpr bool mla = DKQ == 576;

    const int nbatch_K2      = c::get_nbatch_K2_host     (cc, ncols);
    const int nbatch_V2      = c::get_nbatch_K2_host     (cc, ncols);
    const int nbatch_combine = c::get_nbatch_combine_host(cc, ncols);

    static_assert(DKQ   % tile_B::J     == 0, "bad DKQ");
    static_assert(DV    % tile_A::J     == 0, "bad DV");
    static_assert(ncols % cols_per_warp == 0, "bad ncols");

    const size_t nbytes_shared_KV_1stage = c::nbatch_fa         * std::max(nbatch_K2 + 4,  nbatch_V2 + 4) * sizeof(half2);
    const size_t nbytes_shared_KV_2stage = c::nbatch_fa         *         (nbatch_K2 + 4 + nbatch_V2 + 4) * sizeof(half2);
    const size_t nbytes_shared_Q         = ncols                * (DKQ/2 + 4)                             * sizeof(half2);
    const size_t nbytes_shared_mask      = ncols1               * (c::nbatch_fa/2 + 4)                    * sizeof(half2);
    const size_t nbytes_shared_combine   = nwarps*cols_per_warp * (nbatch_combine + 4)                    * sizeof(half2);

    const size_t nbytes_shared_KV = nstages <= 1 ? nbytes_shared_KV_1stage : nbytes_shared_KV_2stage;

    const size_t nbytes_shared_total = std::max(nbytes_shared_combine, c::Q_in_reg ?
        std::max(nbytes_shared_Q,  nbytes_shared_KV + nbytes_shared_mask) :
                 nbytes_shared_Q + nbytes_shared_KV + nbytes_shared_mask);

    float logit_softcap;
    memcpy(&logit_softcap, (const float *) KQV->op_params + 2, sizeof(float));

    fattn_kernel_t fattn_kernel;
    if (logit_softcap == 0.0f) {
        constexpr bool use_logit_softcap = false;
        fattn_kernel = flash_attn_ext_f16<DKQ, DV, ncols1, ncols2, nwarps, ntiles, use_logit_softcap, mla>;

#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)) && !defined(GGML_USE_MUSA)
        static bool shared_memory_limit_raised[GGML_CUDA_MAX_DEVICES] = {false};
        if (!shared_memory_limit_raised[id]) {
            CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(fattn_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, nbytes_shared_total));
            shared_memory_limit_raised[id] = true;
        }
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)) && !defined(GGML_USE_MUSA)
    } else {
        constexpr bool use_logit_softcap = true;
        fattn_kernel = flash_attn_ext_f16<DKQ, DV, ncols1, ncols2, nwarps, ntiles, use_logit_softcap, mla>;

#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)) && !defined(GGML_USE_MUSA)
        static bool shared_memory_limit_raised[GGML_CUDA_MAX_DEVICES] = {false};
        if (!shared_memory_limit_raised[id]) {
            CUDA_CHECK(hipFuncSetAttribute(reinterpret_cast<const void*>(fattn_kernel), hipFuncAttributeMaxDynamicSharedMemorySize, nbytes_shared_total));
            shared_memory_limit_raised[id] = true;
        }
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__)) && !defined(GGML_USE_MUSA)
    }

    launch_fattn_new_mma<DV, ncols1, ncols2>
        (ctx, dst, fattn_kernel, nwarps, nbytes_shared_total, FATTN_KQ_STRIDE, true, true, true);
}

template <int DKQ, int DV, int ncols2>
static void ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * Q = dst->src[0];

    if constexpr (ncols2 <= 8) {
        if (Q->ne[1] <= 8/ncols2) {
            ggml_cuda_flash_attn_ext_mma_f16_case<DKQ, DV, 8/ncols2, ncols2>(ctx, dst);
            return;
        }
    }

    if (Q->ne[1] <= 16/ncols2) {
        ggml_cuda_flash_attn_ext_mma_f16_case<DKQ, DV, 16/ncols2, ncols2>(ctx, dst);
        return;
    }

    if (Q->ne[1] <= 32/ncols2) {
        ggml_cuda_flash_attn_ext_mma_f16_case<DKQ, DV, 32/ncols2, ncols2>(ctx, dst);
        return;
    }

    ggml_cuda_flash_attn_ext_mma_f16_case<DKQ, DV, 64/ncols2, ncols2>(ctx, dst);
}

template <int DKQ, int DV>
static void ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV  = dst;
    const ggml_tensor * Q    = dst->src[0];
    const ggml_tensor * K    = dst->src[1];
    const ggml_tensor * mask = dst->src[3];

    float max_bias = 0.0f;
    memcpy(&max_bias, (const float *) KQV->op_params + 1, sizeof(float));

    const bool use_gqa_opt = mask && max_bias == 0.0f;

    GGML_ASSERT(Q->ne[2] % K->ne[2] == 0);
    const int gqa_ratio = Q->ne[2] / K->ne[2];

    if (use_gqa_opt && gqa_ratio % 8 == 0) {
        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<DKQ, DV, 8>(ctx, dst);
        return;
    }

    if (use_gqa_opt && gqa_ratio % 4 == 0) {
        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<DKQ, DV, 4>(ctx, dst);
        return;
    }

    if (use_gqa_opt && gqa_ratio % 2 == 0) {
        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<DKQ, DV, 2>(ctx, dst);
        return;
    }

    ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<DKQ, DV, 1>(ctx, dst);
}

void ggml_cuda_flash_attn_ext_mma_new(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * KQV  = dst;
    const ggml_tensor * Q    = dst->src[0];
    const ggml_tensor * K    = dst->src[1];
    const ggml_tensor * V    = dst->src[2];
    const ggml_tensor * mask = dst->src[3];

    GGML_ASSERT(Q->ne[0] == 576 && K->ne[0] == 576 && V->ne[0] == 512);

    float max_bias = 0.0f;
    memcpy(&max_bias, (const float *) KQV->op_params + 1, sizeof(float));

    const bool use_gqa_opt = mask && max_bias == 0.0f;
    GGML_ASSERT(use_gqa_opt);

    GGML_ASSERT(Q->ne[2] % K->ne[2] == 0);
    const int gqa_ratio = Q->ne[2] / K->ne[2];
    GGML_ASSERT(gqa_ratio % 16 == 0);
    ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<576, 512, 16>(ctx, dst);

    //switch (Q->ne[0]) {
    //    case 64:
    //        GGML_ASSERT(V->ne[0] == 64);
    //        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2< 64,  64>(ctx, dst);
    //        break;
    //    case 80:
    //        GGML_ASSERT(V->ne[0] == 80);
    //        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2< 80,  80>(ctx, dst);
    //        break;
    //    case 96:
    //        GGML_ASSERT(V->ne[0] == 96);
    //        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2< 96,  96>(ctx, dst);
    //        break;
    //    case 112:
    //        GGML_ASSERT(V->ne[0] == 112);
    //        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2<112, 112>(ctx, dst);
    //        break;
    //    case 128:
    //        GGML_ASSERT(V->ne[0] == 128);
    //        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2<128, 128>(ctx, dst);
    //        break;
    //    case 192:
    //        GGML_ASSERT(V->ne[0] == 128);
    //        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2<192, 128>(ctx, dst);
    //        break;
    //    case 256:
    //        GGML_ASSERT(V->ne[0] == 256);
    //        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols2<256, 256>(ctx, dst);
    //        break;
    //    case 576: {
    //        // For Deepseek, go straight to the ncols1 switch to avoid compiling unnecessary kernels.
    //        GGML_ASSERT(V->ne[0] == 512);
    //        float max_bias = 0.0f;
    //        memcpy(&max_bias, (const float *) KQV->op_params + 1, sizeof(float));

    //        const bool use_gqa_opt = mask && max_bias == 0.0f;
    //        GGML_ASSERT(use_gqa_opt);

    //        GGML_ASSERT(Q->ne[2] % K->ne[2] == 0);
    //        const int gqa_ratio = Q->ne[2] / K->ne[2];
    //        GGML_ASSERT(gqa_ratio % 16 == 0);
    //        ggml_cuda_flash_attn_ext_mma_f16_switch_ncols1<576, 512, 16>(ctx, dst);
    //    } break;
    //    default:
    //        GGML_ABORT("fatal error");
    //        break;
    //}
}

