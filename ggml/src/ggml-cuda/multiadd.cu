#include "hip/hip_runtime.h"
#include "multiadd.cuh"

static __global__ void multi_add_f32(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, const char * src0, char * dst) {
    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;
    int64_t k = ne0*ne1;
    if (i >= k) {
        return;
    }
    int i1 = i / ne0;
    int i0 = i % ne0;
    float * result = (float *)(dst + i1*nb1);
    const float * s = (const float *)(src0 + i1*nb01) + i0;
    if (nused == 1) {
        result[i0] = s[0];
    } else {
        float sum = s[0] + s[ne0];
        for (int j = 2; j < nused; ++j) sum += s[j*ne0];
        result[i0] = sum;
    }
}

static void multi_add_f32_cuda(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, const char * src0, char * dst, hipStream_t stream) {
    int64_t k = ne0 * ne1;
    const int num_blocks = (k + CUDA_MULTI_ADD_BLOCK_SIZE - 1) / CUDA_MULTI_ADD_BLOCK_SIZE;
    multi_add_f32<<<num_blocks, CUDA_MULTI_ADD_BLOCK_SIZE, 0, stream>>>(nused, ne0, ne1, nb1, nb01, src0, dst);
}

void ggml_cuda_op_multi_add(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    GGML_ASSERT(dst->type == GGML_TYPE_F32);
    GGML_ASSERT(dst->ne[2] == 1 && dst->ne[3] == 1);
    GGML_ASSERT(dst->nb[0] == sizeof(float));
    int nused = dst->op_params[0];
    GGML_ASSERT(nused >= 1);
    const char * src0 = (const char *)dst->src[0]->data;
    hipStream_t stream = ctx.stream();
    multi_add_f32_cuda(nused, dst->ne[0], dst->ne[1], dst->nb[1], dst->src[0]->nb[1], src0, (char *)dst->data, stream);
}


static __global__ void mul_multi_add_f32(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, int64_t nb02, int64_t nb11, int64_t nb12, const char * src0, const char * src1, char * dst) {
    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;
    int64_t k = ne0*ne1;
    if (i >= k) {
        return;
    }
    int i1 = i / ne0;
    int i0 = i % ne0;
    float * result = (float *)(dst + i1*nb1);

    auto c0 = src0 + i1*nb02;
    auto c1 = src1 + i1*nb12;

    float sum = 0;
    for (int j = 0; j < nused; ++j) {
        auto x0 = (const float *)c0;
        auto x1 = (const float *)c1;
        sum += x0[i0] * x1[0];
        c0 += nb01;
        c1 += nb11;
    }
    result[i0] = sum;
}

static void mul_multi_add_f32_cuda(int nused, int64_t ne0, int64_t ne1, int64_t nb1, int64_t nb01, int64_t nb02, int64_t nb11, int64_t nb12,
        const char * src0, const char * src1, char * dst, hipStream_t stream) {
    int64_t k = ne0 * ne1;
    const int num_blocks = (k + CUDA_MULTI_ADD_BLOCK_SIZE - 1) / CUDA_MULTI_ADD_BLOCK_SIZE;
    mul_multi_add_f32<<<num_blocks, CUDA_MULTI_ADD_BLOCK_SIZE, 0, stream>>>(nused, ne0, ne1, nb1, nb01, nb02, nb11, nb12, src0, src1, dst);
}

void ggml_cuda_op_mul_multi_add(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    auto src0 = dst->src[0];
    auto src1 = dst->src[1];
    GGML_ASSERT(src0->type == GGML_TYPE_F32);
    GGML_ASSERT(src1->type == GGML_TYPE_F32);
    GGML_ASSERT( dst->type == GGML_TYPE_F32);
    GGML_ASSERT(src0->ne[0] ==  dst->ne[0]);
    GGML_ASSERT(src0->ne[2] ==  dst->ne[1]);
    GGML_ASSERT(src0->ne[1] == src1->ne[1]);
    GGML_ASSERT(src0->ne[2] == src1->ne[2]);
    GGML_ASSERT(src0->ne[3] == src1->ne[3]);
    GGML_ASSERT(src0->ne[3] == 1);
    GGML_ASSERT(src1->ne[0] == 1);

    mul_multi_add_f32_cuda(src0->ne[1], dst->ne[0], dst->ne[1], dst->nb[1], src0->nb[1], src0->nb[2], src1->nb[1], src1->nb[2],
            (const char *)src0->data, (const char *)src1->data, (char *)dst->data, ctx.stream());
}
