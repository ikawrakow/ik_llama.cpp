#include "hip/hip_runtime.h"
#include "../iqk_mmvq_templates.cuh"

__device__ __forceinline__ void vec_dot_iq1_m_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const half * dptr = (const half *)vbq;
    const block_iq1_m_r4 * bq1 = (const block_iq1_m_r4 *)(dptr + 4) + kbx;

    // iqs is 0 or 2
    const float d8 = __low2float(bq8_1->ds);
    const int32_t  * q8 = (const int *)bq8_1->qs;

    int32_t grid32[2];
    const int * igrid = (const int *)grid32;

    int minus1 = ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+0], ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+1], 0));
    int minus2 = ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+2], ggml_cuda_dp4a(0x01010101, q8[4*(iqs/2)+3], 0));

    for (int i = 0; i < 4; ++i) {
        float dl = __half2float(dptr[i])*((bq1->scales[i] >> 4*(iqs/2)) & 0xf) * d8;
        float ml1 = dl * (bq1->qh[4*(iqs/2)+i] & 0x08 ? -1-IQ1M_DELTA : -1+IQ1M_DELTA);
        float ml2 = dl * (bq1->qh[4*(iqs/2)+i] & 0x80 ? -1-IQ1M_DELTA : -1+IQ1M_DELTA);
        grid32[0] = iq1s_grid_gpu[bq1->qs[4*iqs+i] | ((bq1->qh[4*(iqs/2)+i] & 0x07) << 8)];
        grid32[1] = (grid32[0] >> 4) & 0x0f0f0f0f;
        grid32[0] &= 0x0f0f0f0f;
        int sumi = ggml_cuda_dp4a(igrid[0], q8[4*(iqs/2)+0], ggml_cuda_dp4a(igrid[1], q8[4*(iqs/2)+1], 0));
        grid32[0] = iq1s_grid_gpu[bq1->qs[4*iqs+i+4] | ((bq1->qh[4*(iqs/2)+i] & 0x70) << 4)];
        grid32[1] = (grid32[0] >> 4) & 0x0f0f0f0f;
        grid32[0] &= 0x0f0f0f0f;
        sumi = ggml_cuda_dp4a(igrid[0], q8[4*(iqs/2)+2], ggml_cuda_dp4a(igrid[1], q8[4*(iqs/2)+3], sumi));
        result[i] += dl * sumi + ml1 * minus1 + ml2*minus2;
    }
}

void mul_mat_vec_iq1_m_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ1_M_R4, 2, vec_dot_iq1_m_r4_q8_1, 4>(args, stream);
}

