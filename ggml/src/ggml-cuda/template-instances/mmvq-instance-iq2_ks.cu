#include "hip/hip_runtime.h"
#include "../iqk_mmvq_templates.cuh"

__device__ __forceinline__ void vec_dot_iq2_ks_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    float scale = *(const half *)vbq;
    const block_iq2_ks * bq2 = (const block_iq2_ks *)((const char *)vbq + sizeof(half)) + kbx;

    int i4 = iqs/4;  // 0...7. We will process q8 blocks 4*(i4/4), 4*(i4/4)+1, 4*(i4/4)+2, 4*(i4/4)+3
    const int32_t  * q8_1 = (const int *)bq8_1[4*(i4/4)+0].qs + 2*(i4%4);
    const int32_t  * q8_2 = (const int *)bq8_1[4*(i4/4)+1].qs + 2*(i4%4);
    const int32_t  * q8_3 = (const int *)bq8_1[4*(i4/4)+2].qs + 2*(i4%4);
    const int32_t  * q8_4 = (const int *)bq8_1[4*(i4/4)+3].qs + 2*(i4%4);

    const uint16_t * q2 = (const uint16_t *)bq2->qs + 16*(i4/4) + 4*(i4%4);
    const uint16_t extra = bq2->extra >> 4*(i4/4);

    uint32_t val1 = q2[0] | (q2[1] << 16), val2 = q2[2] | (q2[3] << 16);

    int32_t scales32;
    const uint16_t * scales16 = (const uint16_t *)bq2->scales;
    scales32 = __vsub4((scales16[i4/4] | (scales16[i4/4] << 12)) & 0x0f0f0f0f, 0x10101010);
    int8_t * s8 = (int8_t *)&scales32;
    s8[0] += ((extra >> 4) & 0x10);
    s8[1] += ((extra >> 6) & 0x10);
    s8[2] += ((extra >> 5) & 0x10);
    s8[3] += ((extra >> 7) & 0x10);

#ifdef __CUDA_ARCH__

    uint32_t extra32 = uint32_t(extra & 0xf) * 0x01010101;

    uint32_t this_extra = ((extra32 << 2) & 0x04040404) | ((extra32 << 4) & 0x40404040);
    uint32_t idx1 = ((val1 >> 0) & 0x33333333) | this_extra;
    uint32_t idx2 = ((val2 >> 0) & 0x33333333) | this_extra;
    int2 v1 = get_int_from_table_8(idx1, iq2nl_values);
    int2 v2 = get_int_from_table_8(idx2, iq2nl_values);

    int sumi1 = ggml_cuda_dp4a(v2.x, q8_1[1], ggml_cuda_dp4a(v1.x, q8_1[0], 0)) * s8[0];
    int sumi3 = ggml_cuda_dp4a(v2.y, q8_3[1], ggml_cuda_dp4a(v1.y, q8_3[0], 0)) * s8[1];

    this_extra = ((extra32 << 1) & 0x04040404) | ((extra32 << 3) & 0x40404040);
    idx1 = ((val1 >> 2) & 0x33333333) | this_extra;
    idx2 = ((val2 >> 2) & 0x33333333) | this_extra;
    v1 = get_int_from_table_8(idx1, iq2nl_values);
    v2 = get_int_from_table_8(idx2, iq2nl_values);

    int sumi2 = ggml_cuda_dp4a(v2.x, q8_2[1], ggml_cuda_dp4a(v1.x, q8_2[0], 0)) * s8[2];
    int sumi4 = ggml_cuda_dp4a(v2.y, q8_4[1], ggml_cuda_dp4a(v1.y, q8_4[0], 0)) * s8[3];

#else
    uint32_t aux32[2];
    int v1, v2;
    const int * all_values = (const int *)iq2k_table;
    const int * values;

    aux32[0] = ((val1 >> 0) & 0x03030303); aux32[1] = ((val2 >> 0) & 0x03030303); values = all_values + ((extra & 0x01) << 8);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi1 = ggml_cuda_dp4a(v2, q8_1[1], ggml_cuda_dp4a(v1, q8_1[0], 0)) * s8[0];

    aux32[0] = ((val1 >> 2) & 0x03030303); aux32[1] = ((val2 >> 2) & 0x03030303); values = all_values + ((extra & 0x02) << 7);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi2 = ggml_cuda_dp4a(v2, q8_2[1], ggml_cuda_dp4a(v1, q8_2[0], 0)) * s8[2];

    aux32[0] = ((val1 >> 4) & 0x03030303); aux32[1] = ((val2 >> 4) & 0x03030303); values = all_values + ((extra & 0x04) << 6);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi3 = ggml_cuda_dp4a(v2, q8_3[1], ggml_cuda_dp4a(v1, q8_3[0], 0)) * s8[1];

    aux32[0] = ((val1 >> 6) & 0x03030303); aux32[1] = ((val2 >> 6) & 0x03030303); values = all_values + ((extra & 0x08) << 5);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi4 = ggml_cuda_dp4a(v2, q8_4[1], ggml_cuda_dp4a(v1, q8_4[0], 0)) * s8[3];
#endif

    *result += scale * (__low2float(bq8_1[4*(i4/4)+0].ds) * sumi1
                     +  __low2float(bq8_1[4*(i4/4)+1].ds) * sumi2
                     +  __low2float(bq8_1[4*(i4/4)+2].ds) * sumi3
                     +  __low2float(bq8_1[4*(i4/4)+3].ds) * sumi4);
}

void mul_mat_vec_iq2_ks_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_KS, VDR_IQ2_KS_Q8_1_MMVQ, vec_dot_iq2_ks_q8_1>(args, stream);
}

