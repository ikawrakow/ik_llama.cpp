#include "../iqk_mmvq_templates.cuh"

void mul_mat_vec_iq3_ks_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ3_KS, VDR_IQ3_K_Q8_1_MMVQ, vec_dot_iq3_ks_q8_1>(args, stream);
}

