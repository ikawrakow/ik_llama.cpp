#include "hip/hip_runtime.h"
#include "../iqk_mmvq_templates.cuh"

__device__ __forceinline__ void vec_dot_iq5_k_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq5_k_r4 * bq5 = (const block_iq5_k_r4 *)vbq + kbx;

    // iqs is 0...28 in steps of 2
    const int ib16 = iqs/2;
    const float d8 = __low2float(bq8_1[ib16/2].ds);
    const int32_t  * q8 = (const int *)bq8_1[ib16/2].qs + 4*(ib16%2);

    int ib32 = ib16/2;
    int is   = ib16%2;
    int scales;
    const uint32_t * scales_l = (const uint32_t *)bq5->scales_l;
    const uint32_t * scales_h = (const uint32_t *)bq5->scales_h;
    scales = __vsub4(((scales_l[2*(ib32%4)+is] >> 4*(ib32/4)) & 0x0f0f0f0f) | (((scales_h[2*(ib32%2)+is] >> 2*(ib32/2)) & 0x03030303) << 4), 0x20202020);
    const int8_t * s8 = (const int8_t *)&scales;
    int2 val1;
    const int * q4 = (const int *)bq5->qs + 16*ib32;
    const int * qh = (const int *)bq5->qh +  4*ib32;
    int aux32[2];
    const uint8_t * aux8 = (const uint8_t *)aux32;
    for (int i = 0; i < 4; ++i) {
        auto values1 = iq5nl_values + (((bq5->extra[i+4*is] >> ib32) & 1) << 5);
        int sumi1 = 0;
        aux32[0] = ((q4[i+4*is+0] >> 0) & 0x0f0f0f0f) | (((qh[i] >> (2*is+0)) & 0x01010101) << 4);
        aux32[1] = ((q4[i+4*is+0] >> 4) & 0x0f0f0f0f) | (((qh[i] >> (2*is+1)) & 0x01010101) << 4);
        val1.x  = int_from_table(aux8+0, (const uint8_t *)values1);
        val1.y  = int_from_table(aux8+4, (const uint8_t *)values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[0], ggml_cuda_dp4a(val1.y, q8[2], sumi1));
        aux32[0] = ((q4[i+4*is+8] >> 0) & 0x0f0f0f0f) | (((qh[i] >> (2*is+4)) & 0x01010101) << 4);
        aux32[1] = ((q4[i+4*is+8] >> 4) & 0x0f0f0f0f) | (((qh[i] >> (2*is+5)) & 0x01010101) << 4);
        val1.x  = int_from_table(aux8+0, (const uint8_t *)values1);
        val1.y  = int_from_table(aux8+4, (const uint8_t *)values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[1], ggml_cuda_dp4a(val1.y, q8[3], sumi1));
        const float d = __half2float(bq5->d[i]) * d8;
        result[i] += d * sumi1 * s8[i];
    }
}

void mul_mat_vec_iq5_k_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ5_K_R4, 2, vec_dot_iq5_k_r4_q8_1, 4>(args, stream);
}

