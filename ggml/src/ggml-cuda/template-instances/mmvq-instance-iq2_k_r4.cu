#include "../iqk_mmvq_templates.cuh"

void mul_mat_vec_iq2_k_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_K_R4, 2, vec_dot_iq2_k_r4_q8_1, 4>(args, stream);
}

