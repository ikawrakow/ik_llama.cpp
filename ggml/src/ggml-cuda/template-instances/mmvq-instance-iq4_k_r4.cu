#include "hip/hip_runtime.h"
#include "../iqk_mmvq_templates.cuh"

__device__ __forceinline__ void vec_dot_iq4_k_r4_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    const block_iq4_k_r4 * bq4 = (const block_iq4_k_r4 *)vbq + kbx;

    // iqs is 0...28 in steps of 2
    const int ib16 = iqs/2;
    const float d8 = __low2float(bq8_1[ib16/2].ds);
    const int32_t  * q8 = (const int *)bq8_1[ib16/2].qs + 4*(ib16%2);

    int ib32 = ib16/2;
    int is   = ib16%2;
    int scales;
    const uint32_t * scales_l = (const uint32_t *)bq4->scales_l;
    const uint32_t * scales_h = (const uint32_t *)bq4->scales_h;
    scales = __vsub4(((scales_l[2*(ib32%4)+is] >> 4*(ib32/4)) & 0x0f0f0f0f) | (((scales_h[2*(ib32%2)+is] >> 2*(ib32/2)) & 0x03030303) << 4), 0x20202020);
    const int8_t * s8 = (const int8_t *)&scales;
    int2 val1;
    const int * q4 = (const int *)bq4->qs + 16*ib32;
    for (int i = 0; i < 4; ++i) {
        auto values1 = iq4k_values + (((bq4->extra[i+4*is] >> ib32) & 1) << 4);
        int sumi1 = 0;
        val1  = get_int_from_table_16(q4[i+4*is+0], values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[0], ggml_cuda_dp4a(val1.y, q8[2], sumi1));
        val1  = get_int_from_table_16(q4[i+4*is+8], values1);
        sumi1 = ggml_cuda_dp4a(val1.x, q8[1], ggml_cuda_dp4a(val1.y, q8[3], sumi1));
        const float d = __half2float(bq4->d[i]) * d8;
        result[i] += d * sumi1 * s8[i];
    }
}

void mul_mat_vec_iq4_k_r4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ4_K_R4, 2, vec_dot_iq4_k_r4_q8_1, 4>(args, stream);
}

