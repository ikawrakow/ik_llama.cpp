#include "hip/hip_runtime.h"
#include "../iqk_mmvq_templates.cuh"

__device__ __forceinline__ void vec_dot_iq2_k_q8_1(
    const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs, float * result) {

    // iqs is 0, 4, 8, 12, 16, 20, 24, 28
    // we have 16 packed quants (when cast to int)

    int i4 = iqs/4;  // 0...7. We will process q8 blocks 4*(i4/4), 4*(i4/4)+1, 4*(i4/4)+2, 4*(i4/4)+3
    const int32_t  * q8_1 = (const int *)bq8_1[4*(i4/4)+0].qs + 2*(i4%4);
    const int32_t  * q8_2 = (const int *)bq8_1[4*(i4/4)+1].qs + 2*(i4%4);
    const int32_t  * q8_3 = (const int *)bq8_1[4*(i4/4)+2].qs + 2*(i4%4);
    const int32_t  * q8_4 = (const int *)bq8_1[4*(i4/4)+3].qs + 2*(i4%4);

    const block_iq2_k * bq2 = (const block_iq2_k *) vbq + kbx;
    const uint32_t * q2 = (const uint32_t *)bq2->qs + 8*(i4/4) + 2*(i4%4);
    const uint16_t extra = bq2->extra >> (8*(i4/4) + (i4%4)/2);

    const uint32_t * scales = (const uint32_t *)bq2->scales;
    uint32_t s32 = __vsub4((scales[i4/4] >> 4*(((i4%4)/2)%2)) & 0x0f0f0f0f, 0x08080808);
    const int8_t * s8 = (const int8_t *)&s32;

    // Block of 16: (32*(4*(i4/4)+k)+8*(i4%4))/16 = 8*(i4/4) + 2*k + (i4%4)/2
    // -> scales_l[4*(i4/4) + k] >> 4*(((i4%4)/2)%2)

#ifdef __CUDA_ARCH__
    uint32_t extra32 = uint32_t(extra & 0xff) * 0x01010101;
    uint32_t extra32_1 = (extra32 << 2) & 0x44444444;
    uint32_t extra32_2 = (extra32 << 0) & 0x44444444;

    uint32_t val1, val2;

    val1 = ((q2[0] >> 0) & 0x33333333) | extra32_1; val2 = ((q2[1] >> 0) & 0x33333333) | extra32_1;
    int2 v1 = get_int_from_table_8(val1, iq2nl_values);
    int2 v2 = get_int_from_table_8(val2, iq2nl_values);
    int sumi1 = ggml_cuda_dp4a(v2.x, q8_1[1], ggml_cuda_dp4a(v1.x, q8_1[0], 0)) * s8[0];
    int sumi3 = ggml_cuda_dp4a(v2.y, q8_3[1], ggml_cuda_dp4a(v1.y, q8_3[0], 0)) * s8[2];

    val1 = ((q2[0] >> 2) & 0x33333333) | extra32_2; val2 = ((q2[1] >> 2) & 0x33333333) | extra32_2;
    v1 = get_int_from_table_8(val1, iq2nl_values);
    v2 = get_int_from_table_8(val2, iq2nl_values);
    int sumi2 = ggml_cuda_dp4a(v2.x, q8_2[1], ggml_cuda_dp4a(v1.x, q8_2[0], 0)) * s8[1];
    int sumi4 = ggml_cuda_dp4a(v2.y, q8_4[1], ggml_cuda_dp4a(v1.y, q8_4[0], 0)) * s8[3];

#else

    const int * all_values = (const int *)iq2k_table;
    const int * values;

    uint32_t val1 = q2[0], val2 = q2[1];

    uint32_t aux32[2];
    int v1, v2;

    aux32[0] = ((val1 >> 0) & 0x03030303); aux32[1] = ((val2 >> 0) & 0x03030303); values = all_values + ((extra & 0x01) << 8);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi1 = ggml_cuda_dp4a(v2, q8_1[1], ggml_cuda_dp4a(v1, q8_1[0], 0)) * s8[0];

    aux32[0] = ((val1 >> 2) & 0x03030303); aux32[1] = ((val2 >> 2) & 0x03030303); values = all_values + ((extra & 0x04) << 6);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi2 = ggml_cuda_dp4a(v2, q8_2[1], ggml_cuda_dp4a(v1, q8_2[0], 0)) * s8[1];

    aux32[0] = ((val1 >> 4) & 0x03030303); aux32[1] = ((val2 >> 4) & 0x03030303); values = all_values + ((extra & 0x10) << 4);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi3 = ggml_cuda_dp4a(v2, q8_3[1], ggml_cuda_dp4a(v1, q8_3[0], 0)) * s8[2];

    aux32[0] = ((val1 >> 6) & 0x03030303); aux32[1] = ((val2 >> 6) & 0x03030303); values = all_values + ((extra & 0x40) << 2);
    v1 = int_from_table_4(aux32[0], values);
    v2 = int_from_table_4(aux32[1], values);
    int sumi4 = ggml_cuda_dp4a(v2, q8_4[1], ggml_cuda_dp4a(v1, q8_4[0], 0)) * s8[3];
#endif

    *result += __half2float(bq2->d) * (__low2float(bq8_1[4*(i4/4)+0].ds) * sumi1
                                    +  __low2float(bq8_1[4*(i4/4)+1].ds) * sumi2
                                    +  __low2float(bq8_1[4*(i4/4)+2].ds) * sumi3
                                    +  __low2float(bq8_1[4*(i4/4)+3].ds) * sumi4);
}

void mul_mat_vec_iq2_k_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    iqk_mul_mat_vec_q_cuda<GGML_TYPE_IQ2_K, VDR_IQ2_K_Q8_1_MMVQ, vec_dot_iq2_k_q8_1>(args, stream);
}

