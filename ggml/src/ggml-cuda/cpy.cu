#include "hip/hip_runtime.h"
#include "cpy.cuh"
#include "convert.cuh"

typedef void (*cpy_kernel_t)(const char * cx, char * cdst);

static __device__ void cpy_1_f32_f32(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    float * dsti = (float *) cdsti;

    *dsti = *xi;
}

static __device__ void cpy_1_f32_f16(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    half * dsti = (half *) cdsti;

    *dsti = __float2half(*xi);
}

static __device__ void cpy_1_f32_bf16(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    hip_bfloat16 * dsti = (hip_bfloat16 *) cdsti;

    *dsti = __float2bfloat16(*xi);
}

static __device__ void cpy_1_f16_f16(const char * cxi, char * cdsti) {
    const half * xi = (const half *) cxi;
    half * dsti = (half *) cdsti;

    *dsti = *xi;
}

static __device__ void cpy_1_f16_f32(const char * cxi, char * cdsti) {
    const half * xi = (const half *) cxi;
    float * dsti = (float *) cdsti;

    *dsti = *xi;
}

template <cpy_kernel_t cpy_1>
static __global__ void cpy_f32_f16(const char * cx, char * cdst, const int ne,
                                   const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
                                   const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11,
                                   const int nb12, const int nb13) {
    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= ne) {
        return;
    }

    // determine indices i03/i13, i02/i12, i01/i11, i00/i10 as a function of index i of flattened tensor
    // then combine those indices with the corresponding byte offsets to get the total offsets
    const int64_t i03 = i/(ne00 * ne01 * ne02);
    const int64_t i02 = (i - i03*ne00*ne01*ne02 )/ (ne00*ne01);
    const int64_t i01 = (i - i03*ne00*ne01*ne02  -  i02*ne01*ne00) / ne00;
    const int64_t i00 = i - i03*ne00*ne01*ne02 - i02*ne01*ne00 - i01*ne00;
    const int64_t x_offset = i00*nb00 + i01*nb01 + i02*nb02 + i03 * nb03;

    const int64_t i13 = i/(ne10 * ne11 * ne12);
    const int64_t i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int64_t i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int64_t i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;
    const int64_t dst_offset = i10*nb10 + i11*nb11 + i12*nb12 + i13 * nb13;

    cpy_1(cx + x_offset, cdst + dst_offset);
}

template <typename dst_t>
static __global__ void k_cpy_q8_0_to_float(const char * cx, dst_t * dst, const int ne,
                                   const int ne00, const int ne01, const int ne02, const int nb01, const int nb02, const int nb03) {
    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;

    if (i >= ne) {
        return;
    }

    const int64_t i03 = i/(ne00 * ne01 * ne02);
    const int64_t i02 = (i - i03*ne00*ne01*ne02) / (ne00*ne01);
    const int64_t i01 = (i - i03*ne00*ne01*ne02 - i02*ne00*ne01) / ne00;
    const int64_t i00 = i - i03*ne00*ne01*ne02 - i02*ne00*ne01 - i01*ne00;

    const block_q8_0 * q8 = (const block_q8_0 *)(cx + i01*nb01 + i02*nb02 + i03*nb03);
    const int ib = i00/QK8_0;
    const int iq = i00%QK8_0;

    if constexpr (std::is_same_v<dst_t, hip_bfloat16>) {
        dst[i00 + i01*ne00 + i02*ne00*ne01 + i03*ne00*ne01*ne02] = __float2bfloat16(__half2float(q8[ib].d)*q8[ib].qs[iq]);
    } else {
        dst[i00 + i01*ne00 + i02*ne00*ne01 + i03*ne00*ne01*ne02] = __half2float(q8[ib].d)*q8[ib].qs[iq];
    }
}

static __global__ void k_transpose_q8_0(const char * cx, char * cdst,
                                   const int ne10, const int ne11, const int ne12,
                                   const int nb01, const int nb02, const int nb03,
                                   const int nb11, const int nb12, const int nb13) {
    const int64_t i = blockDim.x*blockIdx.x + threadIdx.x;

    const int64_t i13 = i/(ne10 * ne11 * ne12);
    const int64_t i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int64_t i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int64_t i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;

    //const int64_t ne00 = ne11;
    //const int64_t ne01 = ne10;
    //const int64_t ne02 = ne12;
    const int64_t i03 = i13;
    const int64_t i02 = i12;
    const int64_t i01 = i10; //(i - i03*ne00*ne01*ne02 - i02*ne00*ne01) / ne00;
    const int64_t i00 = i11; //i - i03*ne00*ne01*ne02 - i02*ne00*ne01 - i01*ne00;

    const block_q8_0 * q8 = (const block_q8_0 *)(cx + i01*nb01 + i02*nb02 + i03*nb03);
    const int ib0 = i00/QK8_0;
    const int iq0 = i00%QK8_0;

    float xi = __half2float(q8[ib0].d)*q8[ib0].qs[iq0];
    float amax = fabsf(xi);
    amax = warp_reduce_max(amax);

    //printf("%d, %d, %d: i = %ld, i11 = %ld i10 = %ld, xi = %g, amax = %g\n", blockDim.x, blockIdx.x, threadIdx.x, i, i11, i10, xi, amax);

    float d = amax/127;
    int8_t q = amax == 0.0f ? 0 : roundf(xi / d);

    block_q8_0 * dst = (block_q8_0 *)(cdst + i11*nb11 + i12*nb12 + i13*nb13);
    dst[i10 / QK8_0].qs[i10 % QK8_0] = q;

    if (threadIdx.x == 0) {
        dst[i10 / QK8_0].d = __float2half(d);
    }
}

static __device__ void cpy_blck_f32_q8_0(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q8_0 * dsti = (block_q8_0 *) cdsti;

    float amax = 0.0f; // absolute max

    for (int j = 0; j < QK8_0; j++) {
        const float v = xi[j];
        amax = fmaxf(amax, fabsf(v));
    }

    const float d = amax / ((1 << 7) - 1);
    const float id = d ? 1.0f/d : 0.0f;

    dsti->d = d;

    for (int j = 0; j < QK8_0; ++j) {
        const float x0 = xi[j]*id;

        dsti->qs[j] = roundf(x0);
    }
}

static __device__ void cpy_blck_f32_q4_0(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q4_0 * dsti = (block_q4_0 *) cdsti;

    float amax = 0.0f;
    float vmax = 0.0f;

    for (int j = 0; j < QK4_0; ++j) {
        const float v = xi[j];
        if (amax < fabsf(v)) {
            amax = fabsf(v);
            vmax = v;
        }
    }

    const float d  = vmax / -8;
    const float id = d ? 1.0f/d : 0.0f;

    dsti->d = d;

    for (int j = 0; j < QK4_0/2; ++j) {
        const float x0 = xi[0       + j]*id;
        const float x1 = xi[QK4_0/2 + j]*id;

        const uint8_t xi0 = min(15, (int8_t)(x0 + 8.5f));
        const uint8_t xi1 = min(15, (int8_t)(x1 + 8.5f));

        dsti->qs[j]  = xi0;
        dsti->qs[j] |= xi1 << 4;
    }
}

static __device__ void cpy_blck_f32_q4_1(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q4_1 * dsti = (block_q4_1 *) cdsti;

    float vmin = FLT_MAX;
    float vmax = -FLT_MAX;

    for (int j = 0; j < QK4_1; ++j) {
        const float v = xi[j];

        if (v < vmin) vmin = v;
        if (v > vmax) vmax = v;
    }

    const float d  = (vmax - vmin) / ((1 << 4) - 1);
    const float id = d ? 1.0f/d : 0.0f;

    dsti->dm.x = d;
    dsti->dm.y = vmin;

    for (int j = 0; j < QK4_1/2; ++j) {
        const float x0 = (xi[0       + j] - vmin)*id;
        const float x1 = (xi[QK4_1/2 + j] - vmin)*id;

        const uint8_t xi0 = min(15, (int8_t)(x0 + 0.5f));
        const uint8_t xi1 = min(15, (int8_t)(x1 + 0.5f));

        dsti->qs[j]  = xi0;
        dsti->qs[j] |= xi1 << 4;
    }
}

static __device__ void cpy_blck_f32_q5_0(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q5_0 * dsti = (block_q5_0 *) cdsti;

    float amax = 0.0f;
    float vmax = 0.0f;

    for (int j = 0; j < QK5_0; ++j) {
        const float v = xi[j];
        if (amax < fabsf(v)) {
            amax = fabsf(v);
            vmax = v;
        }
    }

    const float d  = vmax / -16;
    const float id = d ? 1.0f/d : 0.0f;

    dsti->d = d;

    uint32_t qh = 0;
    for (int j = 0; j < QK5_0/2; ++j) {
        const float x0 = xi[0       + j]*id;
        const float x1 = xi[QK5_0/2 + j]*id;

        const uint8_t xi0 = min(31, (int8_t)(x0 + 16.5f));
        const uint8_t xi1 = min(31, (int8_t)(x1 + 16.5f));

        dsti->qs[j]  = (xi0 & 0xf) | ((xi1 & 0xf) << 4);
        qh |= ((xi0 & 0x10u) >> 4) << (j + 0);
        qh |= ((xi1 & 0x10u) >> 4) << (j + QK5_0/2);
    }
    memcpy(dsti->qh, &qh, sizeof(qh));
}

static __device__ void cpy_blck_f32_q5_1(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q5_1 * dsti = (block_q5_1 *) cdsti;

    float min = xi[0];
    float max = xi[0];

    for (int j = 1; j < QK5_1; ++j) {
        const float v = xi[j];
        min = v < min ? v : min;
        max = v > max ? v : max;
    }

    const float d  = (max - min) / 31;
    const float id = d ? 1.0f/d : 0.0f;

    dsti->dm.x = d;
    dsti->dm.y = min;

    uint32_t qh = 0;
    for (int j = 0; j < QK5_1/2; ++j) {
        const float x0 = (xi[0       + j] - min)*id;
        const float x1 = (xi[QK5_1/2 + j] - min)*id;

        const uint8_t xi0 = (uint8_t)(x0 + 0.5f);
        const uint8_t xi1 = (uint8_t)(x1 + 0.5f);

        dsti->qs[j]  = (xi0 & 0xf) | ((xi1 & 0xf) << 4);
        qh |= ((xi0 & 0x10u) >> 4) << (j + 0);
        qh |= ((xi1 & 0x10u) >> 4) << (j + QK5_1/2);
    }
    memcpy(dsti->qh, &qh, sizeof(qh));
}

static __device__ void cpy_blck_f32_q6_0(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_q6_0 * dsti = (block_q6_0 *) cdsti;

    float amax = 0.0f;
    float vmax = 0.0f;

    for (int j = 0; j < QK6_0; ++j) {
        const float v  = xi[j];
        const float av = fabsf(xi[j]);
        if (amax < av) {
            amax = av;
            vmax = v;
        }
    }

    const float d  = vmax / -32;
    const float id = d ? 1.0f/d : 0.0f;

    dsti->d = d;
    memset(dsti->qh, 0, QK6_0/4);

    for (int j = 0; j < QK6_0/2; ++j) {
        const float x0 = xi[0       + j]*id;
        const float x1 = xi[QK4_0/2 + j]*id;

        const uint8_t xi0 = min(63, (int8_t)(x0 + 32.5f));
        const uint8_t xi1 = min(63, (int8_t)(x1 + 32.5f));

        dsti->qs[j]  = (xi0 & 0xf) | ((xi1 & 0xf) << 4);
        const uint8_t h = (xi0 >> 4) | ((xi1 >> 4) << 2);
        dsti->qh[j%(QK6_0/4)] |= (h << 4*(j/(QK6_0/4)));
    }
}

static __device__ const int8_t iq4nl_index[241] = {
     0,  0,  0,  0,  0,  0,  0,  0,  0,  0,  0, 16, 16,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,  1,
     1, 17, 17,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2,  2, 18,  3,  3,  3,  3,  3,  3,  3,  3,  3,  3,
     3,  3,  3,  3,  3,  3, 19,  4,  4,  4,  4,  4,  4,  4,  4,  4,  4,  4,  4,  4,  4, 20,  5,  5,  5,  5,  5,  5,  5,  5,  5,  5,
     5,  5, 21, 21,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6,  6, 22,  7,  7,  7,  7,  7,  7,  7,  7,  7,  7, 23, 23,  8,  8,  8,  8,
     8,  8,  8,  8,  8,  8, 24,  9,  9,  9,  9,  9,  9,  9,  9,  9,  9,  9, 25, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 10, 26, 26,
    11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 11, 27, 27, 12, 12, 12, 12, 12, 12, 12, 12, 12, 12, 12, 12, 12, 12, 28, 13, 13, 13,
    13, 13, 13, 13, 13, 13, 13, 13, 13, 13, 13, 13, 13, 13, 29, 14, 14, 14, 14, 14, 14, 14, 14, 14, 14, 14, 14, 14, 14, 14, 14, 14,
    14, 14, 14, 14, 30, 15, 15, 15, 15, 15, 15, 15, 15, 15, 15, 15, 15
};
static __device__ __forceinline__ int best_index_iq4nl(const int8_t * values, float x) {
    int ix = (int)x - values[0];
    if (ix < 0 || ix >= 241) return ix < 0 ? 0 : 15;
    ix = iq4nl_index[ix];
    return ix < 16 ? ix : x - values[ix-16] < values[ix-15] - x ? ix-16 : ix-15;
}

static __device__ void cpy_blck_f32_iq4_nl(const char * cxi, char * cdsti) {
    const float * xi = (const float *) cxi;
    block_iq4_nl * dsti = (block_iq4_nl *) cdsti;

    float amax = 0.0f;
    float vmax = 0.0f;

    for (int j = 0; j < QK4_NL; ++j) {
        const float v = xi[j];
        if (amax < fabsf(v)) {
            amax = fabsf(v);
            vmax = v;
        }
    }

    float d = vmax / kvalues_iq4nl[0];
    const float id = d ? 1.0f/d : 0.0f;

    //dsti->d = d;

    float sumqx = 0, sumq2 = 0;
    for (int j = 0; j < QK4_NL/2; ++j) {
        const float x0 = xi[0        + j]*id;
        const float x1 = xi[QK4_NL/2 + j]*id;
        const uint8_t xi0 = best_index_iq4nl(kvalues_iq4nl, x0);
        const uint8_t xi1 = best_index_iq4nl(kvalues_iq4nl, x1);
        dsti->qs[j] = xi0 | (xi1 << 4);
        const float v0 = kvalues_iq4nl[xi0];
        const float v1 = kvalues_iq4nl[xi1];
        const float w0 = xi[0        + j]*xi[0        + j];
        const float w1 = xi[QK4_NL/2 + j]*xi[QK4_NL/2 + j];
        sumqx += w0*v0*xi[j] + w1*v1*xi[QK4_NL/2 + j];
        sumq2 += w0*v0*v0 + w1*v1*v1;
    }

    dsti->d = sumq2 > 0 ? sumqx/sumq2 : d;
}

template <cpy_kernel_t cpy_blck, int qk>
static __global__ void cpy_f32_q(const char * cx, char * cdst, const int ne,
                                 const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
                                 const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11,
                                 const int nb12, const int nb13) {
    const int i = (blockDim.x*blockIdx.x + threadIdx.x)*qk;

    if (i >= ne) {
        return;
    }

    const int i03 = i/(ne00 * ne01 * ne02);
    const int i02 = (i - i03*ne00*ne01*ne02 )/ (ne00*ne01);
    const int i01 = (i - i03*ne00*ne01*ne02  -  i02*ne01*ne00) / ne00;
    const int i00 = i - i03*ne00*ne01*ne02 - i02*ne01*ne00 - i01*ne00;
    const int x_offset = i00*nb00 + i01*nb01 + i02*nb02 + i03 * nb03;

    const int i13 = i/(ne10 * ne11 * ne12);
    const int i12 = (i - i13*ne10*ne11*ne12) / (ne10*ne11);
    const int i11 = (i - i13*ne10*ne11*ne12 - i12*ne10*ne11) / ne10;
    const int i10 = i - i13*ne10*ne11*ne12 - i12*ne10*ne11 - i11*ne10;
    const int dst_offset = (i10/qk)*nb10 + i11*nb11 + i12*nb12 + i13*nb13;

    cpy_blck(cx + x_offset, cdst + dst_offset);
}

static void ggml_cpy_f16_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f16_f32><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_f32_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_f32><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_f16_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_f16><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_bf16_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f32_bf16><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q8_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK8_0 == 0);
    const int num_blocks = ne / QK8_0;
    cpy_f32_q<cpy_blck_f32_q8_0, QK8_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q4_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK4_0 == 0);
    const int num_blocks = ne / QK4_0;
    cpy_f32_q<cpy_blck_f32_q4_0, QK4_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q4_1_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK4_1 == 0);
    const int num_blocks = ne / QK4_1;
    cpy_f32_q<cpy_blck_f32_q4_1, QK4_1><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q5_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK5_0 == 0);
    const int num_blocks = ne / QK5_0;
    cpy_f32_q<cpy_blck_f32_q5_0, QK5_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q5_1_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK5_1 == 0);
    const int num_blocks = ne / QK5_1;
    cpy_f32_q<cpy_blck_f32_q5_1, QK5_1><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_q6_0_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK6_0 == 0);
    const int num_blocks = ne / QK6_0;
    cpy_f32_q<cpy_blck_f32_q6_0, QK6_0><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f32_iq4_nl_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    GGML_ASSERT(ne % QK4_NL == 0);
    const int num_blocks = ne / QK4_NL;
    cpy_f32_q<cpy_blck_f32_iq4_nl, QK4_NL><<<num_blocks, 1, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void ggml_cpy_f16_f16_cuda(
    const char * cx, char * cdst, const int ne,
    const int ne00, const int ne01, const int ne02, const int nb00, const int nb01, const int nb02,
    const int nb03, const int ne10, const int ne11, const int ne12, const int nb10, const int nb11, const int nb12, const int nb13, hipStream_t stream) {

    const int num_blocks = (ne + CUDA_CPY_BLOCK_SIZE - 1) / CUDA_CPY_BLOCK_SIZE;
    cpy_f32_f16<cpy_1_f16_f16><<<num_blocks, CUDA_CPY_BLOCK_SIZE, 0, stream>>>
        (cx, cdst, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13);
}

static void transpose_q8_0(ggml_backend_cuda_context & ctx, const ggml_tensor * src, ggml_tensor * dst) {
    auto stream = ctx.stream();
    auto num_blocks = ggml_nelements(dst)/QK8_0;
    k_transpose_q8_0<<<num_blocks, QK8_0, 0, stream>>>(
            (const char *)src->data, (char *)dst->data,
            dst->ne[0], dst->ne[1], dst->ne[2], src->nb[0], src->nb[2], src->nb[3],
            dst->nb[1], dst->nb[2], dst->nb[3]);
}

static void copy_q8_0_to_float(ggml_backend_cuda_context & ctx, const ggml_tensor * src, ggml_tensor * dst) {
    auto stream = ctx.stream();
    auto num_blocks = ggml_nelements(dst)/QK8_0;
    if (dst->type == GGML_TYPE_F16) {
        k_cpy_q8_0_to_float<<<num_blocks, QK8_0, 0, stream>>>((const char *)src->data, (half *)dst->data, ggml_nelements(dst),
                src->ne[0], src->ne[1], src->ne[2], src->nb[1], src->nb[2], src->nb[3]);
    }
    else if (dst->type == GGML_TYPE_F32) {
        k_cpy_q8_0_to_float<<<num_blocks, QK8_0, 0, stream>>>((const char *)src->data, (float *)dst->data, ggml_nelements(dst),
                src->ne[0], src->ne[1], src->ne[2], src->nb[1], src->nb[2], src->nb[3]);
    }
    else if (dst->type == GGML_TYPE_BF16) {
        k_cpy_q8_0_to_float<<<num_blocks, QK8_0, 0, stream>>>((const char *)src->data, (hip_bfloat16 *)dst->data, ggml_nelements(dst),
                src->ne[0], src->ne[1], src->ne[2], src->nb[1], src->nb[2], src->nb[3]);
    }
    else {
        GGML_ABORT("fatal error");
    }
}

void ggml_cuda_cpy(ggml_backend_cuda_context & ctx, const ggml_tensor * src0, ggml_tensor * src1) {
    const int64_t ne = ggml_nelements(src0);
    GGML_ASSERT(ne == ggml_nelements(src1));

    GGML_ASSERT(ggml_nbytes(src0) <= INT_MAX);
    GGML_ASSERT(ggml_nbytes(src1) <= INT_MAX);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne01 = src0->ne[1];
    const int64_t ne02 = src0->ne[2];

    //GGML_ASSERT(src0->ne[3] == 1);

    const int64_t nb00 = src0->nb[0];
    const int64_t nb01 = src0->nb[1];
    const int64_t nb02 = src0->nb[2];
    const int64_t nb03 = src0->nb[3];

    const int64_t ne10 = src1->ne[0];
    const int64_t ne11 = src1->ne[1];
    const int64_t ne12 = src1->ne[2];

    //GGML_ASSERT(src1->ne[3] == 1);

    const int64_t nb10 = src1->nb[0];
    const int64_t nb11 = src1->nb[1];
    const int64_t nb12 = src1->nb[2];
    const int64_t nb13 = src1->nb[3];

    hipStream_t main_stream = ctx.stream();

    char * src0_ddc = (char *) src0->data;
    char * src1_ddc = (char *) src1->data;

    if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_f32_f32_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_f32_f16_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_BF16) {
        ggml_cpy_f32_bf16_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q8_0) {
        ggml_cpy_f32_q8_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_0) {
        ggml_cpy_f32_q4_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_1) {
        ggml_cpy_f32_q4_1_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_0) {
        ggml_cpy_f32_q5_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q6_0) {
        ggml_cpy_f32_q6_0_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_IQ4_NL) {
        ggml_cpy_f32_iq4_nl_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_1) {
        ggml_cpy_f32_q5_1_cuda(src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16) {
        ggml_cpy_f16_f16_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_BF16 && src1->type == GGML_TYPE_BF16) {
        ggml_cpy_f16_f16_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32) {
        ggml_cpy_f16_f32_cuda (src0_ddc, src1_ddc, ne, ne00, ne01, ne02, nb00, nb01, nb02, nb03, ne10, ne11, ne12, nb10, nb11, nb12, nb13, main_stream);
    } else if (ggml_are_same_shape(src0, src1) && src0->type == GGML_TYPE_Q8_0 &&
            (src1->type == GGML_TYPE_F16 || src1->type == GGML_TYPE_BF16 || src1->type == GGML_TYPE_F32)) {
        copy_q8_0_to_float(ctx, src0, src1);
    } else if (ggml_is_contiguous(src0) && ggml_are_same_shape(src0, src1)) {
        if (src1->type == GGML_TYPE_F16) {
            auto to_fp16 = ggml_get_to_fp16_cuda(src0->type);
            if (to_fp16) {
                to_fp16(src0->data, (half *)src1->data, ggml_nrows(src0), src0->ne[0], main_stream);
            }
        }
        else if (src1->type == GGML_TYPE_F32) {
            auto to_fp32 = ggml_get_to_fp32_cuda(src0->type);
            if (to_fp32) {
                to_fp32(src0->data, (float *)src1->data, ggml_nrows(src0), src0->ne[0], main_stream);
            }
        }
        else if (src1->type == GGML_TYPE_BF16) {
            auto to_bf16 = ggml_get_to_bf16_cuda(src0->type);
            if (to_bf16) {
                to_bf16(src0->data, (hip_bfloat16 *)src1->data, ggml_nrows(src0), src0->ne[0], main_stream);
            }
        }
    } else if (ggml_are_same_shape(src0, src1) && src0->type == GGML_TYPE_Q8_0 && src1->type == GGML_TYPE_Q8_0) {
        transpose_q8_0(ctx, src0, src1);
    } else {
        fprintf(stderr, "%s: unsupported type combination (%s to %s)\n", __func__,
                ggml_type_name(src0->type), ggml_type_name(src1->type));
        fprintf(stderr, "%s: %ld x %ld x %ld; %zu x %zu %zu -> %ld x %ld x %ld; %zu x %zu x %zu\n", __func__,
                src0->ne[0], src0->ne[1], src0->ne[2], src0->nb[1], src0->nb[2], src0->nb[3],
                src1->ne[0], src1->ne[1], src1->ne[2], src1->nb[1], src1->nb[2], src1->nb[3]);
        GGML_ABORT("fatal error");
    }
}

void ggml_cuda_dup(ggml_backend_cuda_context & ctx, ggml_tensor * dst) {
    const ggml_tensor * src0 = dst->src[0];
    ggml_cuda_cpy(ctx, src0, dst);
}

void* ggml_cuda_cpy_fn(const ggml_tensor * src0, ggml_tensor * src1) {
    if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F32) {
            return (void*) cpy_f32_f16<cpy_1_f32_f32>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_F16) {
            return (void*) cpy_f32_f16<cpy_1_f32_f16>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_BF16) {
            return (void*) cpy_f32_f16<cpy_1_f32_bf16>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q8_0) {
            return (void*) cpy_f32_q<cpy_blck_f32_q8_0, QK8_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_0) {
            return (void*) cpy_f32_q<cpy_blck_f32_q4_0, QK4_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q4_1) {
            return (void*) cpy_f32_q<cpy_blck_f32_q4_1, QK4_1>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_0) {
            return (void*) cpy_f32_q<cpy_blck_f32_q5_0, QK5_0>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_IQ4_NL) {
            return (void*) cpy_f32_q<cpy_blck_f32_iq4_nl, QK4_NL>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q5_1) {
            return (void*) cpy_f32_q<cpy_blck_f32_q5_1, QK5_1>;
    } else if (src0->type == GGML_TYPE_F32 && src1->type == GGML_TYPE_Q6_0) {
            return (void*) cpy_f32_q<cpy_blck_f32_q6_0, QK6_0>;
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F16) {
            return (void*) cpy_f32_f16<cpy_1_f16_f16>;
    } else if (src0->type == GGML_TYPE_F16 && src1->type == GGML_TYPE_F32) {
            return (void*) cpy_f32_f16<cpy_1_f16_f32>;
    } else if (ggml_are_same_shape(src0, src1) && src0->type == GGML_TYPE_Q8_0 &&
            (src1->type == GGML_TYPE_F16 || src1->type == GGML_TYPE_BF16 || src1->type == GGML_TYPE_F32)) {
        return (void*)copy_q8_0_to_float;
    } else if (ggml_is_contiguous(src0) && ggml_are_same_shape(src0, src1)) {
        if (src1->type == GGML_TYPE_F16) {
            auto to_fp16 = ggml_get_to_fp16_cuda(src0->type);
            if (to_fp16) return (void*)to_fp16;
        }
        else if (src1->type == GGML_TYPE_F32) {
            auto to_fp32 = ggml_get_to_fp32_cuda(src0->type);
            if (to_fp32) return (void*)to_fp32;
        }
        else if (src1->type == GGML_TYPE_BF16) {
            auto to_bf16 = ggml_get_to_bf16_cuda(src0->type);
            if (to_bf16) return (void*)to_bf16;
        }
    }
    else if (ggml_are_same_shape(src0, src1) && src0->type == GGML_TYPE_Q8_0 && src1->type == GGML_TYPE_Q8_0) {
        return (void *)transpose_q8_0;
    }
    fprintf(stderr, "%s: unsupported type combination (%s to %s)\n", __func__,
            ggml_type_name(src0->type), ggml_type_name(src1->type));
    fprintf(stderr, "%s: %ld x %ld x %ld; %zu x %zu %zu -> %ld x %ld x %ld; %zu x %zu x %zu\n", __func__,
                src0->ne[0], src0->ne[1], src0->ne[2], src0->nb[1], src0->nb[2], src0->nb[3],
                src1->ne[0], src1->ne[1], src1->ne[2], src1->nb[1], src1->nb[2], src1->nb[3]);
    GGML_ABORT("fatal error");
}
