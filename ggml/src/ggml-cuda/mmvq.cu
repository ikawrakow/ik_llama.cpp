#include "hip/hip_runtime.h"
//
// Copyright (C) 2023-2024 The ggml authors
// Copyright (C) 2024 Iwan Kawrakow
// MIT license
// SPDX-License-Identifier: MIT
//

#include "mmvq.cuh"
#include "iqk_mmvq.cuh"
#include "vecdotq.cuh"
#include "mmvq-args.h"

typedef float (*vec_dot_q_cuda_t)(const void * __restrict__ vbq, const block_q8_1 * __restrict__ bq8_1, const int & kbx, const int & iqs);

static constexpr __device__ vec_dot_q_cuda_t get_vec_dot_q_cuda(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0   : return vec_dot_q4_0_q8_1;
        case GGML_TYPE_Q4_1   : return vec_dot_q4_1_q8_1;
        case GGML_TYPE_Q5_0   : return vec_dot_q5_0_q8_1;
        case GGML_TYPE_Q5_1   : return vec_dot_q5_1_q8_1;
        case GGML_TYPE_Q6_0   : return vec_dot_q6_0_q8_1;
        case GGML_TYPE_Q8_0   : return vec_dot_q8_0_q8_1;
        case GGML_TYPE_Q2_K   : return vec_dot_q2_K_q8_1;
        case GGML_TYPE_Q3_K   : return vec_dot_q3_K_q8_1;
        case GGML_TYPE_Q4_K   : return vec_dot_q4_K_q8_1;
        case GGML_TYPE_Q5_K   : return vec_dot_q5_K_q8_1;
        case GGML_TYPE_Q6_K   : return vec_dot_q6_K_q8_1;
        case GGML_TYPE_IQ2_XXS: return vec_dot_iq2_xxs_q8_1;
        case GGML_TYPE_IQ2_XS : return vec_dot_iq2_xs_q8_1;
        case GGML_TYPE_IQ2_S  : return vec_dot_iq2_s_q8_1;
        case GGML_TYPE_IQ3_XXS: return vec_dot_iq3_xxs_q8_1;
        case GGML_TYPE_IQ1_S  : return vec_dot_iq1_s_q8_1;
        case GGML_TYPE_IQ1_M  : return vec_dot_iq1_m_q8_1;
        case GGML_TYPE_IQ4_NL : return vec_dot_iq4_nl_q8_1;
        case GGML_TYPE_MXFP4  : return vec_dot_mxfp4_q8_1;
        case GGML_TYPE_IQ4_XS : return vec_dot_iq4_xs_q8_1;
        case GGML_TYPE_IQ3_S  : return vec_dot_iq3_s_q8_1;
        default               : return nullptr;
    }
}

static constexpr __device__ int get_vdr_mmvq(ggml_type type) {
    switch (type) {
        case GGML_TYPE_Q4_0    : return VDR_Q4_0_Q8_1_MMVQ;
        case GGML_TYPE_Q4_1    : return VDR_Q4_1_Q8_1_MMVQ;
        case GGML_TYPE_Q5_0    : return VDR_Q5_0_Q8_1_MMVQ;
        case GGML_TYPE_Q5_1    : return VDR_Q5_1_Q8_1_MMVQ;
        case GGML_TYPE_Q6_0    : return VDR_Q6_0_Q8_1_MMVQ;
        case GGML_TYPE_Q8_0    : return VDR_Q8_0_Q8_1_MMVQ;
        case GGML_TYPE_Q2_K    : return VDR_Q2_K_Q8_1_MMVQ;
        case GGML_TYPE_Q3_K    : return VDR_Q3_K_Q8_1_MMVQ;
        case GGML_TYPE_Q4_K    : return VDR_Q4_K_Q8_1_MMVQ;
        case GGML_TYPE_Q5_K    : return VDR_Q5_K_Q8_1_MMVQ;
        case GGML_TYPE_Q6_K    : return VDR_Q6_K_Q8_1_MMVQ;
        case GGML_TYPE_IQ2_XXS : return VDR_IQ2_XXS_Q8_1_MMVQ;
        case GGML_TYPE_IQ2_XS  : return VDR_IQ2_XS_Q8_1_MMVQ;
        case GGML_TYPE_IQ2_S   : return VDR_IQ2_S_Q8_1_MMVQ;
        case GGML_TYPE_IQ3_XXS : return VDR_IQ3_XXS_Q8_1_MMVQ;
        case GGML_TYPE_IQ3_S   : return VDR_IQ3_S_Q8_1_MMVQ;
        case GGML_TYPE_IQ4_NL  : return VDR_IQ4_NL_Q8_1_MMVQ;
        case GGML_TYPE_MXFP4   : return VDR_MXFP4_Q8_1_MMVQ;
        case GGML_TYPE_IQ4_XS  : return VDR_IQ4_XS_Q8_1_MMVQ;
        default                : return 1;
    }
}

template <ggml_type type, int ncols_y, int nwarps>
static __device__ void mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy,
    const float * bias, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;
    constexpr int vdr = get_vdr_mmvq(type);

    constexpr vec_dot_q_cuda_t vec_dot_q_cuda = get_vec_dot_q_cuda(type);

    //int64_t rows_per_cuda_block = ggml_cuda_info().devices[id].cc < CC_RDNA2 ?
    //    ncols_y < 4 ? 1 : 2 : 1;

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int rows_per_cuda_block = 1;
#else
    constexpr int rows_per_cuda_block = ncols_y < 4 ? 1 : 2;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp[j][i] += vec_dot_q_cuda(vx, &y[j*blocks_per_col_y + kby], (row0 + i)*blocks_per_row_x + kbx, kqs);
            }
        }
    }

    __shared__ float tmp_shared[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared[threadIdx.y-1][j][i][threadIdx.x] = tmp[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp[j][i] += tmp_shared[l][j][i][threadIdx.x];
            }
            tmp[j][i] = warp_reduce_sum(tmp[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + threadIdx.x < nrows_dst)) {
            dst[j*nrows_dst + row0 + threadIdx.x] = bias ? tmp[j][threadIdx.x] + bias[j*nrows_dst + row0 + threadIdx.x] : tmp[j][threadIdx.x];
        }
    }
}

template <ggml_type type, int ncols_y, int nwarps>
static __device__ void fused_mul_mat_vec_q(
    const void * __restrict__ vup, const void * __restrict__ vgate,
    const float * __restrict__ bias_u, const float * __restrict__ bias_g,
    const void * __restrict__ vy, float * __restrict__ dst,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst, ggml_unary_op unary_op) {

    constexpr int qk  = ggml_cuda_type_traits<type>::qk;
    constexpr int qi  = ggml_cuda_type_traits<type>::qi;
    constexpr int vdr = get_vdr_mmvq(type);

    constexpr vec_dot_q_cuda_t vec_dot_q_cuda = get_vec_dot_q_cuda(type);

    //int64_t rows_per_cuda_block = ggml_cuda_info().devices[id].cc < CC_RDNA2 ?
    //    ncols_y < 4 ? 1 : 2 : 1;

#if defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && (defined(RDNA2) || defined(RDNA3))
    constexpr int rows_per_cuda_block = 1;
#else
    constexpr int rows_per_cuda_block = ncols_y < 4 ? 1 : 2;
#endif // defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__) && !defined(RDNA2) && !defined(RDNA3)

    const     int tid = WARP_SIZE*threadIdx.y + threadIdx.x;
    const     int row0 = rows_per_cuda_block*blockIdx.x;
    const     int blocks_per_row_x = ncols_x / qk;
    const     int blocks_per_col_y = nrows_y / QK8_1;
    constexpr int blocks_per_iter = vdr * nwarps*WARP_SIZE / qi;

// partial sum for each thread
    float tmp_u[ncols_y][rows_per_cuda_block] = {0.0f};
    float tmp_g[ncols_y][rows_per_cuda_block] = {0.0f};

    const block_q8_1 * y = (const block_q8_1 *) vy;

    for (int kbx = tid / (qi/vdr); kbx < blocks_per_row_x; kbx += blocks_per_iter) {
        const int kby = kbx * (qk/QK8_1); // y block index that aligns with kbx

        // x block quant index when casting the quants to int
        const int kqs = vdr * (tid % (qi/vdr));

#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_u[j][i] += vec_dot_q_cuda(vup  , &y[j*blocks_per_col_y + kby], (row0 + i)*blocks_per_row_x + kbx, kqs);
                tmp_g[j][i] += vec_dot_q_cuda(vgate, &y[j*blocks_per_col_y + kby], (row0 + i)*blocks_per_row_x + kbx, kqs);
            }
        }
    }

    __shared__ float tmp_shared_u[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    __shared__ float tmp_shared_g[nwarps-1 > 0 ? nwarps-1 : 1][ncols_y][rows_per_cuda_block][WARP_SIZE];
    if (threadIdx.y > 0) {
#pragma unroll
        for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
            for (int i = 0; i < rows_per_cuda_block; ++i) {
                tmp_shared_u[threadIdx.y-1][j][i][threadIdx.x] = tmp_u[j][i];
                tmp_shared_g[threadIdx.y-1][j][i][threadIdx.x] = tmp_g[j][i];
            }
        }
    }
    __syncthreads();
    if (threadIdx.y > 0) {
        return;
    }

    // sum up partial sums and write back result
#pragma unroll
    for (int j = 0; j < ncols_y; ++j) {
#pragma unroll
        for (int i = 0; i < rows_per_cuda_block; ++i) {
#pragma unroll
            for (int l = 0; l < nwarps-1; ++l) {
                tmp_u[j][i] += tmp_shared_u[l][j][i][threadIdx.x];
                tmp_g[j][i] += tmp_shared_g[l][j][i][threadIdx.x];
            }
            tmp_u[j][i] = warp_reduce_sum(tmp_u[j][i]);
            tmp_g[j][i] = warp_reduce_sum(tmp_g[j][i]);
        }

        if (threadIdx.x < rows_per_cuda_block && (rows_per_cuda_block == 1 || row0 + threadIdx.x < nrows_dst)) {
            float u = tmp_u[j][threadIdx.x];
            float g = tmp_g[j][threadIdx.x];
            float r;
            switch (unary_op) {
                case GGML_UNARY_OP_SILU: r = u*g/(1 + expf(-g)); break;
                case GGML_UNARY_OP_RELU: r = fmaxf(g, 0.0f) * u; break;
                case GGML_UNARY_OP_GELU: {
                    constexpr float GELU_COEF_A    = 0.044715f;
                    constexpr float SQRT_2_OVER_PI = 0.79788456080286535587989211986876f;
                    r = 0.5f*g*u*(1.0f + tanhf(SQRT_2_OVER_PI*g*(1.0f + GELU_COEF_A*g*g)));
                } break;
                // we assume that the supported ops have been checked by the caller
                default: {
                  constexpr float alpha = 1.702f;
                  constexpr float limit = 7.0f;
                  g += bias_g[j*nrows_dst + row0 + threadIdx.x];
                  u += bias_u[j*nrows_dst + row0 + threadIdx.x];
                  g = fminf(g, limit);
                  u = fmaxf(fminf(u, limit), -limit);
                  r = g / (1.0f + expf(-g * alpha)) * (1.0f + u);
                } break;
            }
            dst[j*nrows_dst + row0 + threadIdx.x] = r;
        }
    }
}

template <ggml_type type, int ncols_y, int nwarps>
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
// tell the compiler to use as many registers as it wants, see nwarps definition below
__launch_bounds__(nwarps*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void mul_mat_vec_q(
    const void * __restrict__ vx, const void * __restrict__ vy, float * __restrict__ dst,
    const char * __restrict__ ids_data, const void * __restrict__ bias,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst,
    const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, const int64_t ids_nb0, const int64_t bias_nb1) {

    int i2 = blockIdx.y;
    char * cdst = (char *)dst + i2*nb2;
    int i02 = ids_data ? *(const int *)(ids_data + i2*ids_nb0) : i2;
    if (i02 < 0) {
        return;
    }
    const char * cx = (const char *)vx + i02*nb02;
    const char * cy = (const char *)vy + i2*nb12;
    const float * b = (const float *)(bias ? ids_data ? (const char *)bias + i02*bias_nb1 : bias : nullptr);
    mul_mat_vec_q<type, ncols_y, nwarps>(cx, cy, b, (float *)cdst, ncols_x, nrows_x, nrows_y, nrows_dst);
}

template <ggml_type type, int ncols_y, int nwarps>
#if !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
// tell the compiler to use as many registers as it wants, see nwarps definition below
__launch_bounds__(nwarps*WARP_SIZE, 1)
#endif // !(defined(GGML_USE_HIPBLAS) && defined(__HIP_PLATFORM_AMD__))
static __global__ void fused_mul_mat_vec_q(
    const void * __restrict__ vup, const void * __restrict__ vgate,
    const void * __restrict__ vy, float * __restrict__ dst, const char * __restrict__ ids_data,
    const void * __restrict__ bias_u, const void * __restrict__ bias_g, const uint64_t bias_nb1,
    const int ncols_x, const int nrows_x, const int nrows_y, const int nrows_dst,
    const uint64_t nb02, const uint64_t nb12, const uint64_t nb2, const int64_t ids_nb0, ggml_unary_op unary_op) {

    int i2 = blockIdx.y;
    char * cdst = (char *)dst + i2*nb2;
    int i02 = ids_data ? *(const int *)(ids_data + i2*ids_nb0) : i2;
    if (i02 < 0) {
        return;
    }
    const char * cx_u = (const char *)vup   + i02*nb02;
    const char * cx_g = (const char *)vgate + i02*nb02;
    const float * cx_u_b = bias_u ? (const float *)((const char *)bias_u + i02*bias_nb1) : nullptr;
    const float * cx_g_b = bias_g ? (const float *)((const char *)bias_g + i02*bias_nb1) : nullptr;
    const char * cy = (const char *)vy + i2*nb12;
    fused_mul_mat_vec_q<type, ncols_y, nwarps>(cx_u, cx_g, cx_u_b, cx_g_b, cy, (float *)cdst, ncols_x, nrows_x, nrows_y, nrows_dst, unary_op);
}

template <ggml_type type, int nwarps>
static void mul_mat_vec_q_cuda_T(const mmvq_args & args, hipStream_t stream) {

    GGML_ASSERT(args.ncols_x % ggml_blck_size(type) == 0);
    GGML_ASSERT(args.ncols_y <= MMVQ_MAX_BATCH_SIZE);

    int id = ggml_cuda_get_device();

    int64_t rows_per_cuda_block = ggml_cuda_info().devices[id].cc < CC_RDNA2 ?
        args.ncols_y < 4 ? 1 : 2 : 1;

    const int64_t nblocks = (args.nrows_x + rows_per_cuda_block - 1) / rows_per_cuda_block;
    const dim3 block_nums(nblocks, args.ne2, 1);
    const dim3 block_dims(WARP_SIZE, nwarps, 1);

    if (args.vx_u && args.vx_g && args.unary_op != GGML_UNARY_OP_COUNT) {
    switch (args.ncols_y) {
        case 1:
            fused_mul_mat_vec_q<type, 1, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vx_g, args.vy,
                    args.dst, args.ids_data, args.bias_u, args.bias_g, args.bias_nb1,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.unary_op);
            break;
        case 2:
            fused_mul_mat_vec_q<type, 2, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vx_g, args.vy,
                    args.dst, args.ids_data, args.bias_u, args.bias_g, args.bias_nb1,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.unary_op);
            break;
        case 3:
            fused_mul_mat_vec_q<type, 3, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vx_g, args.vy,
                    args.dst, args.ids_data, args.bias_u, args.bias_g, args.bias_nb1,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.unary_op);
            break;
        case 4:
            fused_mul_mat_vec_q<type, 4, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vx_g, args.vy,
                    args.dst, args.ids_data, args.bias_u, args.bias_g, args.bias_nb1,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.unary_op);
            break;
        case 5:
            fused_mul_mat_vec_q<type, 5, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vx_g, args.vy,
                    args.dst, args.ids_data, args.bias_u, args.bias_g, args.bias_nb1,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.unary_op);
            break;
        case 6:
            fused_mul_mat_vec_q<type, 6, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vx_g, args.vy,
                    args.dst, args.ids_data, args.bias_u, args.bias_g, args.bias_nb1,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.unary_op);
            break;
        case 7:
            fused_mul_mat_vec_q<type, 7, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vx_g, args.vy,
                    args.dst, args.ids_data, args.bias_u, args.bias_g, args.bias_nb1,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.unary_op);
            break;
        case 8:
            fused_mul_mat_vec_q<type, 8, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vx_g, args.vy,
                    args.dst, args.ids_data, args.bias_u, args.bias_g, args.bias_nb1,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.unary_op);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
    } else {
    switch (args.ncols_y) {
        case 1:
            mul_mat_vec_q<type, 1, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vy, args.dst, args.ids_data, args.bias_u,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.bias_nb1);
            break;
        case 2:
            mul_mat_vec_q<type, 2, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vy, args.dst, args.ids_data, args.bias_u,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.bias_nb1);
            break;
        case 3:
            mul_mat_vec_q<type, 3, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vy, args.dst, args.ids_data, args.bias_u,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.bias_nb1);
            break;
        case 4:
            mul_mat_vec_q<type, 4, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vy, args.dst, args.ids_data, args.bias_u,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.bias_nb1);
            break;
        case 5:
            mul_mat_vec_q<type, 5, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vy, args.dst, args.ids_data, args.bias_u,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.bias_nb1);
            break;
        case 6:
            mul_mat_vec_q<type, 6, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vy, args.dst, args.ids_data, args.bias_u,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.bias_nb1);
            break;
        case 7:
            mul_mat_vec_q<type, 7, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vy, args.dst, args.ids_data, args.bias_u,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.bias_nb1);
            break;
        case 8:
            mul_mat_vec_q<type, 8, nwarps><<<block_nums, block_dims, 0, stream>>>(args.vx_u, args.vy, args.dst, args.ids_data, args.bias_u,
                    args.ncols_x, args.nrows_x, args.nrows_y, args.nrows_dst, args.nb02, args.nb12, args.nb2, args.ids_nb0, args.bias_nb1);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }
    }
}

template <ggml_type type>
static void mul_mat_vec_q_cuda(const mmvq_args & args, hipStream_t stream) {
    int nwarps = 1;
    int id = ggml_cuda_get_device();
    if (args.ne2 < 2 && ggml_cuda_info().devices[id].cc < CC_RDNA2) { // NVIDIA and AMD older than RDNA2
        nwarps = args.ncols_y <= 4 ? 4 : 2;
    }
    switch (nwarps) {
        case 1:
            mul_mat_vec_q_cuda_T<type, 1>(args, stream);
            break;
        case 2:
            mul_mat_vec_q_cuda_T<type, 2>(args, stream);
            break;
        default:
            mul_mat_vec_q_cuda_T<type, 4>(args, stream);
    }
}

static void mul_mat_vec_q4_0_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q4_0>(args, stream);
}

static void mul_mat_vec_q4_1_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q4_1>(args, stream);
}

static void mul_mat_vec_q5_0_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q5_0>(args, stream);
}

static void mul_mat_vec_q5_1_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q5_1>(args, stream);
}

static void mul_mat_vec_q6_0_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q6_0>(args, stream);
}

static void mul_mat_vec_q8_0_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q8_0>(args, stream);
}

static void mul_mat_vec_q2_K_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q2_K>(args, stream);
}

static void mul_mat_vec_q3_K_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q3_K>(args, stream);
}

static void mul_mat_vec_q4_K_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q4_K>(args, stream);
}

static void mul_mat_vec_q5_K_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q5_K>(args, stream);
}

static void mul_mat_vec_q6_K_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_Q6_K>(args, stream);
}

static void mul_mat_vec_iq2_xxs_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ2_XXS>(args, stream);
}

static void mul_mat_vec_iq2_xs_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ2_XS>(args, stream);
}

static void mul_mat_vec_iq2_s_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ2_S>(args, stream);
}

static void mul_mat_vec_iq3_xxs_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ3_XXS>(args, stream);
}

static void mul_mat_vec_iq1_s_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ1_S>(args, stream);
}

static void mul_mat_vec_iq1_m_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ1_M>(args, stream);
}

static void mul_mat_vec_iq4_nl_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ4_NL>(args, stream);
}

static void mul_mat_vec_mxfp4_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_MXFP4>(args, stream);
}

static void mul_mat_vec_iq4_xs_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ4_XS>(args, stream);
}

static void mul_mat_vec_iq3_s_q8_1_cuda(const mmvq_args & args, hipStream_t stream) {
    mul_mat_vec_q_cuda<GGML_TYPE_IQ3_S>(args, stream);
}

static void ggml_cuda_op_mul_mat_vec_q_impl(ggml_backend_cuda_context & ctx, ggml_type type,
        const int64_t ne00, const int64_t ne0, const int64_t ne2,
        const int64_t nb02, const int64_t nb12, const int64_t nb2, const int64_t ids_nb0, const int64_t bias_nb1,
        const char * src0_dd_u, const char * src0_dd_g, const char * src1_ddq_i, float * dst_dd_i, const char * ids_data,
        const void * bias_u, const void * bias_g,
        const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
        const int64_t src1_padded_row_size, ggml_unary_op unary_op, hipStream_t stream) {

    const int64_t row_diff = row_high - row_low;

    int id = ggml_cuda_get_device();

    // the main device has a larger memory buffer to hold the results from all GPUs
    // nrows_dst == nrows of the matrix that the kernel writes into
    const int64_t nrows_dst = id == ctx.device ? ne0 : row_diff;

    mmvq_args args{/* vx_u     */ src0_dd_u,
                   /* vx_g     */ src0_dd_g,
                   /* bias_u   */ bias_u,
                   /* bias_g   */ bias_g,
                   /* vy       */ src1_ddq_i,
                   /* dst      */ dst_dd_i,
                   /* ids_data */ ids_data,
                   /* ncols_x  */ int(ne00),
                   /* nrows_x  */ int(row_diff),
                   /* nrows_y  */ int(src1_padded_row_size),
                   /* ncols_y  */ int(src1_ncols),
                   /* nrows_dst*/ int(nrows_dst),
                   /* ne2      */ int(ne2),
                   /* nb02     */ uint64_t(nb02),
                   /* nb12     */ uint64_t(nb12),
                   /* nb2      */ uint64_t(nb2),
                   /* ids_nb0  */ uint64_t(ids_nb0),
                   /* bias_nb1 */ uint64_t(bias_nb1),
                   /* unary_op */ unary_op
    };

    switch (type) {
        case GGML_TYPE_Q4_0:
            mul_mat_vec_q4_0_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q4_1:
            mul_mat_vec_q4_1_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q5_0:
            mul_mat_vec_q5_0_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q5_1:
            mul_mat_vec_q5_1_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q6_0:
            mul_mat_vec_q6_0_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q8_0:
            mul_mat_vec_q8_0_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q2_K:
            mul_mat_vec_q2_K_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q3_K:
            mul_mat_vec_q3_K_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q4_K:
            mul_mat_vec_q4_K_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q5_K:
            mul_mat_vec_q5_K_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_Q6_K:
            mul_mat_vec_q6_K_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_XXS:
            mul_mat_vec_iq2_xxs_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_XS:
            mul_mat_vec_iq2_xs_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ2_S:
            mul_mat_vec_iq2_s_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ3_XXS:
            mul_mat_vec_iq3_xxs_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ3_S:
            mul_mat_vec_iq3_s_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ1_S:
            mul_mat_vec_iq1_s_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ1_M:
            mul_mat_vec_iq1_m_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ4_NL:
            mul_mat_vec_iq4_nl_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_MXFP4:
            mul_mat_vec_mxfp4_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ4_XS:
            mul_mat_vec_iq4_xs_q8_1_cuda(args, stream);
            break;
        case GGML_TYPE_IQ1_BN:
        case GGML_TYPE_IQ2_BN:
        case GGML_TYPE_IQ2_K:
        case GGML_TYPE_IQ3_K:
        case GGML_TYPE_IQ2_KL:
        case GGML_TYPE_IQ3_KS:
        case GGML_TYPE_IQ4_K:
        case GGML_TYPE_IQ4_KS:
        case GGML_TYPE_IQ4_KSS:
        case GGML_TYPE_IQ1_KT:
        case GGML_TYPE_IQ2_KT:
        case GGML_TYPE_IQ3_KT:
        case GGML_TYPE_IQ4_KT:
        case GGML_TYPE_IQ2_KS:
        case GGML_TYPE_IQ5_K:
        case GGML_TYPE_IQ5_KS:
        case GGML_TYPE_IQ6_K:
        case GGML_TYPE_IQ2_K_R4:
        case GGML_TYPE_IQ3_K_R4:
        case GGML_TYPE_IQ4_K_R4:
        case GGML_TYPE_IQ4_KS_R4:
        case GGML_TYPE_IQ5_K_R4:
        case GGML_TYPE_IQ5_KS_R4:
        case GGML_TYPE_IQ1_S_R4:
        case GGML_TYPE_IQ1_M_R4:
            iqk_mul_mat_vec_q(type, args, stream);
            break;
        default:
            GGML_ABORT("fatal error");
            break;
    }

}

void ggml_cuda_op_mul_mat_vec_q_3D(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t ne10 = src1->ne[0];
    GGML_ASSERT(ne10 % QK8_1 == 0);
    GGML_ASSERT(src0->ne[3] == 1 && src1->ne[3] == 1 && dst->ne[3] == 1);
    GGML_ASSERT(src0->ne[2] == src1->ne[2] && src0->ne[2] == dst->ne[2]);

    const int64_t ne0 = dst->ne[0];

    const int64_t src1_row_size = ggml_row_size(GGML_TYPE_Q8_1, src1_padded_row_size);

    ggml_cuda_op_mul_mat_vec_q_impl(ctx, src0->type,
        ne00, ne0, dst->ne[2],
        src0->nb[2], src1_row_size, dst->nb[2], 0, 0,
        src0_dd_i, nullptr, src1_ddq_i, dst_dd_i, nullptr, nullptr, nullptr,
        row_low, row_high, src1_ncols,
        src1_padded_row_size, GGML_UNARY_OP_COUNT, stream);

    GGML_UNUSED(src1_ddf_i);
}

void ggml_cuda_op_mul_mat_vec_q(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, ggml_tensor * dst, const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t ne10 = src1->ne[0];
    GGML_ASSERT(ne10 % QK8_1 == 0);

    const int64_t ne0 = dst->ne[0];

    ggml_cuda_op_mul_mat_vec_q_impl(ctx, src0->type,
        ne00, ne0, 1, 0, 0, 0, 0, 0,
        src0_dd_i, nullptr, src1_ddq_i, dst_dd_i, nullptr, nullptr, nullptr,
        row_low, row_high, src1_ncols,
        src1_padded_row_size, GGML_UNARY_OP_COUNT, stream);

    GGML_UNUSED(src1_ddf_i);
}

void ggml_cuda_op_mul_mat_vec_q_id(
    ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, const ggml_tensor * ids, ggml_tensor * dst,
    const ggml_tensor * bias,
    const char * src0_dd_i, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, hipStream_t stream) {

    const int64_t ne00 = src0->ne[0];
    const int64_t ne10 = src1->ne[0];
    GGML_ASSERT(ne10 % QK8_1 == 0);
    GGML_ASSERT(src0->ne[3] == 1 && src1->ne[3] == 1 && dst->ne[3] == 1);
    GGML_ASSERT(src1->ne[1] == 1 && src1->ne[2] == 1);
    GGML_ASSERT(ids->ne[0] == dst->ne[2]);

    const int64_t ne0 = dst->ne[0];

    if (bias) {
        GGML_ASSERT(bias->type == GGML_TYPE_F32);
        GGML_ASSERT(bias->ne[0] == ne0);
        if (ids) {
            //GGML_ASSERT(bias->ne[1] == src0->ne[2]);
            GGML_ASSERT(bias->ne[2] == 1 && bias->ne[3] == 1);
        } else {
            GGML_ASSERT(ggml_nrows(bias) == 1);
        }
    }

    ggml_cuda_op_mul_mat_vec_q_impl(ctx, src0->type,
        ne00, ne0, dst->ne[2],
        src0->nb[2], src1->nb[2], dst->nb[2], ids->nb[0], bias ? bias->nb[1] : 0,
        src0_dd_i, nullptr, src1_ddq_i, dst_dd_i, (const char *)ids->data, bias ? bias->data : nullptr, nullptr,
        row_low, row_high, src1_ncols,
        src1_padded_row_size, GGML_UNARY_OP_COUNT, stream);

    GGML_UNUSED(src1_ddf_i);
}

void ggml_cuda_op_fused_mul_mat_vec_q_id(ggml_backend_cuda_context & ctx,
    const ggml_tensor * src0, const ggml_tensor * src1, const ggml_tensor * ids, ggml_tensor * dst,
    const ggml_tensor * bias_u, const ggml_tensor * bias_g,
    const char * src0_dd_u, const char * src0_dd_g, const float * src1_ddf_i,
    const char * src1_ddq_i, float * dst_dd_i, const int64_t row_low, const int64_t row_high, const int64_t src1_ncols,
    const int64_t src1_padded_row_size, ggml_unary_op unary_op, hipStream_t stream) {

    if (!bias_u && !bias_g) {
        GGML_ASSERT(unary_op == GGML_UNARY_OP_SILU ||
                    unary_op == GGML_UNARY_OP_RELU ||
                    unary_op == GGML_UNARY_OP_GELU);
    } else {
        GGML_ASSERT(unary_op == GGML_UNARY_OP_SWIGLU_OAI);
        GGML_ASSERT(bias_u && bias_g);
        GGML_ASSERT(bias_u->data && bias_g->data);
        GGML_ASSERT(bias_u->nb[1] == bias_g->nb[1]);
        GGML_ASSERT(bias_u->ne[0] == dst->ne[0]);
        GGML_ASSERT(bias_g->ne[0] == dst->ne[0]);
    }
    GGML_ASSERT(src0_dd_u && src0_dd_g);

    const int64_t ne00 = src0->ne[0];
    const int64_t ne10 = src1->ne[0];
    GGML_ASSERT(ne10 % QK8_1 == 0);
    GGML_ASSERT(src0->ne[3] == 1 && src1->ne[3] == 1 && dst->ne[3] == 1);
    GGML_ASSERT(src1->ne[1] == 1 && src1->ne[2] == 1);
    GGML_ASSERT(!ids || ids->ne[0] == dst->ne[2]);

    const int64_t ne0 = dst->ne[0];

    ggml_cuda_op_mul_mat_vec_q_impl(ctx, src0->type,
        ne00, ne0, dst->ne[2],
        src0->nb[2], src1->nb[2], dst->nb[2], ids ? ids->nb[0] : 0, bias_u ? bias_u->nb[1] : 0,
        src0_dd_u, src0_dd_g, src1_ddq_i, dst_dd_i, ids ? (const char *)ids->data : nullptr,
        bias_u ? bias_u->data : nullptr, bias_g ? bias_g->data : nullptr,
        row_low, row_high, src1_ncols,
        src1_padded_row_size, unary_op, stream);

    GGML_UNUSED(src1_ddf_i);
}


bool ggml_cuda_mmvq_type_supported(ggml_type src0_type) {
    switch (src0_type) {
        case GGML_TYPE_Q4_0:
        case GGML_TYPE_Q4_1:
        case GGML_TYPE_Q5_0:
        case GGML_TYPE_Q5_1:
        case GGML_TYPE_Q6_0:
        case GGML_TYPE_Q8_0:
        case GGML_TYPE_Q2_K:
        case GGML_TYPE_Q3_K:
        case GGML_TYPE_Q4_K:
        case GGML_TYPE_Q5_K:
        case GGML_TYPE_Q6_K:
        case GGML_TYPE_IQ2_XXS:
        case GGML_TYPE_IQ2_XS:
        case GGML_TYPE_IQ2_S:
        case GGML_TYPE_IQ3_XXS:
        case GGML_TYPE_IQ1_S:
        case GGML_TYPE_IQ1_M:
        case GGML_TYPE_IQ1_BN:
        case GGML_TYPE_IQ2_BN:
        case GGML_TYPE_IQ4_NL:
        case GGML_TYPE_MXFP4:
        case GGML_TYPE_IQ4_XS:
        case GGML_TYPE_IQ2_K:
        case GGML_TYPE_IQ2_KL:
        case GGML_TYPE_IQ3_KS:
        case GGML_TYPE_IQ3_K:
        case GGML_TYPE_IQ4_K:
        case GGML_TYPE_IQ4_KS:
        case GGML_TYPE_IQ4_KSS:
        case GGML_TYPE_IQ2_KS:
        case GGML_TYPE_IQ5_K:
        case GGML_TYPE_IQ5_KS:
        case GGML_TYPE_IQ6_K:
        case GGML_TYPE_IQ3_S:
        case GGML_TYPE_IQ2_K_R4:
        case GGML_TYPE_IQ3_K_R4:
        case GGML_TYPE_IQ4_K_R4:
        case GGML_TYPE_IQ4_KS_R4:
        case GGML_TYPE_IQ5_K_R4:
        case GGML_TYPE_IQ5_KS_R4:
        case GGML_TYPE_IQ1_S_R4:
        case GGML_TYPE_IQ1_M_R4:
        case GGML_TYPE_IQ1_KT:
        case GGML_TYPE_IQ2_KT:
        case GGML_TYPE_IQ3_KT:
        case GGML_TYPE_IQ4_KT:
            return true;
        default:
            return false;
    }
}
